/*
 * Copyright 2021-2024 The PySCF Developers. All Rights Reserved.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include <hip/hip_runtime.h>
#include <stdio.h>
#define THREADS        16

__global__
static void _block_diag(double *out, int m, int n, double *diags, int ndiags, int *offsets, int *rows, int *cols)
{
    int r = blockIdx.x;

    if (r >= ndiags){
        return;
    }
    int m0 = rows[r+1] - rows[r];
    int n0 = cols[r+1] - cols[r];
    int diag_offset = offsets[r];
    int row_offset = rows[r];
    int col_offset = cols[r];
    
    for (int i = threadIdx.y; i < m0; i += THREADS){
        for (int j = threadIdx.x; j < n0; j += THREADS){
            out[(i+row_offset)*n + (j+col_offset)] = diags[diag_offset + i*n0 + j];
        }
    }
}

extern "C" {
int block_diag(hipStream_t stream, double *out, int m, int n, double *diags, int ndiags, int *offsets, int *rows, int *cols)
{
    dim3 threads(THREADS, THREADS);
    dim3 blocks(ndiags);
    _block_diag<<<blocks, threads, 0, stream>>>(out, m, n, diags, ndiags, offsets, rows, cols);
    hipError_t err = hipGetLastError();
    if (err != hipSuccess) {
        fprintf(stderr, "CUDA Error in block_diag: %s\n", hipGetErrorString(err));
        return 1;
    }
    return 0;
}
}
