/*
 * gpu4pyscf is a plugin to use Nvidia GPU in PySCF package
 *
 * Copyright (C) 2022 Qiming Sun
 *
 * This program is free software: you can redistribute it and/or modify
 * it under the terms of the GNU General Public License as published by
 * the Free Software Foundation, either version 3 of the License, or
 * (at your option) any later version.
 *
 * This program is distributed in the hope that it will be useful,
 * but WITHOUT ANY WARRANTY; without even the implied warranty of
 * MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
 * GNU General Public License for more details.
 *
 * You should have received a copy of the GNU General Public License
 * along with this program.  If not, see <http://www.gnu.org/licenses/>.
 */

#include <stdio.h>
#include <hip/hip_runtime.h>

#define THREADS     16
#define BLOCK_DIM   16

__global__
void _add_sparse(double *a, const double *b, const int *indices, int n, int m, int k, int count)
{
	int row = blockIdx.x * BLOCK_DIM + threadIdx.x;
    int col = blockIdx.y * BLOCK_DIM + threadIdx.y;
    if (row >= k || col >= k){
        return;
    }
    int ix = indices[row];
    int iy = indices[col];
    if (ix < 0 || iy < 0){
        return;
    }
    int idx_a = ix * n + iy;
    int idx_b = row * m + col;
    for (int i = 0; i < count; i++){
        //a[idx_a + i*n*n] += b[idx_b + i*m*m];
        atomicAdd(a+idx_a+i*n*n, b[idx_b+i*m*m]);
    }
}

__global__
void _reduce_sparse(double *a, const double *b, const int *indices, int n, int m, int k, int count)
{
	int row = blockIdx.x * BLOCK_DIM + threadIdx.x;
    int col = blockIdx.y * BLOCK_DIM + threadIdx.y;
    if (row >= k || col >= k){
        return;
    }

    for (int i = 0; i < count; i++){
        int ix = indices[row + i * k];
        int iy = indices[col + i * k];
        if (ix < 0 || iy < 0){
            continue;
        }
        int idx_a = ix * n + iy;
        int idx_b = row * m + col;
        //a[idx_a + i*n*n] += b[idx_b + i*m*m];
        atomicAdd(a+idx_a, b[idx_b+i*m*m]);
    }
}

extern "C" {
__host__
int add_sparse(hipStream_t stream, double *a, const double *b, 
                const int *indices, int n, int m, int k, int count){
    int ntile = (k + THREADS - 1) / THREADS;
    dim3 threads(THREADS, THREADS);
    dim3 blocks(ntile, ntile);
    _add_sparse<<<blocks, threads, 0, stream>>>(a, b, indices, n, m, k, count);
    hipError_t err = hipGetLastError();
    if (err != hipSuccess) {
        return 1;
    }
    return 0;
}

__host__
int reduce_sparse(hipStream_t stream, double *a, const double *b, 
                const int *indices, int n, int m, int k, int count){
    int ntile = (k + THREADS - 1) / THREADS;
    dim3 threads(THREADS, THREADS);
    dim3 blocks(ntile, ntile);
    _reduce_sparse<<<blocks, threads, 0, stream>>>(a, b, indices, n, m, k, count);
    hipError_t err = hipGetLastError();
    if (err != hipSuccess) {
        return 1;
    }
    return 0;
}
}
