#include "hip/hip_runtime.h"
#include "rys_roots.cuh"

#define SQRTPIE4        .8862269254527580136
#define PIE4            .7853981633974483096

__device__
static void rys_roots(int nroots, double x, double *r, double *w, int block_size)
{
    if (x < 3.e-7){
        int off = nroots * (nroots - 1) / 2;
        for (int i = 0; i < nroots; ++i)  {
            r[i*block_size] = ROOT_SMALLX_R0[off+i] + ROOT_SMALLX_R1[off+i] * x;
            w[i*block_size] = ROOT_SMALLX_W0[off+i] + ROOT_SMALLX_W1[off+i] * x;
        }
        return;
    }

    if (x > 35+nroots*5) {
        int off = nroots * (nroots - 1) / 2;
        double t = sqrt(PIE4/x);
        for (int i = 0; i < nroots; ++i)  {
            r[i*block_size] = ROOT_LARGEX_R_DATA[off+i] / x;
            w[i*block_size] = ROOT_LARGEX_W_DATA[off+i] * t;
        }
        return;
    }

    if (nroots == 1) {
        double tt = sqrt(x);
        double fmt0 = SQRTPIE4 / tt * erf(tt);
        w[0] = fmt0;
        double e = exp(-x);
        double b = .5 / x;
        double fmt1 = b * (fmt0 - e);
        r[0] = fmt1 / fmt0;
        return;
    }

    double *datax = ROOT_RW_DATA + DEGREE1*INTERVALS * nroots*(nroots-1);
    int it = (int)(x * .4);
    double u = (x - it * 2.5) * 0.8 - 1.;
    double u2 = u * 2.;
    double *c = datax;
    for (int rt_id = 0; rt_id < nroots; ++rt_id) {
        //for i in range(2, degree + 1):
        //    c0, c1 = c[degree-i] - c1, c0 + c1*u2
        double c0 = c[it + DEGREE   *INTERVALS];
        double c1 = c[it +(DEGREE-1)*INTERVALS];
        double c2, c3;
#pragma unroll
        for (int n = DEGREE-2; n > 0; n-=2) {
            c2 = c[it + n   *INTERVALS] - c1;
            c3 = c0 + c1*u2;
            c1 = c2 + c3*u2;
            c0 = c[it +(n-1)*INTERVALS] - c3;
        }
        if (DEGREE % 2 == 0) {
            c2 = c[it] - c1;
            c3 = c0 + c1*u2;
            r[rt_id*block_size] = c2 + c3*u;
        } else {
            r[rt_id*block_size] = c0 + c1*u;
        }
        c += DEGREE1 * INTERVALS;

        //for i in range(2, degree + 1):
        //    c0, c1 = c[degree-i] - c1, c0 + c1*u2
        c0 = c[it + DEGREE   *INTERVALS];
        c1 = c[it +(DEGREE-1)*INTERVALS];
#pragma unroll
        for (int n = DEGREE-2; n > 0; n-=2) {
            c2 = c[it + n   *INTERVALS] - c1;
            c3 = c0 + c1*u2;
            c1 = c2 + c3*u2;
            c0 = c[it +(n-1)*INTERVALS] - c3;
        }
        if (DEGREE % 2 == 0) {
            c2 = c[it] - c1;
            c3 = c0 + c1*u2;
            w[rt_id*block_size] = c2 + c3*u;
        } else {
            w[rt_id*block_size] = c0 + c1*u;
        }
        c += DEGREE1 * INTERVALS;
    }
}
