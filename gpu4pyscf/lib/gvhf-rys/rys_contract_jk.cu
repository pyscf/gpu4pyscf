#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdint.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>

#include "vhf.cuh"
#include "rys_roots.cu"
#include "create_tasks.cu"

__device__
static void rys_jk_general(RysIntEnvVars envs, JKMatrix jk, BoundsInfo bounds,
                           ShellQuartet *shl_quartet_idx, int ntasks)
{
    // sq is short for shl_quartet
    int sq_id = threadIdx.x;
    int nsq_per_block = blockDim.x;
    int gout_id = threadIdx.y;
    int gout_stride = blockDim.y;
    int li = bounds.li;
    int lj = bounds.lj;
    int lk = bounds.lk;
    int ll = bounds.ll;
    int nfi = bounds.nfi;
    int nfk = bounds.nfk;
    int nfij = bounds.nfij;
    int nfkl = bounds.nfkl;
    int iprim = bounds.iprim;
    int jprim = bounds.jprim;
    int kprim = bounds.kprim;
    int lprim = bounds.lprim;
    int lij = li + lj;
    int lkl = lk + ll;
    int nroots = bounds.nroots;
    int stride_j = bounds.stride_j;
    int stride_k = bounds.stride_k;
    int stride_l = bounds.stride_l;
    int g_size = stride_l * (ll + 1);
    int *idx_ij = c_g_pair_idx + c_g_pair_offsets[li*LMAX1+lj];
    int *idy_ij = idx_ij + nfij;
    int *idz_ij = idy_ij + nfij;
    int *idx_kl = c_g_pair_idx + c_g_pair_offsets[lk*LMAX1+ll];
    int *idy_kl = idx_kl + nfkl;
    int *idz_kl = idy_kl + nfkl;
    int *bas = envs.bas;
    int *ao_loc = envs.ao_loc;
    int nbas = envs.nbas;
    int nao = ao_loc[nbas];
    //double *env = c_env;
    double *env = envs.env;
    double omega = env[PTR_RANGE_OMEGA];

    extern __shared__ double rw[];
    double *g = rw + nsq_per_block * nroots*2;
    double *Rpa_cicj = g + nsq_per_block * g_size*3;
    double Rqc[3], Rpq[3];
    double gout[GWIDTH];

    for (int task0 = 0; task0 < ntasks; task0 += nsq_per_block) {
        __syncthreads();
        int task_id = task0 + sq_id;
        double fac_sym = PI_FAC;
        ShellQuartet sq;
        if (task_id >= ntasks) {
            // To avoid __syncthreads blocking blocking idle warps, all remaining
            // threads compute a valid shell quartet with zero normalization factor
            sq = shl_quartet_idx[0];
            fac_sym = 0.;
        } else {
            sq = shl_quartet_idx[task_id];
        }
        int ish = sq.i;
        int jsh = sq.j;
        int ksh = sq.k;
        int lsh = sq.l;
        //int sh_ij = (ish % TILE) * TILE + (jsh % TILE);
        if (ish == jsh) fac_sym *= .5;
        if (ksh == lsh) fac_sym *= .5;
        if (ish*nbas+jsh == ksh*nbas+lsh) fac_sym *= .5;
        int i0 = ao_loc[ish];
        int j0 = ao_loc[jsh];
        int k0 = ao_loc[ksh];
        int l0 = ao_loc[lsh];
        double *expi = env + bas[ish*BAS_SLOTS+PTR_EXP];
        double *expj = env + bas[jsh*BAS_SLOTS+PTR_EXP];
        double *expk = env + bas[ksh*BAS_SLOTS+PTR_EXP];
        double *expl = env + bas[lsh*BAS_SLOTS+PTR_EXP];
        double *ci = env + bas[ish*BAS_SLOTS+PTR_COEFF];
        double *cj = env + bas[jsh*BAS_SLOTS+PTR_COEFF];
        double *ck = env + bas[ksh*BAS_SLOTS+PTR_COEFF];
        double *cl = env + bas[lsh*BAS_SLOTS+PTR_COEFF];
        double *ri = env + bas[ish*BAS_SLOTS+PTR_BAS_COORD];
        double *rj = env + bas[jsh*BAS_SLOTS+PTR_BAS_COORD];
        double *rk = env + bas[ksh*BAS_SLOTS+PTR_BAS_COORD];
        double *rl = env + bas[lsh*BAS_SLOTS+PTR_BAS_COORD];
        for (int ij = gout_id; ij < iprim*jprim; ij += gout_stride) {
            int ip = ij / jprim;
            int jp = ij % jprim;
            double ai = expi[ip];
            double aj = expj[jp];
            double aij = ai + aj;
            double aj_aij = aj / aij;
            double xjxi = rj[0] - ri[0];
            double yjyi = rj[1] - ri[1];
            double zjzi = rj[2] - ri[2];
            double *Rpa = Rpa_cicj + ij*4*nsq_per_block;
            Rpa[sq_id+0*nsq_per_block] = xjxi * aj_aij;
            Rpa[sq_id+1*nsq_per_block] = yjyi * aj_aij;
            Rpa[sq_id+2*nsq_per_block] = zjzi * aj_aij;
            double theta_ij = ai * aj / aij;
            double Kab = exp(-theta_ij * (xjxi*xjxi+yjyi*yjyi+zjzi*zjzi));
            Rpa[sq_id+3*nsq_per_block] = fac_sym * ci[ip] * cj[jp] * Kab;
        }
        for (int gout_start = 0; gout_start < nfij*nfkl; gout_start+=gout_stride*GWIDTH) {
#pragma unroll
        for (int n = 0; n < GWIDTH; ++n) { gout[n] = 0; }

        for (int klp = 0; klp < kprim*lprim; ++klp) {
            int kp = klp / lprim;
            int lp = klp % lprim;
            double ak = expk[kp];
            double al = expl[lp];
            double akl = ak + al;
            double al_akl = al / akl;
            double xlxk = rl[0] - rk[0];
            double ylyk = rl[1] - rk[1];
            double zlzk = rl[2] - rk[2];
            Rqc[0] = xlxk * al_akl; // (ak*xk+al*xl)/akl
            Rqc[1] = ylyk * al_akl;
            Rqc[2] = zlzk * al_akl;
            __syncthreads();
            if (gout_id == 0) {
                double theta_kl = ak * al / akl;
                double Kcd = exp(-theta_kl * (xlxk*xlxk+ylyk*ylyk+zlzk*zlzk));
                double ckcl = ck[kp] * cl[lp] * Kcd;
                g[sq_id] = ckcl;
            }
            for (int ijp = 0; ijp < iprim*jprim; ++ijp) {
                int ip = ijp / jprim;
                int jp = ijp % jprim;
                double ai = expi[ip];
                double aj = expj[jp];
                double aij = ai + aj;
                double *Rpa = Rpa_cicj + ijp*4*nsq_per_block;
                double xij = ri[0] + Rpa[sq_id+0*nsq_per_block];
                double yij = ri[1] + Rpa[sq_id+1*nsq_per_block];
                double zij = ri[2] + Rpa[sq_id+2*nsq_per_block];
                double xkl = rk[0] + Rqc[0];
                double ykl = rk[1] + Rqc[1];
                double zkl = rk[2] + Rqc[2];
                double xpq = xij - xkl;
                double ypq = yij - ykl;
                double zpq = zij - zkl;
                Rpq[0] = xpq;
                Rpq[1] = ypq;
                Rpq[2] = zpq;
                __syncthreads();
                if (gout_id == 0) {
                    double cicj = Rpa[sq_id+3*nsq_per_block];
                    g[sq_id + g_size * nsq_per_block] = cicj / (aij*akl*sqrt(aij+akl));
                }
                double rr = xpq*xpq + ypq*ypq + zpq*zpq;
                double theta = aij * akl / (aij + akl);
                double theta_rr = theta * rr;
                if (omega == 0) {
                    rys_roots(nroots, theta_rr, rw);
                } else {
                    double theta_fac = omega * omega / (omega * omega + theta);
                    rys_roots(nroots, theta_fac*theta_rr, rw);
                    __syncthreads();
                    double sqrt_theta_fac = sqrt(theta_fac);
                    for (int irys = gout_id; irys < nroots; irys+=gout_stride) {
                        rw[sq_id+ irys*2   *nsq_per_block] *= theta_fac;
                        rw[sq_id+(irys*2+1)*nsq_per_block] *= sqrt_theta_fac;
                    }
                }
                double s0x, s1x, s2x;
                for (int irys = 0; irys < nroots; ++irys) {
                    __syncthreads();
                    if (gout_id == 0) {
                        g[sq_id + 2*g_size*nsq_per_block] = rw[sq_id+(irys*2+1)*nsq_per_block];
                    }
                    double rt = rw[sq_id + irys*2*nsq_per_block];
                    double rt_aa = rt / (aij + akl);
                    double rt_aij = rt_aa * akl;
                    double rt_akl = rt_aa * aij;
                    double b00 = .5 * rt_aa;
                    double b10 = .5/aij * (1 - rt_aij);
                    double b01 = .5/akl * (1 - rt_akl);

                    // TRR
                    //for i in range(lij):
                    //    trr(i+1,0) = c0 * trr(i,0) + i*b10 * trr(i-1,0)
                    //for k in range(lkl):
                    //    for i in range(lij+1):
                    //        trr(i,k+1) = c0p * trr(i,k) + k*b01 * trr(i,k-1) + i*b00 * trr(i-1,k)
                    if (lij > 0) {
                        __syncthreads();
                        // gx(0,n+1) = c0*gx(0,n) + n*b10*gx(0,n-1)
                        for (int n = gout_id; n < 3; n += gout_stride) {
                            double *_gx = g + n * g_size * nsq_per_block;
                            int ir = sq_id + n * nsq_per_block;
                            double c0x = Rpa[ir] - rt_aij * Rpq[n];
                            s0x = _gx[sq_id];
                            s1x = c0x * s0x;
                            _gx[sq_id + nsq_per_block] = s1x;
                            for (int i = 1; i < lij; ++i) {
                                s2x = c0x * s1x + i * b10 * s0x;
                                _gx[sq_id + (i+1)*nsq_per_block] = s2x;
                                s0x = s1x;
                                s1x = s2x;
                            }
                        }
                    }

                    if (lkl > 0) {
                        int lij3 = (lij+1)*3;
                        for (int n = gout_id; n < lij3+gout_id; n += gout_stride) {
                            __syncthreads();
                            int i = n / 3; //for i in range(lij+1):
                            int _ix = n % 3; // TODO: remove _ix for nroots > 2
                            double *_gx = g + (i + _ix * g_size) * nsq_per_block;
                            double cpx = Rqc[_ix] + rt_akl * Rpq[_ix];
                            //for i in range(lij+1):
                            //    trr(i,1) = c0p * trr(i,0) + i*b00 * trr(i-1,0)
                            if (n < lij3) {
                                s0x = _gx[sq_id];
                                s1x = cpx * s0x;
                                if (i > 0) {
                                    s1x += i * b00 * _gx[sq_id-nsq_per_block];
                                }
                                _gx[sq_id + stride_k*nsq_per_block] = s1x;
                            }

                            //for k in range(1, lkl):
                            //    for i in range(lij+1):
                            //        trr(i,k+1) = cp * trr(i,k) + k*b01 * trr(i,k-1) + i*b00 * trr(i-1,k)
                            for (int k = 1; k < lkl; ++k) {
                                __syncthreads();
                                if (n < lij3) {
                                    s2x = cpx*s1x + k*b01*s0x;
                                    if (i > 0) {
                                        s2x += i * b00 * _gx[sq_id + (k*stride_k-1)*nsq_per_block];
                                    }
                                    _gx[sq_id + (k*stride_k+stride_k)*nsq_per_block] = s2x;
                                    s0x = s1x;
                                    s1x = s2x;
                                }
                            }
                        }
                    }

                    // hrr
                    // g(i,j+1) = rirj * g(i,j) +  g(i+1,j)
                    // g(...,k,l+1) = rkrl * g(...,k,l) + g(...,k+1,l)
                    if (lj > 0) {
                        __syncthreads();
                        if (task_id < ntasks) {
                            int lkl3 = (lkl+1)*3;
                            for (int m = gout_id; m < lkl3; m += gout_stride) {
                                int k = m / 3;
                                int _ix = m % 3;
                                double xixj = ri[_ix] - rj[_ix];
                                double *_gx = g + (_ix*g_size + k*stride_k) * nsq_per_block;
                                for (int j = 0; j < lj; ++j) {
                                    int ij = (lij-j) + j*stride_j;
                                    s1x = _gx[sq_id + ij*nsq_per_block];
                                    for (--ij; ij >= j*stride_j; --ij) {
                                        s0x = _gx[sq_id + ij*nsq_per_block];
                                        _gx[sq_id + (ij+stride_j)*nsq_per_block] = xixj * s0x + s1x;
                                        s1x = s0x;
                                    }
                                }
                            }
                        }
                    }
                    if (ll > 0) {
                        __syncthreads();
                        if (task_id < ntasks) {
                            for (int n = gout_id; n < stride_k*3; n += gout_stride) {
                                int i = n / 3;
                                int _ix = n % 3;
                                double xkxl = rk[_ix] - rl[_ix];
                                double *_gx = g + (_ix*g_size + i) * nsq_per_block;
                                for (int l = 0; l < ll; ++l) {
                                    int kl = (lkl-l)*stride_k + l*stride_l;
                                    s1x = _gx[sq_id + kl*nsq_per_block];
                                    for (kl-=stride_k; kl >= l*stride_l; kl-=stride_k) {
                                        s0x = _gx[sq_id + kl*nsq_per_block];
                                        _gx[sq_id + (kl+stride_l)*nsq_per_block] = xkxl * s0x + s1x;
                                        s1x = s0x;
                                    }
                                }
                            }
                        }
                    }

                    __syncthreads();
                    double *gx = g;
                    double *gy = gx + nsq_per_block * g_size;
                    double *gz = gy + nsq_per_block * g_size;
#pragma unroll
                    for (int n = 0; n < GWIDTH; ++n) {
                        int ijkl = (gout_start + n*gout_stride+gout_id);
                        int kl = ijkl / nfij;
                        int ij = ijkl % nfij;
                        if (kl >= nfkl) break;
                        int addrx = sq_id + (idx_ij[ij] + idx_kl[kl] * stride_k) * nsq_per_block;
                        int addry = sq_id + (idy_ij[ij] + idy_kl[kl] * stride_k) * nsq_per_block;
                        int addrz = sq_id + (idz_ij[ij] + idz_kl[kl] * stride_k) * nsq_per_block;
                        gout[n] += gx[addrx] * gy[addry] * gz[addrz];
                    }
                }
            }
        }
        if (task_id >= ntasks) {
            continue;
        }
        double *dm = jk.dm;
        double *vj = jk.vj;
        double *vk = jk.vk;
        int do_j = vj != NULL;
        int do_k = vk != NULL;
        for (int i_dm = 0; i_dm < jk.n_dm; ++i_dm) {
#pragma unroll
            for (int n = 0; n < GWIDTH; ++n) {
                int ijkl = (gout_start + n*gout_stride+gout_id);
                int kl = ijkl / nfij;
                int ij = ijkl % nfij;
                if (kl >= nfkl) break;
                double s = gout[n];
                int i = ij % nfi;
                int j = ij / nfi;
                int k = kl % nfk;
                int l = kl / nfk;
                int _i = i + i0;
                int _j = j + j0;
                int _k = k + k0;
                int _l = l + l0;
                if (do_j) {
                    int _ji = _j*nao+_i;
                    int _lk = _l*nao+_k;
                    atomicAdd(vj+_lk, s * dm[_ji]);
                    atomicAdd(vj+_ji, s * dm[_lk]);
                }
                if (do_k) {
                    int _jl = _j*nao+_l;
                    int _jk = _j*nao+_k;
                    int _il = _i*nao+_l;
                    int _ik = _i*nao+_k;
                    atomicAdd(vk+_ik, s * dm[_jl]);
                    atomicAdd(vk+_il, s * dm[_jk]);
                    atomicAdd(vk+_jk, s * dm[_il]);
                    atomicAdd(vk+_jl, s * dm[_ik]);
                }
            }
            vj += nao * nao;
            vk += nao * nao;
            dm += nao * nao;
        }
    } }
}

__device__
static void rys_sr_jk_general(RysIntEnvVars envs, JKMatrix jk, BoundsInfo bounds,
                              ShellQuartet *shl_quartet_idx, int ntasks)
{
    // sq is short for shl_quartet
    int sq_id = threadIdx.x;
    int nsq_per_block = blockDim.x;
    int gout_id = threadIdx.y;
    int gout_stride = blockDim.y;
    int li = bounds.li;
    int lj = bounds.lj;
    int lk = bounds.lk;
    int ll = bounds.ll;
    int nfi = bounds.nfi;
    int nfk = bounds.nfk;
    int nfij = bounds.nfij;
    int nfkl = bounds.nfkl;
    int iprim = bounds.iprim;
    int jprim = bounds.jprim;
    int kprim = bounds.kprim;
    int lprim = bounds.lprim;
    int lij = li + lj;
    int lkl = lk + ll;
    int nroots = bounds.nroots;
    int stride_j = bounds.stride_j;
    int stride_k = bounds.stride_k;
    int stride_l = bounds.stride_l;
    int g_size = stride_l * (ll + 1);
    int *idx_ij = c_g_pair_idx + c_g_pair_offsets[li*LMAX1+lj];
    int *idy_ij = idx_ij + nfij;
    int *idz_ij = idy_ij + nfij;
    int *idx_kl = c_g_pair_idx + c_g_pair_offsets[lk*LMAX1+ll];
    int *idy_kl = idx_kl + nfkl;
    int *idz_kl = idy_kl + nfkl;
    int *bas = envs.bas;
    int *ao_loc = envs.ao_loc;
    int nbas = envs.nbas;
    int nao = ao_loc[nbas];
    //double *env = c_env;
    double *env = envs.env;
    double omega = env[PTR_RANGE_OMEGA];

    extern __shared__ double rw[];
    double *g = rw + nsq_per_block * nroots*2;
    double *Rpa_cicj = g + nsq_per_block * g_size*3;
    double Rqc[3], Rpq[3];
    double gout[GWIDTH];

    for (int task0 = 0; task0 < ntasks; task0 += nsq_per_block) {
        __syncthreads();
        int task_id = task0 + sq_id;
        double fac_sym = PI_FAC;
        ShellQuartet sq;
        if (task_id >= ntasks) {
            // To avoid __syncthreads blocking blocking idle warps, all remaining
            // threads compute a valid shell quartet with zero normalization factor
            sq = shl_quartet_idx[0];
            fac_sym = 0.;
        } else {
            sq = shl_quartet_idx[task_id];
        }
        int ish = sq.i;
        int jsh = sq.j;
        int ksh = sq.k;
        int lsh = sq.l;
        //int sh_ij = (ish % TILE) * TILE + (jsh % TILE);
        if (ish == jsh) fac_sym *= .5;
        if (ksh == lsh) fac_sym *= .5;
        if (ish*nbas+jsh == ksh*nbas+lsh) fac_sym *= .5;
        int i0 = ao_loc[ish];
        int j0 = ao_loc[jsh];
        int k0 = ao_loc[ksh];
        int l0 = ao_loc[lsh];
        double *expi = env + bas[ish*BAS_SLOTS+PTR_EXP];
        double *expj = env + bas[jsh*BAS_SLOTS+PTR_EXP];
        double *expk = env + bas[ksh*BAS_SLOTS+PTR_EXP];
        double *expl = env + bas[lsh*BAS_SLOTS+PTR_EXP];
        double *ci = env + bas[ish*BAS_SLOTS+PTR_COEFF];
        double *cj = env + bas[jsh*BAS_SLOTS+PTR_COEFF];
        double *ck = env + bas[ksh*BAS_SLOTS+PTR_COEFF];
        double *cl = env + bas[lsh*BAS_SLOTS+PTR_COEFF];
        double *ri = env + bas[ish*BAS_SLOTS+PTR_BAS_COORD];
        double *rj = env + bas[jsh*BAS_SLOTS+PTR_BAS_COORD];
        double *rk = env + bas[ksh*BAS_SLOTS+PTR_BAS_COORD];
        double *rl = env + bas[lsh*BAS_SLOTS+PTR_BAS_COORD];
        for (int ij = gout_id; ij < iprim*jprim; ij += gout_stride) {
            int ip = ij / jprim;
            int jp = ij % jprim;
            double ai = expi[ip];
            double aj = expj[jp];
            double aij = ai + aj;
            double aj_aij = aj / aij;
            double xjxi = rj[0] - ri[0];
            double yjyi = rj[1] - ri[1];
            double zjzi = rj[2] - ri[2];
            double *Rpa = Rpa_cicj + ij*4*nsq_per_block;
            Rpa[sq_id+0*nsq_per_block] = xjxi * aj_aij;
            Rpa[sq_id+1*nsq_per_block] = yjyi * aj_aij;
            Rpa[sq_id+2*nsq_per_block] = zjzi * aj_aij;
            double theta_ij = ai * aj / aij;
            double Kab = exp(-theta_ij * (xjxi*xjxi+yjyi*yjyi+zjzi*zjzi));
            Rpa[sq_id+3*nsq_per_block] = fac_sym * ci[ip] * cj[jp] * Kab;
        }
        for (int gout_start = 0; gout_start < nfij*nfkl; gout_start+=gout_stride*GWIDTH) {
#pragma unroll
        for (int n = 0; n < GWIDTH; ++n) { gout[n] = 0; }

        for (int klp = 0; klp < kprim*lprim; ++klp) {
            int kp = klp / lprim;
            int lp = klp % lprim;
            double ak = expk[kp];
            double al = expl[lp];
            double akl = ak + al;
            double al_akl = al / akl;
            double xlxk = rl[0] - rk[0];
            double ylyk = rl[1] - rk[1];
            double zlzk = rl[2] - rk[2];
            Rqc[0] = xlxk * al_akl; // (ak*xk+al*xl)/akl
            Rqc[1] = ylyk * al_akl;
            Rqc[2] = zlzk * al_akl;
            __syncthreads();
            if (gout_id == 0) {
                double theta_kl = ak * al / akl;
                double Kcd = exp(-theta_kl * (xlxk*xlxk+ylyk*ylyk+zlzk*zlzk));
                double ckcl = ck[kp] * cl[lp] * Kcd;
                g[sq_id] = ckcl;
            }
            for (int ijp = 0; ijp < iprim*jprim; ++ijp) {
                int ip = ijp / jprim;
                int jp = ijp % jprim;
                double ai = expi[ip];
                double aj = expj[jp];
                double aij = ai + aj;
                double *Rpa = Rpa_cicj + ijp*4*nsq_per_block;
                double xij = ri[0] + Rpa[sq_id+0*nsq_per_block];
                double yij = ri[1] + Rpa[sq_id+1*nsq_per_block];
                double zij = ri[2] + Rpa[sq_id+2*nsq_per_block];
                double xkl = rk[0] + Rqc[0];
                double ykl = rk[1] + Rqc[1];
                double zkl = rk[2] + Rqc[2];
                double xpq = xij - xkl;
                double ypq = yij - ykl;
                double zpq = zij - zkl;
                Rpq[0] = xpq;
                Rpq[1] = ypq;
                Rpq[2] = zpq;
                __syncthreads();
                if (gout_id == 0) {
                    double cicj = Rpa[sq_id+3*nsq_per_block];
                    g[sq_id + g_size * nsq_per_block] = cicj / (aij*akl*sqrt(aij+akl));
                }
                double rr = xpq*xpq + ypq*ypq + zpq*zpq;
                double theta = aij * akl / (aij + akl);
                double theta_rr = theta * rr;
                int _nroots = nroots/2;
                rys_roots(_nroots, theta_rr, rw+nroots*nsq_per_block);
                double theta_fac = omega * omega / (omega * omega + theta);
                rys_roots(_nroots, theta_fac*theta_rr, rw);
                __syncthreads();
                double sqrt_theta_fac = -sqrt(theta_fac);
                for (int irys = gout_id; irys < _nroots; irys+=gout_stride) {
                    rw[sq_id+ irys*2   *nsq_per_block] *= theta_fac;
                    rw[sq_id+(irys*2+1)*nsq_per_block] *= sqrt_theta_fac;
                }
                double s0x, s1x, s2x;
                for (int irys = 0; irys < nroots; ++irys) {
                    __syncthreads();
                    if (gout_id == 0) {
                        g[sq_id + 2*g_size*nsq_per_block] = rw[sq_id+(irys*2+1)*nsq_per_block];
                    }
                    double rt = rw[sq_id + irys*2*nsq_per_block];
                    double rt_aa = rt / (aij + akl);
                    double rt_aij = rt_aa * akl;
                    double rt_akl = rt_aa * aij;
                    double b00 = .5 * rt_aa;
                    double b10 = .5/aij * (1 - rt_aij);
                    double b01 = .5/akl * (1 - rt_akl);

                    // TRR
                    //for i in range(lij):
                    //    trr(i+1,0) = c0 * trr(i,0) + i*b10 * trr(i-1,0)
                    //for k in range(lkl):
                    //    for i in range(lij+1):
                    //        trr(i,k+1) = c0p * trr(i,k) + k*b01 * trr(i,k-1) + i*b00 * trr(i-1,k)
                    if (lij > 0) {
                        __syncthreads();
                        // gx(0,n+1) = c0*gx(0,n) + n*b10*gx(0,n-1)
                        for (int n = gout_id; n < 3; n += gout_stride) {
                            double *_gx = g + n * g_size * nsq_per_block;
                            int ir = sq_id + n * nsq_per_block;
                            double c0x = Rpa[ir] - rt_aij * Rpq[n];
                            s0x = _gx[sq_id];
                            s1x = c0x * s0x;
                            _gx[sq_id + nsq_per_block] = s1x;
                            for (int i = 1; i < lij; ++i) {
                                s2x = c0x * s1x + i * b10 * s0x;
                                _gx[sq_id + (i+1)*nsq_per_block] = s2x;
                                s0x = s1x;
                                s1x = s2x;
                            }
                        }
                    }

                    if (lkl > 0) {
                        int lij3 = (lij+1)*3;
                        for (int n = gout_id; n < lij3+gout_id; n += gout_stride) {
                            __syncthreads();
                            int i = n / 3; //for i in range(lij+1):
                            int _ix = n % 3; // TODO: remove _ix for nroots > 2
                            double *_gx = g + (i + _ix * g_size) * nsq_per_block;
                            double cpx = Rqc[_ix] + rt_akl * Rpq[_ix];
                            //for i in range(lij+1):
                            //    trr(i,1) = c0p * trr(i,0) + i*b00 * trr(i-1,0)
                            if (n < lij3) {
                                s0x = _gx[sq_id];
                                s1x = cpx * s0x;
                                if (i > 0) {
                                    s1x += i * b00 * _gx[sq_id-nsq_per_block];
                                }
                                _gx[sq_id + stride_k*nsq_per_block] = s1x;
                            }

                            //for k in range(1, lkl):
                            //    for i in range(lij+1):
                            //        trr(i,k+1) = cp * trr(i,k) + k*b01 * trr(i,k-1) + i*b00 * trr(i-1,k)
                            for (int k = 1; k < lkl; ++k) {
                                __syncthreads();
                                if (n < lij3) {
                                    s2x = cpx*s1x + k*b01*s0x;
                                    if (i > 0) {
                                        s2x += i * b00 * _gx[sq_id + (k*stride_k-1)*nsq_per_block];
                                    }
                                    _gx[sq_id + (k*stride_k+stride_k)*nsq_per_block] = s2x;
                                    s0x = s1x;
                                    s1x = s2x;
                                }
                            }
                        }
                    }

                    // hrr
                    // g(i,j+1) = rirj * g(i,j) +  g(i+1,j)
                    // g(...,k,l+1) = rkrl * g(...,k,l) + g(...,k+1,l)
                    if (lj > 0) {
                        __syncthreads();
                        if (task_id < ntasks) {
                            int lkl3 = (lkl+1)*3;
                            for (int m = gout_id; m < lkl3; m += gout_stride) {
                                int k = m / 3;
                                int _ix = m % 3;
                                double xixj = ri[_ix] - rj[_ix];
                                double *_gx = g + (_ix*g_size + k*stride_k) * nsq_per_block;
                                for (int j = 0; j < lj; ++j) {
                                    int ij = (lij-j) + j*stride_j;
                                    s1x = _gx[sq_id + ij*nsq_per_block];
                                    for (--ij; ij >= j*stride_j; --ij) {
                                        s0x = _gx[sq_id + ij*nsq_per_block];
                                        _gx[sq_id + (ij+stride_j)*nsq_per_block] = xixj * s0x + s1x;
                                        s1x = s0x;
                                    }
                                }
                            }
                        }
                    }
                    if (ll > 0) {
                        __syncthreads();
                        if (task_id < ntasks) {
                            for (int n = gout_id; n < stride_k*3; n += gout_stride) {
                                int i = n / 3;
                                int _ix = n % 3;
                                double xkxl = rk[_ix] - rl[_ix];
                                double *_gx = g + (_ix*g_size + i) * nsq_per_block;
                                for (int l = 0; l < ll; ++l) {
                                    int kl = (lkl-l)*stride_k + l*stride_l;
                                    s1x = _gx[sq_id + kl*nsq_per_block];
                                    for (kl-=stride_k; kl >= l*stride_l; kl-=stride_k) {
                                        s0x = _gx[sq_id + kl*nsq_per_block];
                                        _gx[sq_id + (kl+stride_l)*nsq_per_block] = xkxl * s0x + s1x;
                                        s1x = s0x;
                                    }
                                }
                            }
                        }
                    }

                    __syncthreads();
                    if (task_id >= ntasks) {
                        continue;
                    }
                    double *gx = g;
                    double *gy = gx + nsq_per_block * g_size;
                    double *gz = gy + nsq_per_block * g_size;
#pragma unroll
                    for (int n = 0; n < GWIDTH; ++n) {
                        int ijkl = gout_start + n*gout_stride+gout_id;
                        int kl = ijkl / nfij;
                        int ij = ijkl % nfij;
                        if (kl >= nfkl) break;
                        int addrx = sq_id + (idx_ij[ij] + idx_kl[kl] * stride_k) * nsq_per_block;
                        int addry = sq_id + (idy_ij[ij] + idy_kl[kl] * stride_k) * nsq_per_block;
                        int addrz = sq_id + (idz_ij[ij] + idz_kl[kl] * stride_k) * nsq_per_block;
                        gout[n] += gx[addrx] * gy[addry] * gz[addrz];
                    }
                }
            }
        }
        if (task_id >= ntasks) {
            continue;
        }
        double *dm = jk.dm;
        double *vj = jk.vj;
        double *vk = jk.vk;
        int do_j = vj != NULL;
        int do_k = vk != NULL;
        for (int i_dm = 0; i_dm < jk.n_dm; ++i_dm) {
#pragma unroll
            for (int n = 0; n < GWIDTH; ++n) {
                int ijkl = (gout_start + n*gout_stride+gout_id);
                int kl = ijkl / nfij;
                int ij = ijkl % nfij;
                if (kl >= nfkl) break;
                double s = gout[n];
                int i = ij % nfi;
                int j = ij / nfi;
                int k = kl % nfk;
                int l = kl / nfk;
                int _i = i + i0;
                int _j = j + j0;
                int _k = k + k0;
                int _l = l + l0;
                if (do_j) {
                    int _ji = _j*nao+_i;
                    int _lk = _l*nao+_k;
                    atomicAdd(vj+_lk, s * dm[_ji]);
                    atomicAdd(vj+_ji, s * dm[_lk]);
                }
                if (do_k) {
                    int _jl = _j*nao+_l;
                    int _jk = _j*nao+_k;
                    int _il = _i*nao+_l;
                    int _ik = _i*nao+_k;
                    atomicAdd(vk+_ik, s * dm[_jl]);
                    atomicAdd(vk+_il, s * dm[_jk]);
                    atomicAdd(vk+_jk, s * dm[_il]);
                    atomicAdd(vk+_jl, s * dm[_ik]);
                }
            }
            vj += nao * nao;
            vk += nao * nao;
            dm += nao * nao;
        }
    } }
}

__global__
void rys_jk_kernel(RysIntEnvVars envs, JKMatrix jk, BoundsInfo bounds,
                   ShellQuartet *pool, uint32_t *batch_head)
{
    int b_id = blockIdx.x;
    int t_id = threadIdx.y * blockDim.x + threadIdx.x;
    ShellQuartet *shl_quartet_idx = pool + b_id * QUEUE_DEPTH;
    __shared__ int batch_id;
    if (t_id == 0) {
        batch_id = atomicAdd(batch_head, 1);
    }
    __syncthreads();
    int nbatches_kl = (bounds.ntile_kl_pairs + TILES_IN_BATCH - 1) / TILES_IN_BATCH;
    int nbatches = bounds.ntile_ij_pairs * nbatches_kl;
    while (batch_id < nbatches) {
        int batch_ij = batch_id / nbatches_kl;
        int batch_kl = batch_id % nbatches_kl;
        int ntasks = _fill_jk_tasks(shl_quartet_idx, envs, jk, bounds,
                                    batch_ij, batch_kl);
        if (t_id == 0) {
            batch_id = atomicAdd(batch_head, 1);
            atomicAdd(batch_head+1, ntasks);
        }
        if (ntasks > 0) {
            rys_jk_general(envs, jk, bounds, shl_quartet_idx, ntasks);
        }
        __syncthreads();
    }
}

__global__
void rys_sr_jk_kernel(RysIntEnvVars envs, JKMatrix jk, BoundsInfo bounds,
                   ShellQuartet *pool, uint32_t *batch_head)
{
    int b_id = blockIdx.x;
    int t_id = threadIdx.y * blockDim.x + threadIdx.x;
    ShellQuartet *shl_quartet_idx = pool + b_id * QUEUE_DEPTH;
    __shared__ int batch_id;
    if (t_id == 0) {
        batch_id = atomicAdd(batch_head, 1);
    }
    __syncthreads();
    int nbatches_kl = (bounds.ntile_kl_pairs + TILES_IN_BATCH - 1) / TILES_IN_BATCH;
    int nbatches = bounds.ntile_ij_pairs * nbatches_kl;
    while (batch_id < nbatches) {
        int batch_ij = batch_id / nbatches_kl;
        int batch_kl = batch_id % nbatches_kl;
        int ntasks = _fill_sr_jk_tasks(shl_quartet_idx, envs, jk, bounds,
                                       batch_ij, batch_kl);
        if (ntasks > 0) {
            rys_sr_jk_general(envs, jk, bounds, shl_quartet_idx, ntasks);
        }
        if (t_id == 0) {
            batch_id = atomicAdd(batch_head, 1);
            atomicAdd(batch_head+1, ntasks);
        }
        __syncthreads();
    }
}
