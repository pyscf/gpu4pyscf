
#include <hip/hip_runtime.h>
#include <float.h>
#define LSUM_MAX        (LMAX*4)
#define EPS_FLOAT64     DBL_EPSILON
#define SQRTPIE4        .886226925452758013

__device__
static void eval_gamma_inc_fn(double *f, double t, int m)
{
    int t_id = threadIdx.x + blockDim.x * threadIdx.y;
    int block_size = blockDim.x * blockDim.y;
    if (t < EPS_FLOAT64) {
        f[t_id] = 1.;
        for (int i = 1; i <= m; i++) {
            f[t_id + i*block_size] = 1./(2*i+1);
        }
    } else if (m > 0 && t < m*.5+.5) {
        double bi = m + .5;
        double e = .5 * exp(-t);
        double x = e;
        double s = e;
        double tol = EPS_FLOAT64 * e;
        while (x > tol) {
            bi += 1.;
            x *= t / bi;
            s += x;
        }
        double b = m + 0.5;
        double fval = s / b;
        f[t_id + m*block_size] = fval;
        for (int i = m-1; i >= 0; i--) {
            b -= 1.;
            fval = (e + t * fval) / b;
            f[t_id + i*block_size] = fval;
        }
    } else {
        double tt = sqrt(t);
        double fval = SQRTPIE4 / tt * erf(tt);
        f[t_id] = fval;
        if (m > 0) {
            double e = .5 * exp(-t);
            double b = 1. / t;
            double b1 = .5;
            for (int i = 1; i <= m; i++) {
                fval = b * (b1 * fval - e);
                f[t_id + i*block_size] = fval;
                b1 += 1.;
            }
        }
    }
}
