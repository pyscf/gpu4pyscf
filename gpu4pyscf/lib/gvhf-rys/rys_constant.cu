/*
 * Copyright 2025 The PySCF Developers. All Rights Reserved.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <gvhf-rys/vhf.cuh>

__constant__ int _c_cartesian_lexical_xyz[] = {
    // s, offset = 0
    0, 0, 0,
    0, 0, 0, // padding
    0, 0, 0, // padding
    // p, offset = 9
    1, 0, 0,
    0, 1, 0,
    0, 0, 1,
    // d, offset = 9 * 2
    2, 0, 0,
    1, 1, 0,
    1, 0, 1,
    0, 2, 0,
    0, 1, 1,
    0, 0, 2,
    // f, offset = 9 * 4
    3, 0, 0,
    2, 1, 0,
    2, 0, 1,
    1, 2, 0,
    1, 1, 1,
    1, 0, 2,
    0, 3, 0,
    0, 2, 1,
    0, 1, 2,
    0, 0, 3,
    0, 0, 0, // padding
    0, 0, 0, // padding
    // g, offset = 9 * 8
    4, 0, 0,
    3, 1, 0,
    3, 0, 1,
    2, 2, 0,
    2, 1, 1,
    2, 0, 2,
    1, 3, 0,
    1, 2, 1,
    1, 1, 2,
    1, 0, 3,
    0, 4, 0,
    0, 3, 1,
    0, 2, 2,
    0, 1, 3,
    0, 0, 4,
};

__constant__ GXYZOffset c_gxyz_offset[625];
