#include "hip/hip_runtime.h"
/* Copyright 2023 The GPU4PySCF Authors. All Rights Reserved.
 *
 * This program is free software: you can redistribute it and/or modify
 * it under the terms of the GNU General Public License as published by
 * the Free Software Foundation, either version 3 of the License, or
 * (at your option) any later version.
 *
 * This program is distributed in the hope that it will be useful,
 * but WITHOUT ANY WARRANTY; without even the implied warranty of
 * MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
 * GNU General Public License for more details.
 *
 * You should have received a copy of the GNU General Public License
 * along with this program.  If not, see <http://www.gnu.org/licenses/>.
 */

// Unrolled version
template <int LI, int LJ, int LK> __global__
void GINTint3c2e_ip1_jk_kernel(GINTEnvVars envs, JKMatrix jk, BasisProdOffsets offsets)
{
    int ntasks_ij = offsets.ntasks_ij;
    int ntasks_kl = offsets.ntasks_kl;
    int task_ij = blockIdx.x * blockDim.x + threadIdx.x;
    int task_kl = blockIdx.y * blockDim.y + threadIdx.y;
    bool active = true;
    if (task_ij >= ntasks_ij || task_kl >= ntasks_kl) {
        active = false;
        task_ij = 0;
        task_kl = 0;
    }
    double norm = envs.fac;
    int bas_ij = offsets.bas_ij + task_ij;
    int bas_kl = offsets.bas_kl + task_kl;
    int nprim_ij = envs.nprim_ij;
    int nprim_kl = envs.nprim_kl;
    int prim_ij = offsets.primitive_ij + task_ij * nprim_ij;
    int prim_kl = offsets.primitive_kl + task_kl * nprim_kl;
    int *bas_pair2bra = c_bpcache.bas_pair2bra;
    int *bas_pair2ket = c_bpcache.bas_pair2ket;
    int ish = bas_pair2bra[bas_ij];
    int jsh = bas_pair2ket[bas_ij];
    int ksh = bas_pair2bra[bas_kl];
    
    double* __restrict__ exp = c_bpcache.a1;
    constexpr int LI_CEIL = LI + 1;
    constexpr int NROOTS = (LI_CEIL+LJ+LK)/2 + 1;
    constexpr int GSIZE = 3 * NROOTS * (LI_CEIL+1)*(LJ+1)*(LK+1);

    double g[2*GSIZE];
    double *f = g + GSIZE;

    const int as_ish = envs.ibase ? ish: jsh; 
    const int as_jsh = envs.ibase ? jsh: ish; 

    constexpr int nfi = (LI+1)*(LI+2)/2;
    double j3[nfi * 3];
    double k3[nfi * 3];
    for (int k = 0; k < nfi * 3; k++){
        j3[k] = 0.0;
        k3[k] = 0.0;
    }
    if (active) {
        for (int ij = prim_ij; ij < prim_ij+nprim_ij; ++ij) {
            for (int kl = prim_kl; kl < prim_kl+nprim_kl; ++kl) {
                GINTg0_int3c2e<LI_CEIL, LJ, LK>(envs, g, norm, as_ish, as_jsh, ksh, ij, kl);
                double ai2 = -2.0*exp[ij];
                GINTnabla1i_2e<LI, LJ, LK, NROOTS>(envs, f, g, ai2);
                GINTkernel_int3c2e_ip1_getjk_direct<LI, LJ, LK>(envs, jk, j3, k3, f, g, ish, jsh, ksh);
            }
        }
    }

    write_int3c2e_ip1_jk(jk, j3, k3, ish);
}

template <int NROOTS, int GSIZE> __global__
void GINTint3c2e_ip1_jk_kernel(GINTEnvVars envs, JKMatrix jk, BasisProdOffsets offsets)
{
    int ntasks_ij = offsets.ntasks_ij;
    int ntasks_kl = offsets.ntasks_kl;
    int task_ij = blockIdx.x * blockDim.x + threadIdx.x;
    int task_kl = blockIdx.y * blockDim.y + threadIdx.y;
    bool active = true;
    if (task_ij >= ntasks_ij || task_kl >= ntasks_kl) {
        active = false;
        task_ij = 0;
        task_kl = 0;
    }
    double norm = envs.fac;
    int bas_ij = offsets.bas_ij + task_ij;
    int bas_kl = offsets.bas_kl + task_kl;
    int nprim_ij = envs.nprim_ij;
    int nprim_kl = envs.nprim_kl;
    int prim_ij = offsets.primitive_ij + task_ij * nprim_ij;
    int prim_kl = offsets.primitive_kl + task_kl * nprim_kl;
    int *bas_pair2bra = c_bpcache.bas_pair2bra;
    int *bas_pair2ket = c_bpcache.bas_pair2ket;
    int ish = bas_pair2bra[bas_ij];
    int jsh = bas_pair2ket[bas_ij];
    int ksh = bas_pair2bra[bas_kl];
    
    double* __restrict__ exp = c_bpcache.a1;
    double g[2*GSIZE];
    double * __restrict__ f = g + GSIZE;

    const int as_ish = envs.ibase ? ish: jsh; 
    const int as_jsh = envs.ibase ? jsh: ish; 

    double j3[GPU_AO_NF * 3];
    double k3[GPU_AO_NF * 3];
    for (int k = 0; k < GPU_AO_NF * 3; k++){
        j3[k] = 0.0;
        k3[k] = 0.0;
    }
    if (active) {
        for (int ij = prim_ij; ij < prim_ij+nprim_ij; ++ij) {
            for (int kl = prim_kl; kl < prim_kl+nprim_kl; ++kl) {
            GINTg0_int3c2e<NROOTS>(envs, g, norm, as_ish, as_jsh, ksh, ij, kl);
            double ai2 = -2.0*exp[ij];
            GINTnabla1i_2e<NROOTS>(envs, f, g, ai2, envs.i_l, envs.j_l, envs.k_l);
            GINTkernel_int3c2e_ip1_getjk_direct<NROOTS>(envs, jk, j3, k3, f, g, ish, jsh, ksh);
            }
        }
    }

    write_int3c2e_ip1_jk(jk, j3, k3, ish);
}

__global__
static void GINTint3c2e_ip1_jk_kernel000(GINTEnvVars envs, JKMatrix jk, BasisProdOffsets offsets)
{
    int ntasks_ij = offsets.ntasks_ij;
    int ntasks_kl = offsets.ntasks_kl;
    int task_ij = blockIdx.x * blockDim.x + threadIdx.x;
    int task_kl = blockIdx.y * blockDim.y + threadIdx.y;
    bool active = true;
    if (task_ij >= ntasks_ij || task_kl >= ntasks_kl) {
        active = false;
        task_ij = 0;
        task_kl = 0;
    }
    int bas_ij = offsets.bas_ij + task_ij;
    int bas_kl = offsets.bas_kl + task_kl;
    double norm = envs.fac;
    double omega = envs.omega;
    int *bas_pair2bra = c_bpcache.bas_pair2bra;
    int *bas_pair2ket = c_bpcache.bas_pair2ket;
    int ish = bas_pair2bra[bas_ij];
    int jsh = bas_pair2ket[bas_ij];
    int ksh = bas_pair2bra[bas_kl];
    int nprim_ij = envs.nprim_ij;
    int nprim_kl = envs.nprim_kl;
    int prim_ij = offsets.primitive_ij + task_ij * nprim_ij;
    int prim_kl = offsets.primitive_kl + task_kl * nprim_kl;
    double* __restrict__ a12 = c_bpcache.a12;
    double* __restrict__ e12 = c_bpcache.e12;
    double* __restrict__ x12 = c_bpcache.x12;
    double* __restrict__ y12 = c_bpcache.y12;
    double* __restrict__ z12 = c_bpcache.z12;
    double* __restrict__ a1 = c_bpcache.a1;
    int ij, kl;
    int prim_ij0, prim_ij1, prim_kl0, prim_kl1;
    int nbas = c_bpcache.nbas;
    double* __restrict__ bas_x = c_bpcache.bas_coords;
    double* __restrict__ bas_y = bas_x + nbas;
    double* __restrict__ bas_z = bas_y + nbas;

    double gout0 = 0;
    double gout1 = 0;
    double gout2 = 0;
    double xi = bas_x[ish];
    double yi = bas_y[ish];
    double zi = bas_z[ish];
    prim_ij0 = prim_ij;
    prim_ij1 = prim_ij + nprim_ij;
    prim_kl0 = prim_kl;
    prim_kl1 = prim_kl + nprim_kl;
    for (ij = prim_ij0; ij < prim_ij1; ++ij) {
    for (kl = prim_kl0; kl < prim_kl1; ++kl) {
        double ai2 = -2.0*a1[ij];
        double aij = a12[ij];
        double eij = e12[ij];
        double xij = x12[ij];
        double yij = y12[ij];
        double zij = z12[ij];
        double akl = a12[kl];
        double ekl = e12[kl];
        double xkl = x12[kl];
        double ykl = y12[kl];
        double zkl = z12[kl];
        double xijxkl = xij - xkl;
        double yijykl = yij - ykl;
        double zijzkl = zij - zkl;
        double aijkl = aij + akl;
        double a1 = aij * akl;
        double a0 = a1 / aijkl;
        double theta = omega > 0.0 ? omega * omega / (omega * omega + a0) : 1.0;
        a0 *= theta;
        double x = a0 * (xijxkl * xijxkl + yijykl * yijykl + zijzkl * zijzkl);
        double fac = eij * ekl * sqrt(a0 / (a1 * a1 * a1));
        double root0, weight0;
        if (x < 3.e-7) {
            root0 = 0.5;
            weight0 = 1.;
        } else {
            double tt = sqrt(x);
            double fmt0 = SQRTPIE4 / tt * erf(tt);
            weight0 = fmt0;
            double e = exp(-x);
            double b = .5 / x;
            double fmt1 = b * (fmt0 - e);
            root0 = fmt1 / (fmt0 - fmt1);
        }
        root0 /= root0 + 1 - root0 * theta;
        double u2 = a0 * root0;
        double tmp2 = akl * u2 / (u2 * aijkl + a1);;
        double c00x = xij - xi - tmp2 * xijxkl;
        double c00y = yij - yi - tmp2 * yijykl;
        double c00z = zij - zi - tmp2 * zijzkl;
        double g_0 = 1;
        double g_1 = c00x;
        double g_2 = 1;
        double g_3 = c00y;
        double g_4 = norm * fac * weight0;
        double g_5 = g_4 * c00z;

        double f_1 = ai2 * g_1;
        double f_3 = ai2 * g_3;
        double f_5 = ai2 * g_5;

        gout0 += f_1 * g_2 * g_4;
        gout1 += g_0 * f_3 * g_4;
        gout2 += g_0 * g_2 * f_5;
    } }

    int *ao_loc = c_bpcache.ao_loc;
    int i0 = ao_loc[ish] - jk.ao_offsets_i;
    int j0 = ao_loc[jsh] - jk.ao_offsets_j;
    int k0 = ao_loc[ksh] - jk.ao_offsets_k;

    int nao = jk.nao;
    double* __restrict__ dm = jk.dm;
    double* __restrict__ rhok = jk.rhok;
    double* __restrict__ rhoj = jk.rhoj;
    double* __restrict__ vj = jk.vj;
    double* __restrict__ vk = jk.vk;

    int tx = threadIdx.x;
    int ty = threadIdx.y;
    __shared__ double sdata[THREADSX][THREADSY];
    if (!active){
        gout0 = 0.0; gout1 = 0.0; gout2 = 0.0;
    }
    if (vj != NULL){
        double rhoj_tmp;
        int off_dm = i0 + nao*j0;
        rhoj_tmp = dm[off_dm] * rhoj[k0];
        double vj_tmp[3];
        vj_tmp[0] = gout0*rhoj_tmp;
        vj_tmp[1] = gout1*rhoj_tmp;
        vj_tmp[2] = gout2*rhoj_tmp;
        for (int j = 0; j < 3; j++){
            sdata[tx][ty] = vj_tmp[j]; __syncthreads();
            if(THREADSY >= 16 && ty<8) sdata[tx][ty] += sdata[tx][ty+8]; __syncthreads();
            if(THREADSY >= 8  && ty<4) sdata[tx][ty] += sdata[tx][ty+4]; __syncthreads();
            if(THREADSY >= 4  && ty<2) sdata[tx][ty] += sdata[tx][ty+2]; __syncthreads();
            if(THREADSY >= 2  && ty<1) sdata[tx][ty] += sdata[tx][ty+1]; __syncthreads();
            if (ty == 0) atomicAdd(vj+i0+j*nao, sdata[tx][0]);
        }
    }
    if (vk != NULL){
        double rhok_tmp;
        int off_rhok = i0 + nao*j0 + k0*nao*nao;
        rhok_tmp = rhok[off_rhok];
        double vk_tmp[3];
        vk_tmp[0] = gout0 * rhok_tmp;
        vk_tmp[1] = gout1 * rhok_tmp;
        vk_tmp[2] = gout2 * rhok_tmp;
        for (int j = 0; j < 3; j++){
            sdata[tx][ty] = vk_tmp[j]; __syncthreads();
            if(THREADSY >= 16 && ty<8) sdata[tx][ty] += sdata[tx][ty+8]; __syncthreads();
            if(THREADSY >=  8 && ty<4) sdata[tx][ty] += sdata[tx][ty+4]; __syncthreads();
            if(THREADSY >=  4 && ty<2) sdata[tx][ty] += sdata[tx][ty+2]; __syncthreads();
            if(THREADSY >=  2 && ty<1) sdata[tx][ty] += sdata[tx][ty+1]; __syncthreads();
            if (ty == 0) atomicAdd(vk+i0+j*nao, sdata[tx][0]);
        }
    }
}
