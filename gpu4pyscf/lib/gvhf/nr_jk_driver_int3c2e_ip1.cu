/*
 * Copyright 2021-2024 The PySCF Developers. All Rights Reserved.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include <stdio.h>
#include <stdint.h>
#include <stdlib.h>
#include <string.h>
#include <hip/hip_runtime.h>

#include "gint/gint.h"
#include "gint/config.h"
#include "gint/cuda_alloc.cuh"
#include "gint/g2e.h"
#include "gint/cint2e.cuh"

#include "contract_jk.cu"
#include "gint/rys_roots.cu"
#include "gint/g2e.cu"
#include "g3c2e.cuh"
#include "g3c2e_ip1.cu"

__host__
static int GINTrun_tasks_int3c2e_ip1_jk(JKMatrix *jk, BasisProdOffsets *offsets, GINTEnvVars *envs, hipStream_t stream)
{
    int nrys_roots = envs->nrys_roots;
    int ntasks_ij = offsets->ntasks_ij;
    int ntasks_kl = offsets->ntasks_kl;
    assert(ntasks_kl < 65536*THREADSY);
    dim3 threads(THREADSX, THREADSY);
    dim3 blocks((ntasks_ij+THREADSX-1)/THREADSX, (ntasks_kl+THREADSY-1)/THREADSY);
    int li = envs->i_l;
    int lj = envs->j_l;
    int lk = envs->k_l;
    int type_ijk = li * 100 + lj * 10 + lk;

    switch (type_ijk) {
        case   0: GINTint3c2e_ip1_jk_kernel000<<<blocks, threads, 0, stream>>>(*envs, *jk, *offsets); break;
        // li+lj+lk=1
        case 1: GINTint3c2e_ip1_jk_kernel<0,0,1><<<blocks, threads, 0, stream>>>(*envs, *jk, *offsets); break;
        case 10: GINTint3c2e_ip1_jk_kernel<0,1,0><<<blocks, threads, 0, stream>>>(*envs, *jk, *offsets); break;
        case 100: GINTint3c2e_ip1_jk_kernel<1,0,0><<<blocks, threads, 0, stream>>>(*envs, *jk, *offsets); break;
        // li+lj+lk=2
        case 2: GINTint3c2e_ip1_jk_kernel<0,0,2><<<blocks, threads, 0, stream>>>(*envs, *jk, *offsets); break;
        case 11: GINTint3c2e_ip1_jk_kernel<0,1,1><<<blocks, threads, 0, stream>>>(*envs, *jk, *offsets); break;
        case 20: GINTint3c2e_ip1_jk_kernel<0,2,0><<<blocks, threads, 0, stream>>>(*envs, *jk, *offsets); break;
        case 101: GINTint3c2e_ip1_jk_kernel<1,0,1><<<blocks, threads, 0, stream>>>(*envs, *jk, *offsets); break;
        case 110: GINTint3c2e_ip1_jk_kernel<1,1,0><<<blocks, threads, 0, stream>>>(*envs, *jk, *offsets); break;
        case 200: GINTint3c2e_ip1_jk_kernel<2,0,0><<<blocks, threads, 0, stream>>>(*envs, *jk, *offsets); break;
        // li+lj+lk=3
        case 3: GINTint3c2e_ip1_jk_kernel<0,0,3><<<blocks, threads, 0, stream>>>(*envs, *jk, *offsets); break;
        case 12: GINTint3c2e_ip1_jk_kernel<0,1,2><<<blocks, threads, 0, stream>>>(*envs, *jk, *offsets); break;
        case 21: GINTint3c2e_ip1_jk_kernel<0,2,1><<<blocks, threads, 0, stream>>>(*envs, *jk, *offsets); break;
        case 30: GINTint3c2e_ip1_jk_kernel<0,3,0><<<blocks, threads, 0, stream>>>(*envs, *jk, *offsets); break;
        case 102: GINTint3c2e_ip1_jk_kernel<1,0,2><<<blocks, threads, 0, stream>>>(*envs, *jk, *offsets); break;
        case 111: GINTint3c2e_ip1_jk_kernel<1,1,1><<<blocks, threads, 0, stream>>>(*envs, *jk, *offsets); break;
        case 120: GINTint3c2e_ip1_jk_kernel<1,2,0><<<blocks, threads, 0, stream>>>(*envs, *jk, *offsets); break;
        case 201: GINTint3c2e_ip1_jk_kernel<2,0,1><<<blocks, threads, 0, stream>>>(*envs, *jk, *offsets); break;
        case 210: GINTint3c2e_ip1_jk_kernel<2,1,0><<<blocks, threads, 0, stream>>>(*envs, *jk, *offsets); break;
        case 300: GINTint3c2e_ip1_jk_kernel<3,0,0><<<blocks, threads, 0, stream>>>(*envs, *jk, *offsets); break;
        // li+lj+lk=4
        case 4: GINTint3c2e_ip1_jk_kernel<0,0,4><<<blocks, threads, 0, stream>>>(*envs, *jk, *offsets); break;
        case 13: GINTint3c2e_ip1_jk_kernel<0,1,3><<<blocks, threads, 0, stream>>>(*envs, *jk, *offsets); break;
        case 22: GINTint3c2e_ip1_jk_kernel<0,2,2><<<blocks, threads, 0, stream>>>(*envs, *jk, *offsets); break;
        case 31: GINTint3c2e_ip1_jk_kernel<0,3,1><<<blocks, threads, 0, stream>>>(*envs, *jk, *offsets); break;
        case 40: GINTint3c2e_ip1_jk_kernel<0,4,0><<<blocks, threads, 0, stream>>>(*envs, *jk, *offsets); break;
        case 103: GINTint3c2e_ip1_jk_kernel<1,0,3><<<blocks, threads, 0, stream>>>(*envs, *jk, *offsets); break;
        case 112: GINTint3c2e_ip1_jk_kernel<1,1,2><<<blocks, threads, 0, stream>>>(*envs, *jk, *offsets); break;
        case 121: GINTint3c2e_ip1_jk_kernel<1,2,1><<<blocks, threads, 0, stream>>>(*envs, *jk, *offsets); break;
        case 130: GINTint3c2e_ip1_jk_kernel<1,3,0><<<blocks, threads, 0, stream>>>(*envs, *jk, *offsets); break;
        case 202: GINTint3c2e_ip1_jk_kernel<2,0,2><<<blocks, threads, 0, stream>>>(*envs, *jk, *offsets); break;
        case 211: GINTint3c2e_ip1_jk_kernel<2,1,1><<<blocks, threads, 0, stream>>>(*envs, *jk, *offsets); break;
        case 220: GINTint3c2e_ip1_jk_kernel<2,2,0><<<blocks, threads, 0, stream>>>(*envs, *jk, *offsets); break;
        case 301: GINTint3c2e_ip1_jk_kernel<3,0,1><<<blocks, threads, 0, stream>>>(*envs, *jk, *offsets); break;
        case 310: GINTint3c2e_ip1_jk_kernel<3,1,0><<<blocks, threads, 0, stream>>>(*envs, *jk, *offsets); break;
        case 400: GINTint3c2e_ip1_jk_kernel<4,0,0><<<blocks, threads, 0, stream>>>(*envs, *jk, *offsets); break;
        // li+lj+lk=5
        //case 5: GINTint3c2e_ip1_jk_kernel<0,0,5><<<blocks, threads, 0, stream>>>(*envs, *jk, *offsets); break;
        case 14: GINTint3c2e_ip1_jk_kernel<0,1,4><<<blocks, threads, 0, stream>>>(*envs, *jk, *offsets); break;
        case 23: GINTint3c2e_ip1_jk_kernel<0,2,3><<<blocks, threads, 0, stream>>>(*envs, *jk, *offsets); break;
        case 32: GINTint3c2e_ip1_jk_kernel<0,3,2><<<blocks, threads, 0, stream>>>(*envs, *jk, *offsets); break;
        case 41: GINTint3c2e_ip1_jk_kernel<0,4,1><<<blocks, threads, 0, stream>>>(*envs, *jk, *offsets); break;
        //case 50: GINTint3c2e_ip1_jk_kernel<0,5,0><<<blocks, threads, 0, stream>>>(*envs, *jk, *offsets); break;
        case 104: GINTint3c2e_ip1_jk_kernel<1,0,4><<<blocks, threads, 0, stream>>>(*envs, *jk, *offsets); break;
        case 113: GINTint3c2e_ip1_jk_kernel<1,1,3><<<blocks, threads, 0, stream>>>(*envs, *jk, *offsets); break;
        case 122: GINTint3c2e_ip1_jk_kernel<1,2,2><<<blocks, threads, 0, stream>>>(*envs, *jk, *offsets); break;
        case 131: GINTint3c2e_ip1_jk_kernel<1,3,1><<<blocks, threads, 0, stream>>>(*envs, *jk, *offsets); break;
        case 140: GINTint3c2e_ip1_jk_kernel<1,4,0><<<blocks, threads, 0, stream>>>(*envs, *jk, *offsets); break;
        case 203: GINTint3c2e_ip1_jk_kernel<2,0,3><<<blocks, threads, 0, stream>>>(*envs, *jk, *offsets); break;
        case 212: GINTint3c2e_ip1_jk_kernel<2,1,2><<<blocks, threads, 0, stream>>>(*envs, *jk, *offsets); break;
        case 221: GINTint3c2e_ip1_jk_kernel<2,2,1><<<blocks, threads, 0, stream>>>(*envs, *jk, *offsets); break;
        case 230: GINTint3c2e_ip1_jk_kernel<2,3,0><<<blocks, threads, 0, stream>>>(*envs, *jk, *offsets); break;
        case 302: GINTint3c2e_ip1_jk_kernel<3,0,2><<<blocks, threads, 0, stream>>>(*envs, *jk, *offsets); break;
        case 311: GINTint3c2e_ip1_jk_kernel<3,1,1><<<blocks, threads, 0, stream>>>(*envs, *jk, *offsets); break;
        case 320: GINTint3c2e_ip1_jk_kernel<3,2,0><<<blocks, threads, 0, stream>>>(*envs, *jk, *offsets); break;
        case 401: GINTint3c2e_ip1_jk_kernel<4,0,1><<<blocks, threads, 0, stream>>>(*envs, *jk, *offsets); break;
        case 410: GINTint3c2e_ip1_jk_kernel<4,1,0><<<blocks, threads, 0, stream>>>(*envs, *jk, *offsets); break;
        //case 500: GINTint3c2e_ip1_jk_kernel<5,0,0><<<blocks, threads, 0, stream>>>(*envs, *jk, *offsets); break;
#ifdef UNROLL_INT3C2E
#endif
        default: {
            dim3 threads(THREADSX*THREADSY);
            dim3 blocks(ntasks_ij, ntasks_kl);
            const int li_ceil = li + 1;
            const int gsize = 3*nrys_roots*(li_ceil+1)*(lj+1)*(lk+1);
            hipError_t err = hipFuncSetAttribute(reinterpret_cast<const void*>(
                GINTint3c2e_ip1_jk_general_kernel),
                hipFuncAttributeMaxDynamicSharedMemorySize,
                (gsize+16)*sizeof(double));
            if (err != hipSuccess) {
                fprintf(stderr, "hipFuncSetAttribute error: %s\n", hipGetErrorString(err));
                return 1;
            }
            const int shm_size = gsize*sizeof(double);
            GINTint3c2e_ip1_jk_general_kernel<<<blocks, threads, shm_size, stream>>>(*envs, *jk, *offsets);
        }
    }

    hipError_t err = hipGetLastError();
    if (err != hipSuccess) {
        fprintf(stderr, "CUDA Error of GINTint3c2e_ip1_jk_kernel: %s\n", hipGetErrorString(err));
        return 1;
    }
    return 0;
}


extern "C" { __host__
int GINTbuild_int3c2e_ip1_jk(hipStream_t stream, BasisProdCache *bpcache,
                 double *vj, double *vk, double *dm, double *rhoj, double *rhok,
                 int *ao_offsets, int nao, int naux, int n_dm,
                 int *bins_locs_ij, int ntasks_kl, int ncp_ij, int cp_kl_id, double omega)
{
    ContractionProdType *cp_kl = bpcache->cptype + cp_kl_id;

    int ng[4] = {1,0,0,0};

    // move bpcache to constant memory
    checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(c_bpcache), bpcache, sizeof(BasisProdCache)));

    JKMatrix jk;
    jk.n_dm = n_dm;
    jk.nao = nao;
    jk.naux = naux;
    jk.dm = dm;
    jk.vj = vj;
    jk.vk = vk;
    jk.rhoj = rhoj;
    jk.rhok = rhok;
    jk.ao_offsets_i = ao_offsets[0];
    jk.ao_offsets_j = ao_offsets[1];
    jk.ao_offsets_k = ao_offsets[2];
    jk.ao_offsets_l = ao_offsets[3];

    int *bas_pairs_locs = bpcache->bas_pairs_locs;
    int *primitive_pairs_locs = bpcache->primitive_pairs_locs;

    for (int cp_ij_id = 0; cp_ij_id < ncp_ij; cp_ij_id++){
        GINTEnvVars envs;
        ContractionProdType *cp_ij = bpcache->cptype + cp_ij_id;
        GINTinit_EnvVars(&envs, cp_ij, cp_kl, ng);
        envs.omega = omega;
        if (envs.nrys_roots > 9) {
            return 2;
        }
        int ntasks_ij = bins_locs_ij[cp_ij_id+1] - bins_locs_ij[cp_ij_id];
        if (ntasks_ij <= 0) continue;

        BasisProdOffsets offsets;
        offsets.ntasks_ij = ntasks_ij;
        offsets.ntasks_kl = ntasks_kl;
        offsets.bas_ij = bas_pairs_locs[cp_ij_id];
        offsets.bas_kl = bas_pairs_locs[cp_kl_id];
        offsets.primitive_ij = primitive_pairs_locs[cp_ij_id];
        offsets.primitive_kl = primitive_pairs_locs[cp_kl_id];

        int err = GINTrun_tasks_int3c2e_ip1_jk(&jk, &offsets, &envs, stream);

        if (err != 0) {
            return err;
        }
    }
    return 0;
}

}
