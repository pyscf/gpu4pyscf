#include "hip/hip_runtime.h"
/* Copyright 2023 The GPU4PySCF Authors. All Rights Reserved.
 *
 * This program is free software: you can redistribute it and/or modify
 * it under the terms of the GNU General Public License as published by
 * the Free Software Foundation, either version 3 of the License, or
 * (at your option) any later version.
 *
 * This program is distributed in the hope that it will be useful,
 * but WITHOUT ANY WARRANTY; without even the implied warranty of
 * MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
 * GNU General Public License for more details.
 *
 * You should have received a copy of the GNU General Public License
 * along with this program.  If not, see <http://www.gnu.org/licenses/>.
 */

// Unrolled verion
template <int LI, int LJ, int LK> __global__
void GINTint3c2e_ip2_jk_kernel(GINTEnvVars envs, JKMatrix jk, BasisProdOffsets offsets)
{
    int ntasks_ij = offsets.ntasks_ij;
    int ntasks_kl = offsets.ntasks_kl;
    int task_ij = blockIdx.x * blockDim.x + threadIdx.x;
    int task_kl = blockIdx.y * blockDim.y + threadIdx.y;
    bool active = true;
    if (task_ij >= ntasks_ij || task_kl >= ntasks_kl) {
        active = false;
        task_ij = 0;
        task_kl = 0;
    }
    double norm = envs.fac;

    int bas_ij = offsets.bas_ij + task_ij;
    int bas_kl = offsets.bas_kl + task_kl;
    int nprim_ij = envs.nprim_ij;
    int nprim_kl = envs.nprim_kl;
    int prim_ij = offsets.primitive_ij + task_ij * nprim_ij;
    int prim_kl = offsets.primitive_kl + task_kl * nprim_kl;
    int *bas_pair2bra = c_bpcache.bas_pair2bra;
    int *bas_pair2ket = c_bpcache.bas_pair2ket;
    int ish = bas_pair2bra[bas_ij];
    int jsh = bas_pair2ket[bas_ij];
    int ksh = bas_pair2bra[bas_kl];
    int lsh = bas_pair2ket[bas_kl];
    double* __restrict__ exp = c_bpcache.a1;
    constexpr int LK_CEIL = LK + 1;
    constexpr int NROOTS = (LI+LJ+LK_CEIL)/2 + 1;
    constexpr int GSIZE = 3 * NROOTS * (LI+1)*(LJ+1)*(LK_CEIL+1);

    double g[2*GSIZE];
    double *f = g + GSIZE;

    int as_ish, as_jsh, as_ksh, as_lsh;
    if (envs.ibase) {
        as_ish = ish;
        as_jsh = jsh;
    } else {
        as_ish = jsh;
        as_jsh = ish;
    }
    if (envs.kbase) {
        as_ksh = ksh;
        as_lsh = lsh;
    } else {
        as_ksh = lsh;
        as_lsh = ksh;
    }

    constexpr int nfk = (LK+1)*(LK+2)/2;
    double j3[nfk * 3];
    double k3[nfk * 3];
    for (int k = 0; k < nfk * 3; k++){
        j3[k] = 0.0;
        k3[k] = 0.0;
    }
    if (active) {
        for (int ij = prim_ij; ij < prim_ij+nprim_ij; ++ij) {
            for (int kl = prim_kl; kl < prim_kl+nprim_kl; ++kl) {
                GINTg0_int3c2e<LI, LJ, LK_CEIL>(envs, g, norm, as_ish, as_jsh, as_ksh, as_lsh, ij, kl);
                double ak2 = -2.0*exp[kl];
                GINTnabla1k_2e<LI, LJ, LK, NROOTS>(envs, f, g, ak2);
                GINTkernel_int3c2e_ip2_getjk_direct<LI, LJ, LK>(envs, jk, j3, k3, f, g, ish, jsh, ksh);
            }
        }
    }

    write_int3c2e_ip2_jk(jk, j3, k3, ksh);
}

// General version
template <int NROOTS, int GSIZE> __global__
void GINTint3c2e_ip2_jk_kernel(GINTEnvVars envs, JKMatrix jk, BasisProdOffsets offsets)
{
    int ntasks_ij = offsets.ntasks_ij;
    int ntasks_kl = offsets.ntasks_kl;
    int task_ij = blockIdx.x * blockDim.x + threadIdx.x;
    int task_kl = blockIdx.y * blockDim.y + threadIdx.y;
    bool active = true;
    if (task_ij >= ntasks_ij || task_kl >= ntasks_kl) {
        active = false;
        task_ij = 0;
        task_kl = 0;
    }
    double norm = envs.fac;

    int bas_ij = offsets.bas_ij + task_ij;
    int bas_kl = offsets.bas_kl + task_kl;
    int nprim_ij = envs.nprim_ij;
    int nprim_kl = envs.nprim_kl;
    int prim_ij = offsets.primitive_ij + task_ij * nprim_ij;
    int prim_kl = offsets.primitive_kl + task_kl * nprim_kl;
    int *bas_pair2bra = c_bpcache.bas_pair2bra;
    int *bas_pair2ket = c_bpcache.bas_pair2ket;
    int ish = bas_pair2bra[bas_ij];
    int jsh = bas_pair2ket[bas_ij];
    int ksh = bas_pair2bra[bas_kl];
    int lsh = bas_pair2ket[bas_kl];
    double* __restrict__ exp = c_bpcache.a1;
    double g[2*GSIZE];
    double *f = g + GSIZE;

    int ij, kl;
    int as_ish, as_jsh, as_ksh, as_lsh;
    if (envs.ibase) {
        as_ish = ish;
        as_jsh = jsh;
    } else {
        as_ish = jsh;
        as_jsh = ish;
    }
    if (envs.kbase) {
        as_ksh = ksh;
        as_lsh = lsh;
    } else {
        as_ksh = lsh;
        as_lsh = ksh;
    }

    double j3[GPU_AUX_NF * 3];
    double k3[GPU_AUX_NF * 3];
    for (int k = 0; k < GPU_AUX_NF * 3; k++){
        j3[k] = 0.0;
        k3[k] = 0.0;
    }
    if (active) {
        for (ij = prim_ij; ij < prim_ij+nprim_ij; ++ij) {
            for (kl = prim_kl; kl < prim_kl+nprim_kl; ++kl) {
            GINTg0_int3c2e<NROOTS>(envs, g, norm, as_ish, as_jsh, as_ksh, as_lsh, ij, kl);
            double ak2 = -2.0*exp[kl];
            GINTnabla1k_2e<NROOTS>(envs, f, g, ak2, envs.i_l, envs.j_l, envs.k_l);
            GINTkernel_int3c2e_ip2_getjk_direct<NROOTS>(envs, jk, j3, k3, f, g, ish, jsh, ksh);
            }
        }
    }

    write_int3c2e_ip2_jk(jk, j3, k3, ksh);
}

__global__
static void GINTint3c2e_ip2_jk_kernel001(GINTEnvVars envs, JKMatrix jk, BasisProdOffsets offsets)
{
    int ntasks_ij = offsets.ntasks_ij;
    int ntasks_kl = offsets.ntasks_kl;
    int task_ij = blockIdx.x * blockDim.x + threadIdx.x;
    int task_kl = blockIdx.y * blockDim.y + threadIdx.y;
    bool active = true;
    if (task_ij >= ntasks_ij || task_kl >= ntasks_kl) {
        active = false;
        task_ij = 0;
        task_kl = 0;
    }
    int bas_ij = offsets.bas_ij + task_ij;
    int bas_kl = offsets.bas_kl + task_kl;
    double norm = envs.fac;
    double omega = envs.omega;
    int *bas_pair2bra = c_bpcache.bas_pair2bra;
    int *bas_pair2ket = c_bpcache.bas_pair2ket;
    int ish = bas_pair2bra[bas_ij];
    int jsh = bas_pair2ket[bas_ij];
    int ksh = bas_pair2bra[bas_kl];
    int nprim_ij = envs.nprim_ij;
    int nprim_kl = envs.nprim_kl;
    int prim_ij = offsets.primitive_ij + task_ij * nprim_ij;
    int prim_kl = offsets.primitive_kl + task_kl * nprim_kl;
    double* __restrict__ a12 = c_bpcache.a12;
    double* __restrict__ e12 = c_bpcache.e12;
    double* __restrict__ x12 = c_bpcache.x12;
    double* __restrict__ y12 = c_bpcache.y12;
    double* __restrict__ z12 = c_bpcache.z12;
    double* __restrict__ a1 = c_bpcache.a1;
    int ij, kl;
    int prim_ij0, prim_ij1, prim_kl0, prim_kl1;
    int nbas = c_bpcache.nbas;
    double* __restrict__ bas_x = c_bpcache.bas_coords;
    double* __restrict__ bas_y = bas_x + nbas;
    double* __restrict__ bas_z = bas_y + nbas;

    double gout0 = 0;
    double gout1 = 0;
    double gout2 = 0;
    double xk = bas_x[ksh];
    double yk = bas_y[ksh];
    double zk = bas_z[ksh];
    prim_ij0 = prim_ij;
    prim_ij1 = prim_ij + nprim_ij;
    prim_kl0 = prim_kl;
    prim_kl1 = prim_kl + nprim_kl;
    for (ij = prim_ij0; ij < prim_ij1; ++ij) {
    for (kl = prim_kl0; kl < prim_kl1; ++kl) {
        double ak2 = -2.0*a1[kl];
        double aij = a12[ij];
        double eij = e12[ij];
        double xij = x12[ij];
        double yij = y12[ij];
        double zij = z12[ij];
        double akl = a12[kl];
        double ekl = e12[kl];
        double xkl = x12[kl];
        double ykl = y12[kl];
        double zkl = z12[kl];
        double xijxkl = xij - xkl;
        double yijykl = yij - ykl;
        double zijzkl = zij - zkl;
        double aijkl = aij + akl;
        double a1 = aij * akl;
        double a0 = a1 / aijkl;
        double theta = omega > 0.0 ? omega * omega / (omega * omega + a0) : 1.0;
        a0 *= theta;
        double x = a0 * (xijxkl * xijxkl + yijykl * yijykl + zijzkl * zijzkl);
        double fac = norm * eij * ekl * sqrt(a0 / (a1 * a1 * a1));
        double root0, weight0;
        if (x < 3.e-7) {
            root0 = 0.5;
            weight0 = 1.;
        } else {
            double tt = sqrt(x);
            double fmt0 = SQRTPIE4 / tt * erf(tt);
            weight0 = fmt0;
            double e = exp(-x);
            double b = .5 / x;
            double fmt1 = b * (fmt0 - e);
            root0 = fmt1 / (fmt0 - fmt1);
        }
        root0 /= root0 + 1 - root0 * theta;
        double u2 = a0 * root0;
        double tmp4 = .5 / (u2 * aijkl + a1);
        double b00 = u2 * tmp4;
        double tmp1 = 2 * b00;
        double tmp3 = tmp1 * aij;
        double c0px = xkl - xk + tmp3 * xijxkl;
        double c0py = ykl - yk + tmp3 * yijykl;
        double c0pz = zkl - zk + tmp3 * zijzkl;
        double g_0 = 1;
        double g_1 = c0px;
        double g_2 = 1;
        double g_3 = c0py;
        double g_4 = weight0 * fac;
        double g_5 = c0pz * g_4;

        double f_1 = ak2 * g_1;
        double f_3 = ak2 * g_3;
        double f_5 = ak2 * g_5;

        gout0 += f_1 * g_2 * g_4;
        gout1 += g_0 * f_3 * g_4;
        gout2 += g_0 * g_2 * f_5;

    } }

    int *ao_loc = c_bpcache.ao_loc;
    int i0 = ao_loc[ish] - jk.ao_offsets_i;
    int j0 = ao_loc[jsh] - jk.ao_offsets_j;
    int k0 = ao_loc[ksh] - jk.ao_offsets_k;

    int nao = jk.nao;
    int naux = jk.naux;
    double* __restrict__ dm = jk.dm;
    double* __restrict__ rhok = jk.rhok;
    double* __restrict__ rhoj = jk.rhoj;
    double* __restrict__ vj = jk.vj;
    double* __restrict__ vk = jk.vk;

    int tx = threadIdx.x;
    int ty = threadIdx.y;
    __shared__ double sdata[THREADSX][THREADSY];
    if (!active){
        gout0 = 0.0; gout1 = 0.0; gout2 = 0.0;
    }
    if (vj != NULL){
        double rhoj_tmp;
        int off_dm = i0 + nao*j0;
        rhoj_tmp = dm[off_dm] * rhoj[k0];
        double vj_tmp[3];
        vj_tmp[0] = gout0 * rhoj_tmp;
        vj_tmp[1] = gout1 * rhoj_tmp;
        vj_tmp[2] = gout2 * rhoj_tmp;
        for (int j = 0; j < 3; j++){
            sdata[tx][ty] = vj_tmp[j]; __syncthreads();
            if(tx<8) sdata[tx][ty] += sdata[tx+8][ty]; __syncthreads();
            if(tx<4) sdata[tx][ty] += sdata[tx+4][ty]; __syncthreads();
            if(tx<2) sdata[tx][ty] += sdata[tx+2][ty]; __syncthreads();
            if(tx<1) sdata[tx][ty] += sdata[tx+1][ty]; __syncthreads();
            if (tx == 0) atomicAdd(vj+k0+j*naux, sdata[0][ty]);
        }
    }

    if (vk != NULL){
        double rhok_tmp;
        int off_rhok = i0 + nao*j0 + k0*nao*nao;
        rhok_tmp = rhok[off_rhok];
        double vk_tmp[3];
        vk_tmp[0] = gout0 * rhok_tmp;
        vk_tmp[1] = gout1 * rhok_tmp;
        vk_tmp[2] = gout2 * rhok_tmp;
        for (int j = 0; j < 3; j++){
            sdata[tx][ty] = vk_tmp[j]; __syncthreads();
            if(tx<8) sdata[tx][ty] += sdata[tx+8][ty]; __syncthreads();
            if(tx<4) sdata[tx][ty] += sdata[tx+4][ty]; __syncthreads();
            if(tx<2) sdata[tx][ty] += sdata[tx+2][ty]; __syncthreads();
            if(tx<1) sdata[tx][ty] += sdata[tx+1][ty]; __syncthreads();
            if (tx == 0) atomicAdd(vk+k0+j*naux, sdata[0][ty]);
        }
    }
}