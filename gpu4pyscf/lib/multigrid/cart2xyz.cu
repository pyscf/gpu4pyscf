#include "hip/hip_runtime.h"
/*
 * Copyright 2025 The PySCF Developers. All Rights Reserved.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include <stdint.h>
#include "multigrid.cuh"

__device__ inline
void dm_xyz_coeff(double *cx, double xi, double xj, int lmax)
{
    double xij = xi - xj;
    int lmax1 = lmax + 1;
    cx[0] = 1.;
    for (int lx = 1; lx <= lmax; lx++) {
        cx[lx*WARP_SIZE] = cx[(lx-1)*WARP_SIZE] * xij;
    }
    for (int l = 1; l <= lmax; l++){
        double binom = 1.;
        for (int lx = 0; lx <= l; lx++) {
            // binom = binomial(l, lx)
            cx[(l*lmax1+lx)*WARP_SIZE] = binom * cx[(l-lx)*WARP_SIZE];
            binom = (binom * (l-lx)) / (lx+1);
        }
    }
}

__device__ inline
int cart_address(int l, int x, int y, int z)
{
    // (l-x)*(l-x+1)/2+l-x-y
    int yz = l - x;
    return yz * (yz + 3) / 2 - y;
}

__device__ inline
double sub_dm_xyz(int lx, int ly, int lz, int li, int lj, int nao,
                  double *cx, double *cy, double *cz, double *dm)
{
    // TODO: unroll lij < 4
    int lj1 = lj + 1;
    double out = 0.;
    for (int lx_i = MIN(lx, li); lx_i >= 0; lx_i--) {
    for (int ly_i = MIN(ly, li-lx_i); ly_i >= 0; ly_i--) {
        int lz_i = li - lx_i - ly_i;
        if (lz < lz_i) continue;
        int jx = lx - lx_i;
        int jy = ly - ly_i;
        int jz = lz - lz_i;
        int i = cart_address(li, lx_i, ly_i, lz_i);
        // TODO: precomputing index
        for (int lx_j = lj; lx_j >= jx; lx_j--) {
        for (int ly_j = lj-lx_j; ly_j >= jy; ly_j--) {
            int lz_j = lj - lx_j - ly_j;
            if (lz_j < jz) continue;
            int j = cart_address(lj, lx_j, ly_j, lz_j);
            double cxyz = cx[(jx+lx_j*lj1)*WARP_SIZE]
                        * cy[(jy+ly_j*lj1)*WARP_SIZE]
                        * cz[(jz+lz_j*lj1)*WARP_SIZE];
            out += cxyz * dm[i*nao+j];
        } }
    } }
    return out;
}

template <int L> __device__ static
void dm_to_dm_xyz(double *dm_xyz, double *dm, int nao, int li, int lj,
                  double *ri, double *rj, double cicj)
{
    int thread_id = threadIdx.x;
    int sp_id = thread_id % WARP_SIZE;
    int warp_id = thread_id / WARP_SIZE;
    int lj1 = lj + 1;
    extern __shared__ double cache[];
    double *cx = cache + sp_id;
    double *cy = cx + lj1 * lj1 * WARP_SIZE;
    double *cz = cy + lj1 * lj1 * WARP_SIZE;

    for (int n = warp_id; n < 3; n += WARPS) {
        dm_xyz_coeff(cx+n*lj1*lj1*WARP_SIZE, ri[n], rj[n], lj);
    }
    __syncthreads();

    constexpr int nf3 = (L+1)*(L+2)*(L+3)/6;
    Fold3Index *fold3idx = c_i_in_fold3idx + L*nf3/4;
    for (int n = warp_id; n < nf3; n += WARPS) {
        int lx = fold3idx[n].x;
        int ly = fold3idx[n].y;
        int lz = fold3idx[n].z;
        double val = sub_dm_xyz(lx, ly, lz, li, lj, nao, cx, cy, cz, dm);
        dm_xyz[n*WARP_SIZE+sp_id] = val * cicj;
    }
    __syncthreads();
}

template <int L> __device__ static
void dm_xyz_to_dm(double *dm, double *dm_xyz, int nao, int li, int lj,
                  double *ri, double *rj, double cicj, double *cache,
                  int npairs_per_block)
{
    int thread_id = threadIdx.x;
    int sp_id = thread_id % WARP_SIZE;
    int warp_id = thread_id / WARP_SIZE;
    int lj1 = lj + 1;
    constexpr int L1 = L + 1;
    double *cx = cache + sp_id;
    double *cy = cx + lj1 * lj1 * WARP_SIZE;
    double *cz = cy + lj1 * lj1 * WARP_SIZE;

    for (int n = warp_id; n < 3; n += WARPS) {
        dm_xyz_coeff(cx+n*lj1*lj1*WARP_SIZE, ri[n], rj[n], lj);
    }
    __syncthreads();

    if (sp_id >= npairs_per_block) {
        return;
    }

    int nfi = (li + 1) * (li + 2) / 2;
    int nfj = (lj + 1) * (lj + 2) / 2;
    int nfij = nfi * nfj;
    Fold2Index *i_fold2idx = c_i_in_fold2idx + li*nfi/3;
    Fold2Index *j_fold2idx = c_i_in_fold2idx + lj*nfj/3;
    for (int n = warp_id; n < nfij; n += WARPS) {
        int i = n / nfj;
        int j = n % nfj;
        int lx_i = i_fold2idx[i].x;
        int ly_i = i_fold2idx[i].y;
        int lz_i = li - lx_i - ly_i;
        int lx_j = j_fold2idx[j].x;
        int ly_j = j_fold2idx[j].y;
        int lz_j = lj - lx_j - ly_j;
        double dm_ij = 0.;
        // TODO: precomputing index
        for (int jx = 0; jx <= lx_j; ++jx) {
            double fac = cicj * cx[(jx+lx_j*lj1)*WARP_SIZE];
            int lx = lx_i + jx;
            for (int jy = 0; jy <= ly_j; ++jy) {
                double cxy = fac * cy[(jy+ly_j*lj1)*WARP_SIZE];
                int ly = ly_i + jy;
                for (int jz = 0; jz <= lz_j; ++jz) {
                    int lz = lz_i + jz;
                    double cxyz = cxy * cz[(jz+lz_j*lj1)*WARP_SIZE];
                    dm_ij += cxyz * dm_xyz[(ADDR2(L,ly,lz)*L1+lx)*WARP_SIZE];
                }
            }
        }
        atomicAdd(dm+i*nao+j, dm_ij);
    }
}
