#include "hip/hip_runtime.h"
/*
 * Copyright 2025 The PySCF Developers. All Rights Reserved.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include <stdio.h>
#include <stdint.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

#include "gvhf-rys/vhf.cuh"
#include "gvhf-md/boys.cu"
#include "gvhf-md/md_j.cuh"

#define RT2_MAX 9
#define IJ_SIZE 11
// 48KB ~18, 96KB ~41, 160KB ~61
#define RT_TMP_SIZE 31
#define RT2_IDX_CACHE_SIZE (35*56)

extern __constant__ uint16_t c_Rt_idx[];
extern __constant__ int8_t c_Rt_tuv_fac[];
extern __constant__ int8_t c_Rt2_efg_phase[];
extern __device__ int Rt2_idx_offsets[];
extern __device__ uint16_t Rt2_kl_ij[];
extern __device__ uint16_t Rt2_ij_kl[];

#define ADDR(l, t, u, v) \
        ((l+1)*(l+2)*(l+3)/6 - ((l)-(t)+1)*((l)-(t)+2)*((l)-(t)+3)/6 + \
         ((l)-(t)+1)*((l)-(t)+2)/2 - ((l)-(t)-(u)+1)*((l)-(t)-(u)+2)/2 + (v))

__device__
inline void iter_Rt_n(double *Rt, double rx, double ry, double rz, int l,
                      int nsq_per_block, int gout_id, int gout_stride)
{
    int nf2 = (l + 1) * (l + 2) / 2;
    int nf3 = nf2 * (l + 3) / 3;
    int offsets = nf3 * l / 4 - l; //l*(l+1)*(l+2)*(l+3)/24 - l;
    uint16_t *p1 = c_Rt_idx + offsets;
    int8_t *tuv_fac = c_Rt_tuv_fac + offsets;
    double Rt_tmp[RT_TMP_SIZE];
    nf2 -= 1; // Drop the first element in Rt. It is assigned outside
    nf3 -= 1;
    for (int n = 0; n < RT_TMP_SIZE; ++n) {
        int i = n * gout_stride + gout_id;
        if (i >= nf3) break;
        Rt_tmp[n] = tuv_fac[i] * Rt[p1[i]*nsq_per_block];
        if (i < l) {
            Rt_tmp[n] += rz * Rt[i*nsq_per_block];
        } else if (i < nf2) {
            Rt_tmp[n] += ry * Rt[(i-l)*nsq_per_block];
        } else {
            Rt_tmp[n] += rx * Rt[(i-nf2)*nsq_per_block];
        }
    }
    __syncthreads();
    for (int n = 0; n < RT_TMP_SIZE; ++n) {
        int i = n * gout_stride + gout_id;
        if (i >= nf3) break;
        Rt[(i+1)*nsq_per_block] = Rt_tmp[n];
    }
}

// gout_pattern = ((li == 0) >> 3) | ((lj == 0) >> 2) | ((lk == 0) >> 1) | (ll == 0);
__global__ static
void pbc_md_j_kernel(RysIntEnvVars envs, JKMatrix jmat, MDBoundsInfo bounds,
                  int threadsx, int threadsy, int tilex, int tiley,
                  uint16_t *pRt2_kl_ij, int8_t *efg_phase)
{
    int *pair_ij_mapping = bounds.pair_ij_mapping;
    int *pair_kl_mapping = bounds.pair_kl_mapping;
    int bsizex = threadsx * tilex;
    int bsizey = threadsy * tiley;
    int task_ij0 = blockIdx.x * bsizex;
    int task_kl0 = blockIdx.y * bsizey;
    int pair_ij0 = pair_ij_mapping[task_ij0];
    int pair_kl0 = pair_kl_mapping[task_kl0];
    float *q_cond = bounds.q_cond;
    if (q_cond[pair_ij0] + q_cond[pair_kl0] < bounds.cutoff) {
        return;
    }
    if (pair_ij_mapping == pair_kl_mapping &&
        // when ij pattern and kl pattern are identical, the 8-fold permutation
        // symmetry can be utilized. Tiles on in the upper triangular part can
        // be skipped. If the last ij task (task_ij0+bsizex-1) is greater than
        // the first kl task (task_kl0), tile is completely inside the triu part.
        task_ij0+bsizex <= task_kl0) {
        return;
    }

    int sq_id = threadIdx.x;
    int gout_id = threadIdx.y;
    int gout_stride = blockDim.y;
    int nsq_per_block = blockDim.x;
    //assert(nsq_per_block == threadsx * threadsy);
    int t_id = gout_id * nsq_per_block + sq_id;
    int tx = sq_id % threadsx;
    int ty = sq_id / threadsx;
    int threads = nsq_per_block * gout_stride;
    int order = bounds.order;
    int *bas = envs.bas;
    int *pair_ij_loc = bounds.pair_ij_loc;
    int *pair_kl_loc = bounds.pair_kl_loc;
    int nbas = envs.nbas;
    double *env = envs.env;
    double *dm = jmat.dm;
    double *vj = jmat.vj;
    int nf3ij = bounds.nf3ij;
    int nf3kl = bounds.nf3kl;

    int npairs_ij = bounds.npairs_ij;
    int npairs_kl = bounds.npairs_kl;
    extern __shared__ double dm_kl_cache[];
    double *Rq_cache = dm_kl_cache + nf3kl*bsizey;
    double *Rp_cache = dm_kl_cache + bsizey*(4+nf3kl);
    double *gamma_inc = dm_kl_cache + bsizey*(4+nf3kl) + threadsx*4 + sq_id;
    double *Rt = gamma_inc + (order+1) * nsq_per_block;
    uint16_t *Rt2_address = pRt2_kl_ij;
    if (nf3ij * nf3kl <= RT2_IDX_CACHE_SIZE) {
        int l4 = bounds.lij + bounds.lkl;
        int nf3 = (l4 + 1) * (l4 + 2) * (l4 + 3) / 6;
        Rt2_address = (uint16_t *)(Rt - sq_id + nf3 * nsq_per_block);
        for (int n = t_id; n < nf3ij * nf3kl; n += threads) {
            Rt2_address[n] = pRt2_kl_ij[n];
        }
    }
    float *qd_ij_max = bounds.qd_ij_max;
    float *qd_kl_max = bounds.qd_kl_max;

    __syncthreads();
    for (int n = t_id; n < bsizey; n += threads) {
        int task_kl = blockIdx.y * bsizey + n;
        if (task_kl < npairs_kl) {
            int pair_kl = pair_kl_mapping[task_kl];
            int ksh = pair_kl / nbas;
            int lsh = pair_kl % nbas;
            double ak = env[bas[ksh*BAS_SLOTS+PTR_EXP]];
            double al = env[bas[lsh*BAS_SLOTS+PTR_EXP]];
            double *rk = env + bas[ksh*BAS_SLOTS+PTR_BAS_COORD];
            double *rl = env + bas[lsh*BAS_SLOTS+PTR_BAS_COORD];
            double akl = ak + al;
            double xkl = (ak * rk[0] + al * rl[0]) / akl;
            double ykl = (ak * rk[1] + al * rl[1]) / akl;
            double zkl = (ak * rk[2] + al * rl[2]) / akl;
            Rq_cache[n+0*bsizey] = xkl;
            Rq_cache[n+1*bsizey] = ykl;
            Rq_cache[n+2*bsizey] = zkl;
            Rq_cache[n+3*bsizey] = akl;
        } else {
            Rq_cache[n+0*bsizey] = 1e5;
            Rq_cache[n+1*bsizey] = 1e5;
            Rq_cache[n+2*bsizey] = 1e5;
            Rq_cache[n+3*bsizey] = 1.;
        }
    }
    {
        int xslots = threadsx * gout_stride;
        int xslot_id = t_id / threadsy;
        int ty = t_id % threadsy;
        for (int n = xslot_id; n < nf3kl * tiley; n += xslots) {
            int kl = n / tiley;
            int batch_kl = n  - kl * tiley;
            int sq_kl = ty + batch_kl * threadsy;
            int task_kl = blockIdx.y * bsizey + sq_kl;
            if (task_kl < npairs_kl) {
                int kl_loc0 = pair_kl_loc[task_kl];
                dm_kl_cache[sq_kl+kl*bsizey] = dm[kl_loc0+kl];
            }
        }
    }

    for (int batch_ij = 0; batch_ij < tilex; ++batch_ij) {
        int task_ij0 = (blockIdx.x * tilex + batch_ij) * threadsx;
        if (task_ij0 >= npairs_ij) {
            break;
        }
        __syncthreads();
        if (t_id < threadsx) {
            int task_ij = task_ij0 + t_id;
            if (task_ij < npairs_ij) {
                int pair_ij = pair_ij_mapping[task_ij];
                int ish = pair_ij / nbas;
                int jsh = pair_ij % nbas;
                double ai = env[bas[ish*BAS_SLOTS+PTR_EXP]];
                double aj = env[bas[jsh*BAS_SLOTS+PTR_EXP]];
                double *ri = env + bas[ish*BAS_SLOTS+PTR_BAS_COORD];
                double *rj = env + bas[jsh*BAS_SLOTS+PTR_BAS_COORD];
                double aij = ai + aj;
                double xij = (ai * ri[0] + aj * rj[0]) / aij;
                double yij = (ai * ri[1] + aj * rj[1]) / aij;
                double zij = (ai * ri[2] + aj * rj[2]) / aij;
                Rp_cache[t_id+0*threadsx] = xij;
                Rp_cache[t_id+1*threadsx] = yij;
                Rp_cache[t_id+2*threadsx] = zij;
                Rp_cache[t_id+3*threadsx] = aij;
            } else {
                Rp_cache[t_id+0*threadsx] = 2e5;
                Rp_cache[t_id+1*threadsx] = 2e5;
                Rp_cache[t_id+2*threadsx] = 2e5;
                Rp_cache[t_id+3*threadsx] = 1.; // aij
            }
        }
        double vj_ij[IJ_SIZE];
#pragma unroll
        for (int n = 0; n < IJ_SIZE; ++n) {
            vj_ij[n] = 0.;
        }
        for (int batch_kl = 0; batch_kl < tiley; ++batch_kl) {
            int task_kl0 = (blockIdx.y * tiley + batch_kl) * threadsy;
            if (task_kl0 >= npairs_kl) {
                break;
            }
            int pair_ij0 = pair_ij_mapping[task_ij0];
            int pair_kl0 = pair_kl_mapping[task_kl0];
            if (qd_ij_max[blockIdx.x*tilex+batch_ij] + q_cond[pair_kl0] < bounds.cutoff &&
                qd_kl_max[blockIdx.y*tiley+batch_kl] + q_cond[pair_ij0] < bounds.cutoff) {
                continue;
            }

            int sq_kl = ty + batch_kl * threadsy;
            int task_ij = task_ij0 + tx;
            int task_kl = task_kl0 + ty;
            double fac = PI_FAC;
            if (task_ij >= npairs_ij || task_kl >= npairs_kl) {
                fac = 0.;
            }
            __syncthreads();
            int bsizey = threadsy * tiley;
            double xij = Rp_cache[tx+0*threadsx];
            double yij = Rp_cache[tx+1*threadsx];
            double zij = Rp_cache[tx+2*threadsx];
            double aij = Rp_cache[tx+3*threadsx];
            double xkl = Rq_cache[sq_kl+0*bsizey];
            double ykl = Rq_cache[sq_kl+1*bsizey];
            double zkl = Rq_cache[sq_kl+2*bsizey];
            double akl = Rq_cache[sq_kl+3*bsizey];
            double xpq = xij - xkl;
            double ypq = yij - ykl;
            double zpq = zij - zkl;
            double rr = xpq*xpq + ypq*ypq + zpq*zpq;
            double theta = aij * akl / (aij + akl);
            if (gout_id == 0) {
                double omega = jmat.omega;
                boys_fn(gamma_inc, theta, rr, omega, fac/(aij*akl*sqrt(aij+akl)),
                        order, 0, nsq_per_block);
                Rt[0] = gamma_inc[order*nsq_per_block];
            }
            for (int n = 1; n <= order; ++n) {
                __syncthreads();
                if (n == 1) {
                    if (gout_id == 0) {
                        double _Rt_0 = Rt[0];
                        Rt[1*nsq_per_block] = zpq * _Rt_0;
                        Rt[2*nsq_per_block] = ypq * _Rt_0;
                        Rt[3*nsq_per_block] = xpq * _Rt_0;
                        Rt[0] = gamma_inc[(order-n)*nsq_per_block];
                    }
                } else if (n == 2) {
                    if (gout_id == 0) {
                        double _Rt_0 = Rt[0];
                        double _Rt_1 = Rt[1*nsq_per_block];
                        double _Rt_2 = Rt[2*nsq_per_block];
                        double _Rt_3 = Rt[3*nsq_per_block];
                        Rt[1*nsq_per_block] = zpq * _Rt_0;
                        Rt[2*nsq_per_block] = zpq * _Rt_1 + _Rt_0;
                        Rt[3*nsq_per_block] = ypq * _Rt_0;
                        Rt[4*nsq_per_block] = ypq * _Rt_1;
                        Rt[5*nsq_per_block] = ypq * _Rt_2 + _Rt_0;
                        Rt[6*nsq_per_block] = xpq * _Rt_0;
                        Rt[7*nsq_per_block] = xpq * _Rt_1;
                        Rt[8*nsq_per_block] = xpq * _Rt_2;
                        Rt[9*nsq_per_block] = xpq * _Rt_3 + _Rt_0;
                        Rt[0] = gamma_inc[(order-n)*nsq_per_block];
                    }
                } else {
                    iter_Rt_n(Rt, xpq, ypq, zpq, n, nsq_per_block, gout_id, gout_stride);
                    if (gout_id == 0) {
                        Rt[0] = gamma_inc[(order-n)*nsq_per_block];
                    }
                }
            }
            __syncthreads();

            if (task_kl < npairs_kl) {
                for (int k = 0; k < nf3kl; ++k) {
                    double dm_kl = efg_phase[k] * dm_kl_cache[k*bsizey+sq_kl];
                    uint16_t *p1_ij = Rt2_address + k * nf3ij;
#pragma unroll
                    for (int n = 0, i = gout_id; n < IJ_SIZE; ++n, i += gout_stride) {
                        if (i >= nf3ij) break;
                        double s = Rt[p1_ij[i]*nsq_per_block];
                        vj_ij[n] += s * dm_kl;
                    }
                }
            }
        }
        {
            double *vj_cache = Rp_cache + t_id;
            int task_ij = task_ij0 + tx;
            int ij_loc0 = pair_ij_loc[task_ij];
#pragma unroll
            for (int n = 0, i = gout_id; n < IJ_SIZE; ++n, i += gout_stride) {
                if (i >= nf3ij+gout_id) break;
                __syncthreads();
                vj_cache[0] = vj_ij[n];
                for (int stride = threadsy/2; stride > 0; stride /= 2) {
                    __syncthreads();
                    if (ty < stride) {
                        vj_cache[0] += vj_cache[stride*threadsx];
                    }
                }
                __syncthreads();
                if (ty == 0 && i < nf3ij && task_ij < npairs_ij) {
                    atomicAdd(vj+ij_loc0+i, vj_cache[0]);
                }
            }
        }
    }
}

extern "C" {
int PBC_build_j(double *vj, double *dm, int n_dm,
                RysIntEnvVars envs, int *scheme, int *shls_slice,
                int npairs_ij, int npairs_kl,
                int *pair_ij_mapping, int *pair_kl_mapping,
                int *pair_ij_loc, int *pair_kl_loc,
                float **qd_ij_max, float **qd_kl_max,
                float *q_cond, float cutoff,
                int *atm, int natm, int *bas, int nbas, double *env)
{
    int ish0 = shls_slice[0];
    int jsh0 = shls_slice[2];
    int ksh0 = shls_slice[4];
    int lsh0 = shls_slice[6];
    int li = bas[ANG_OF + ish0*BAS_SLOTS];
    int lj = bas[ANG_OF + jsh0*BAS_SLOTS];
    int lk = bas[ANG_OF + ksh0*BAS_SLOTS];
    int ll = bas[ANG_OF + lsh0*BAS_SLOTS];
    int lij = li + lj;
    int lkl = lk + ll;
    int order = lij + lkl;
    int nf3ij = (lij+1)*(lij+2)*(lij+3)/6;
    int nf3kl = (lkl+1)*(lkl+2)*(lkl+3)/6;
    int nf3ijkl = (order+1)*(order+2)*(order+3)/6;
    // 16x16 threads are applied to all unrolled code
    float *tile16_qd_ij_max = qd_ij_max[block_id_for_threads(16)];
    float *tile16_qd_kl_max = qd_kl_max[block_id_for_threads(16)];
    MDBoundsInfo bounds = {li, lj, lk, ll, lij, lkl, order, nf3ij, nf3kl, nf3ijkl,
        npairs_ij, npairs_kl, pair_ij_mapping, pair_kl_mapping,
        pair_ij_loc, pair_kl_loc, tile16_qd_ij_max, tile16_qd_kl_max,
        q_cond, cutoff};

    double omega = env[PTR_RANGE_OMEGA];
    JKMatrix jmat = {vj, NULL, dm, n_dm, 0, omega};

    int threads_ij = scheme[0];
    int threads_kl = scheme[1];
    int gout_stride = scheme[2];
    int tilex = scheme[3];
    int tiley = scheme[4];
    int buflen = scheme[5];
    int bsizex = threads_ij * tilex;
    int bsizey = threads_kl * tiley;
    int nsq_per_block = threads_ij * threads_kl;
    dim3 threads(nsq_per_block, gout_stride);
    int blocks_ij = (npairs_ij + bsizex - 1) / bsizex;
    int blocks_kl = (npairs_kl + bsizey - 1) / bsizey;
    dim3 blocks(blocks_ij, blocks_kl);
    uint16_t *pRt2_kl_ij;
    int8_t *efg_phase;
    hipGetSymbolAddress((void**)&pRt2_kl_ij, Rt2_kl_ij);
    hipGetSymbolAddress((void**)&efg_phase, c_Rt2_efg_phase);
    pRt2_kl_ij += offset_for_Rt2_idx(lij, lkl);
    efg_phase += offset_for_Rt2_idx(0, lkl);
    if (1){//!pbc_md_j_unrolled(&envs, &jmat, &bounds, omega)) {
        bounds.qd_ij_max = qd_ij_max[block_id_for_threads(threads_ij)];
        bounds.qd_kl_max = qd_kl_max[block_id_for_threads(threads_kl)];
        pbc_md_j_kernel<<<blocks, threads, buflen>>>(
            envs, jmat, bounds, threads_ij, threads_kl, tilex, tiley,
            pRt2_kl_ij, efg_phase);
    }
    hipError_t err = hipGetLastError();
    if (err != hipSuccess) {
        fprintf(stderr, "CUDA Error in MD_build_j: %s\n", hipGetErrorString(err));
        return 1;
    }
    return 0;
}

int PBC_build_j_init(int shm_size)
{
    hipFuncSetAttribute(reinterpret_cast<const void*>(pbc_md_j_kernel), hipFuncAttributeMaxDynamicSharedMemorySize, shm_size);
    hipError_t err = hipGetLastError();
    if (err != hipSuccess) {
        fprintf(stderr, "Failed to set CUDA shm size %d: %s\n", shm_size,
                hipGetErrorString(err));
        return 1;
    }
    return 0;
}
}
