#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdint.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>

#include "gvhf-rys/vhf.cuh"
#include "gvhf-rys/gamma_inc.cu"

#define TILEX   2
#define TILEY   4

extern __constant__ uint16_t c_Rt_idx[];
extern __constant__ uint16_t c_Rt_offsets[];

#define ADDR(l, t, u, v) \
        ((l+1)*(l+2)*(l+3)/6 - ((l)-(t)+1)*((l)-(t)+2)*((l)-(t)+3)/6 + \
         ((l)-(t)+1)*((l)-(t)+2)/2 - ((l)-(t)-(u)+1)*((l)-(t)-(u)+2)/2 + (v))

__device__
static void iter_Rt_n(double *out, double *Rt, double rx, double ry, double rz,
                      int l, int sq_id, int nsq_per_block)
{
    uint16_t *p1 = c_Rt_idx + c_Rt_offsets[l];
    double *pout = out + nsq_per_block;
    int k = 0;
    for (int v = 0, i = 0; v < l; ++v) {
        pout[sq_id+k*nsq_per_block] = rz * Rt[sq_id+i*nsq_per_block] + v * Rt[sq_id+p1[k]*nsq_per_block];
        ++k; ++i;
    }
    for (int u = 0, i = 0; u < l; ++u) {
        for (int v = 0; v < l-u; ++v) {
            pout[sq_id+k*nsq_per_block] = ry * Rt[sq_id+i*nsq_per_block] + u * Rt[sq_id+p1[k]*nsq_per_block];
            ++k; ++i;
        }
    }
    //int nf3 = l*(l+1)*(l+2)/6;
    //Fold3Index *fold3idx = c_i_in_fold3idx + (l-1)*nf3/4;;
    //for (int i = 0; i < nf3; ++i) {
    //    Fold3Index f3i = fold3idx[i];
    //    int t = f3i.x;
    //    pout[sq_id+(k+i)*nsq_per_block] = rx * Rt[sq_id+i*nsq_per_block]
    //        + t * Rt[sq_id+p1[k+i]*nsq_per_block];
    //}
    for (int t = 0, i = 0; t < l; ++t) {
        // corresponding to the nested loops
        // for (u = 0; u < l-t; ++u) for (v = 0; v < l-t-u; ++v)
        for (int uv = 0; uv < (l-t) * (l-t+1) / 2; ++uv) {
            pout[sq_id+(k+i)*nsq_per_block] = rx * Rt[sq_id+i*nsq_per_block]
                + t * Rt[sq_id+p1[k+i]*nsq_per_block];
            ++i;
        }
    }
}

#if CUDA_VERSION >= 12040
__global__ __maxnreg__(128)
#else
__global__
#endif
void md_j_kernel(RysIntEnvVars envs, JKMatrix jk, BoundsInfo bounds)
{
    int *pair_ij_mapping = bounds.tile_ij_mapping;
    int *pair_kl_mapping = bounds.tile_kl_mapping;
    int threadsx = blockDim.x;
    int threadsy = blockDim.y;
    int bsizex = threadsx * TILEX;
    int bsizey = threadsy * TILEY;
    int task_ij0 = blockIdx.x * bsizex;
    int task_kl0 = blockIdx.y * bsizey;
    int pair_ij0 = pair_ij_mapping[task_ij0];
    int pair_kl0 = pair_kl_mapping[task_kl0];
    float *q_cond = bounds.q_cond;
    if (q_cond[pair_ij0] + q_cond[pair_kl0] < bounds.cutoff) {
        return;
    }

    int tx = threadIdx.x;
    int ty = threadIdx.y;
    int sq_id = tx + threadsx * ty;
    int nsq_per_block = threadsx * threadsy;
    int gout_id = threadIdx.z;
    int gout_stride = blockDim.z;
    int t_id = sq_id + nsq_per_block * gout_id;
    int threads = nsq_per_block * gout_stride;
    int li = bounds.li;
    int lj = bounds.lj;
    int lk = bounds.lk;
    int ll = bounds.ll;
    int lij = li + lj;
    int lkl = lk + ll;
    int order = lij + lkl;
    int nf3ijkl = (order+1)*(order+2)*(order+3)/6;
    int *bas = envs.bas;
    int *dm_pair_loc = envs.ao_loc;
    int nbas = envs.nbas;
    double *env = envs.env;
    double *dm = jk.dm;
    double *vj = jk.vj;
    int nf3ij = (lij+1)*(lij+2)*(lij+3)/6;
    int nf3kl = (lkl+1)*(lkl+2)*(lkl+3)/6;
    int ij_fold3idx_cum = lij*nf3ij/4;
    int kl_fold3idx_cum = lkl*nf3kl/4;
    Fold3Index *ij_fold3idx = c_i_in_fold3idx + ij_fold3idx_cum;
    Fold3Index *kl_fold3idx = c_i_in_fold3idx + kl_fold3idx_cum;

    int npairs_ij = bounds.npairs_ij;
    int npairs_kl = bounds.npairs_kl;
    extern __shared__ double gamma_inc[];
    double *Rp_cache = gamma_inc + (order+1) * nsq_per_block;
    double *Rq_cache = Rp_cache + bsizex*4;
    double *vj_ij_cache = Rq_cache + bsizey*4;
    double *vj_kl_cache = vj_ij_cache + nf3ij * bsizex;

    // zero out all cache;
    for (int n = t_id; n < (bsizex*4 + bsizey*4 + nf3ij*bsizex + nf3kl*bsizey); n += threads) {
        Rp_cache[n] = 0.;
    }
    __syncthreads();
    if (t_id < bsizex) {
        int task_ij = blockIdx.x * bsizex + t_id;
        if (task_ij < npairs_ij) {
            int pair_ij = pair_ij_mapping[task_ij];
            int ish = pair_ij / nbas;
            int jsh = pair_ij % nbas;
            double ai = env[bas[ish*BAS_SLOTS+PTR_EXP]];
            double aj = env[bas[jsh*BAS_SLOTS+PTR_EXP]];
            double *ri = env + bas[ish*BAS_SLOTS+PTR_BAS_COORD];
            double *rj = env + bas[jsh*BAS_SLOTS+PTR_BAS_COORD];
            double aij = ai + aj;
            double xij = (ai * ri[0] + aj * rj[0]) / aij;
            double yij = (ai * ri[1] + aj * rj[1]) / aij;
            double zij = (ai * ri[2] + aj * rj[2]) / aij;
            Rp_cache[t_id+0*bsizex] = xij;
            Rp_cache[t_id+1*bsizex] = yij;
            Rp_cache[t_id+2*bsizex] = zij;
            Rp_cache[t_id+3*bsizex] = aij;
        } else {
            Rp_cache[t_id+3*bsizex] = 1.;
        }
    }
    if (t_id < bsizey) {
        int task_kl = blockIdx.y * bsizey + t_id;
        if (task_kl < npairs_kl) {
            int pair_kl = pair_kl_mapping[task_kl];
            int ksh = pair_kl / nbas;
            int lsh = pair_kl % nbas;
            double ak = env[bas[ksh*BAS_SLOTS+PTR_EXP]];
            double al = env[bas[lsh*BAS_SLOTS+PTR_EXP]];
            double *rk = env + bas[ksh*BAS_SLOTS+PTR_BAS_COORD];
            double *rl = env + bas[lsh*BAS_SLOTS+PTR_BAS_COORD];
            double akl = ak + al;
            double xkl = (ak * rk[0] + al * rl[0]) / akl;
            double ykl = (ak * rk[1] + al * rl[1]) / akl;
            double zkl = (ak * rk[2] + al * rl[2]) / akl;
            Rq_cache[t_id+0*bsizey] = xkl;
            Rq_cache[t_id+1*bsizey] = ykl;
            Rq_cache[t_id+2*bsizey] = zkl;
            Rq_cache[t_id+3*bsizey] = akl;
        } else {
            Rq_cache[t_id+3*bsizey] = 1.;
        }
    }
    //for (int n = ty+threadsy*gout_id; n < nf3ij*TILEX; n += threadsy*gout_stride) {
    //    int i = n / TILEX;
    //    int tile = n % TILEX;
    //    int task_ij = blockIdx.x * bsizex + tile * threadsx + tx;
    //    if (task_ij < npairs_ij) {
    //        int pair_ij = pair_ij_mapping[task_ij];
    //        int dm_ij_pair0 = dm_pair_loc[pair_ij];
    //        int sq_ij = tx + tile * threadsx;
    //        dm_ij_cache[sq_ij+i*bsizex] = dm[dm_ij_pair0+i];
    //    }
    //}
    //for (int n = tx+threadsx*gout_id; n < nf3kl*TILEY; n += threadsx*gout_stride) {
    //    int i = n / TILEY;
    //    int tile = n % TILEY;
    //    int task_kl = blockIdx.y * bsizey + tile * threadsy + ty;
    //    if (task_kl < npairs_kl) {
    //        int pair_kl = pair_kl_mapping[task_kl];
    //        int dm_kl_pair0 = dm_pair_loc[pair_kl];
    //        int sq_kl = ty + tile * threadsy;
    //        dm_kl_cache[sq_kl+i*bsizey] = dm[dm_kl_pair0+i];
    //    }
    //}
    __syncthreads();

    for (int batch_ij = 0; batch_ij < TILEX; ++batch_ij) {
    for (int batch_kl = 0; batch_kl < TILEY; ++batch_kl) {
        int task_ij0 = blockIdx.x * bsizex + batch_ij * threadsx;
        int task_kl0 = blockIdx.y * bsizey + batch_kl * threadsy;
        if (task_ij0 >= npairs_ij || task_kl0 >= npairs_kl) {
            continue;
        }
        int pair_ij0 = pair_ij_mapping[task_ij0];
        int pair_kl0 = pair_kl_mapping[task_kl0];
        if (q_cond[pair_ij0] + q_cond[pair_kl0] < bounds.cutoff) {
            continue;
        }

        int sq_ij = tx + batch_ij * threadsx;
        int sq_kl = ty + batch_kl * threadsy;
        int task_ij = task_ij0 + tx;
        int task_kl = task_kl0 + ty;
        double fac_sym = PI_FAC;
        if (task_ij >= npairs_ij) {
            task_ij = task_ij0;
            fac_sym = 0.;
        }
        if (task_kl >= npairs_kl) {
            task_kl = task_kl0;
            fac_sym = 0.;
        }
        int pair_ij = pair_ij_mapping[task_ij];
        int pair_kl = pair_kl_mapping[task_kl];

        int ish = pair_ij / nbas;
        int jsh = pair_ij % nbas;
        int ksh = pair_kl / nbas;
        int lsh = pair_kl % nbas;
        if (ish == jsh) fac_sym *= .5;
        if (ksh == lsh) fac_sym *= .5;
        if (pair_ij_mapping == pair_kl_mapping) {
            if (task_ij == task_kl) fac_sym *= .5;
            // TODO: skip certain blocks when task_ij < task_kl
            if (task_ij < task_kl) fac_sym = 0.;
        }
        int dm_ij_pair0 = dm_pair_loc[pair_ij];
        int dm_kl_pair0 = dm_pair_loc[pair_kl];
        double *Rt, *buf;
        if (gout_id == 0) {
            double xij = Rp_cache[sq_ij+0*bsizex];
            double yij = Rp_cache[sq_ij+1*bsizex];
            double zij = Rp_cache[sq_ij+2*bsizex];
            double aij = Rp_cache[sq_ij+3*bsizex];
            double xkl = Rq_cache[sq_kl+0*bsizey];
            double ykl = Rq_cache[sq_kl+1*bsizey];
            double zkl = Rq_cache[sq_kl+2*bsizey];
            double akl = Rq_cache[sq_kl+3*bsizey];
            double fac = fac_sym / (aij*akl*sqrt(aij+akl));
            double xpq = xij - xkl;
            double ypq = yij - ykl;
            double zpq = zij - zkl;
            double rr = xpq*xpq + ypq*ypq + zpq*zpq;
            double theta = aij * akl / (aij + akl);
            double theta_rr = theta * rr;
            eval_gamma_inc_fn(gamma_inc, theta_rr, order, sq_id, nsq_per_block);
            double a2 = -2. * theta;
            gamma_inc[sq_id] *= fac;
            for (int i = 1; i <= order; i++) {
                fac *= a2;
                gamma_inc[sq_id+i*nsq_per_block] *= fac;
            }
            if (order % 2 == 0) {
                Rt = vj_kl_cache + nf3kl*bsizey;
                buf = Rt + nf3ijkl * nsq_per_block;
            } else {
                buf = vj_kl_cache + nf3kl*bsizey;
                Rt = buf + nf3ijkl * nsq_per_block;
            }
            Rt[sq_id] = gamma_inc[sq_id+order*nsq_per_block];
            for (int n = 1; n <= order; ++n) {
                // swap input and output
                double *tmp = buf;
                buf = Rt;
                Rt = tmp;
                Rt[sq_id] = gamma_inc[sq_id+(order-n)*nsq_per_block];
                switch (n) {
                case 1:
                    Rt[sq_id+1*nsq_per_block] = zpq * buf[sq_id+0*nsq_per_block];
                    Rt[sq_id+2*nsq_per_block] = ypq * buf[sq_id+0*nsq_per_block];
                    Rt[sq_id+3*nsq_per_block] = xpq * buf[sq_id+0*nsq_per_block];
                    break;
                case 2:
                    Rt[sq_id+1*nsq_per_block] = zpq * buf[sq_id+0*nsq_per_block];
                    Rt[sq_id+2*nsq_per_block] = zpq * buf[sq_id+1*nsq_per_block] + buf[sq_id+0*nsq_per_block];
                    Rt[sq_id+3*nsq_per_block] = ypq * buf[sq_id+0*nsq_per_block];
                    Rt[sq_id+4*nsq_per_block] = ypq * buf[sq_id+1*nsq_per_block];
                    Rt[sq_id+5*nsq_per_block] = ypq * buf[sq_id+2*nsq_per_block] + buf[sq_id+0*nsq_per_block];
                    Rt[sq_id+6*nsq_per_block] = xpq * buf[sq_id+0*nsq_per_block];
                    Rt[sq_id+7*nsq_per_block] = xpq * buf[sq_id+1*nsq_per_block];
                    Rt[sq_id+8*nsq_per_block] = xpq * buf[sq_id+2*nsq_per_block];
                    Rt[sq_id+9*nsq_per_block] = xpq * buf[sq_id+3*nsq_per_block] + buf[sq_id+0*nsq_per_block];
                    break;
                case 3:
                    Rt[sq_id+1*nsq_per_block] = zpq * buf[sq_id+0*nsq_per_block];
                    Rt[sq_id+2*nsq_per_block] = zpq * buf[sq_id+1*nsq_per_block] + buf[sq_id+0*nsq_per_block];
                    Rt[sq_id+3*nsq_per_block] = zpq * buf[sq_id+2*nsq_per_block] + 2 * buf[sq_id+1*nsq_per_block];
                    Rt[sq_id+4*nsq_per_block] = ypq * buf[sq_id+0*nsq_per_block];
                    Rt[sq_id+5*nsq_per_block] = ypq * buf[sq_id+1*nsq_per_block];
                    Rt[sq_id+6*nsq_per_block] = ypq * buf[sq_id+2*nsq_per_block];
                    Rt[sq_id+7*nsq_per_block] = ypq * buf[sq_id+3*nsq_per_block] + buf[sq_id+0*nsq_per_block];
                    Rt[sq_id+8*nsq_per_block] = ypq * buf[sq_id+4*nsq_per_block] + buf[sq_id+1*nsq_per_block];
                    Rt[sq_id+9*nsq_per_block] = ypq * buf[sq_id+5*nsq_per_block] + 2 * buf[sq_id+3*nsq_per_block];
                    Rt[sq_id+10*nsq_per_block] = xpq * buf[sq_id+0*nsq_per_block];
                    Rt[sq_id+11*nsq_per_block] = xpq * buf[sq_id+1*nsq_per_block];
                    Rt[sq_id+12*nsq_per_block] = xpq * buf[sq_id+2*nsq_per_block];
                    Rt[sq_id+13*nsq_per_block] = xpq * buf[sq_id+3*nsq_per_block];
                    Rt[sq_id+14*nsq_per_block] = xpq * buf[sq_id+4*nsq_per_block];
                    Rt[sq_id+15*nsq_per_block] = xpq * buf[sq_id+5*nsq_per_block];
                    Rt[sq_id+16*nsq_per_block] = xpq * buf[sq_id+6*nsq_per_block] + buf[sq_id+0*nsq_per_block];
                    Rt[sq_id+17*nsq_per_block] = xpq * buf[sq_id+7*nsq_per_block] + buf[sq_id+1*nsq_per_block];
                    Rt[sq_id+18*nsq_per_block] = xpq * buf[sq_id+8*nsq_per_block] + buf[sq_id+3*nsq_per_block];
                    Rt[sq_id+19*nsq_per_block] = xpq * buf[sq_id+9*nsq_per_block] + 2 * buf[sq_id+6*nsq_per_block];
                    break;
                case 4:
                    Rt[sq_id+1*nsq_per_block] = zpq * buf[sq_id+0*nsq_per_block];
                    Rt[sq_id+2*nsq_per_block] = zpq * buf[sq_id+1*nsq_per_block] + buf[sq_id+0*nsq_per_block];
                    Rt[sq_id+3*nsq_per_block] = zpq * buf[sq_id+2*nsq_per_block] + 2 * buf[sq_id+1*nsq_per_block];
                    Rt[sq_id+4*nsq_per_block] = zpq * buf[sq_id+3*nsq_per_block] + 3 * buf[sq_id+2*nsq_per_block];
                    Rt[sq_id+5*nsq_per_block] = ypq * buf[sq_id+0*nsq_per_block];
                    Rt[sq_id+6*nsq_per_block] = ypq * buf[sq_id+1*nsq_per_block];
                    Rt[sq_id+7*nsq_per_block] = ypq * buf[sq_id+2*nsq_per_block];
                    Rt[sq_id+8*nsq_per_block] = ypq * buf[sq_id+3*nsq_per_block];
                    Rt[sq_id+9*nsq_per_block] = ypq * buf[sq_id+4*nsq_per_block] + buf[sq_id+0*nsq_per_block];
                    Rt[sq_id+10*nsq_per_block] = ypq * buf[sq_id+5*nsq_per_block] + buf[sq_id+1*nsq_per_block];
                    Rt[sq_id+11*nsq_per_block] = ypq * buf[sq_id+6*nsq_per_block] + buf[sq_id+2*nsq_per_block];
                    Rt[sq_id+12*nsq_per_block] = ypq * buf[sq_id+7*nsq_per_block] + 2 * buf[sq_id+4*nsq_per_block];
                    Rt[sq_id+13*nsq_per_block] = ypq * buf[sq_id+8*nsq_per_block] + 2 * buf[sq_id+5*nsq_per_block];
                    Rt[sq_id+14*nsq_per_block] = ypq * buf[sq_id+9*nsq_per_block] + 3 * buf[sq_id+7*nsq_per_block];
                    Rt[sq_id+15*nsq_per_block] = xpq * buf[sq_id+0*nsq_per_block];
                    Rt[sq_id+16*nsq_per_block] = xpq * buf[sq_id+1*nsq_per_block];
                    Rt[sq_id+17*nsq_per_block] = xpq * buf[sq_id+2*nsq_per_block];
                    Rt[sq_id+18*nsq_per_block] = xpq * buf[sq_id+3*nsq_per_block];
                    Rt[sq_id+19*nsq_per_block] = xpq * buf[sq_id+4*nsq_per_block];
                    Rt[sq_id+20*nsq_per_block] = xpq * buf[sq_id+5*nsq_per_block];
                    Rt[sq_id+21*nsq_per_block] = xpq * buf[sq_id+6*nsq_per_block];
                    Rt[sq_id+22*nsq_per_block] = xpq * buf[sq_id+7*nsq_per_block];
                    Rt[sq_id+23*nsq_per_block] = xpq * buf[sq_id+8*nsq_per_block];
                    Rt[sq_id+24*nsq_per_block] = xpq * buf[sq_id+9*nsq_per_block];
                    Rt[sq_id+25*nsq_per_block] = xpq * buf[sq_id+10*nsq_per_block] + buf[sq_id+0*nsq_per_block];
                    Rt[sq_id+26*nsq_per_block] = xpq * buf[sq_id+11*nsq_per_block] + buf[sq_id+1*nsq_per_block];
                    Rt[sq_id+27*nsq_per_block] = xpq * buf[sq_id+12*nsq_per_block] + buf[sq_id+2*nsq_per_block];
                    Rt[sq_id+28*nsq_per_block] = xpq * buf[sq_id+13*nsq_per_block] + buf[sq_id+4*nsq_per_block];
                    Rt[sq_id+29*nsq_per_block] = xpq * buf[sq_id+14*nsq_per_block] + buf[sq_id+5*nsq_per_block];
                    Rt[sq_id+30*nsq_per_block] = xpq * buf[sq_id+15*nsq_per_block] + buf[sq_id+7*nsq_per_block];
                    Rt[sq_id+31*nsq_per_block] = xpq * buf[sq_id+16*nsq_per_block] + 2 * buf[sq_id+10*nsq_per_block];
                    Rt[sq_id+32*nsq_per_block] = xpq * buf[sq_id+17*nsq_per_block] + 2 * buf[sq_id+11*nsq_per_block];
                    Rt[sq_id+33*nsq_per_block] = xpq * buf[sq_id+18*nsq_per_block] + 2 * buf[sq_id+13*nsq_per_block];
                    Rt[sq_id+34*nsq_per_block] = xpq * buf[sq_id+19*nsq_per_block] + 3 * buf[sq_id+16*nsq_per_block];
                    break;
                default: iter_Rt_n(Rt, buf, xpq, ypq, zpq, n, sq_id, nsq_per_block);
                }
            }
        }

        Rt = vj_kl_cache + nf3kl*bsizey;
        double *vj_cache = Rt + nf3ijkl * nsq_per_block;
        //for (k = 0, e = 0; e <= l1; ++e) {
        //for (f = 0; f <= l1-e; ++f) {
        //for (g = 0; g <= l1-e-f; ++g, ++k) {
        //    double rho_kl_val = rho_kl[k];
        //    double jvec_kl_val = 0.;
        //    double fac = 1;
        //    if ((e + f + g) % 2 != 0) {
        //        fac = -1;
        //    }
        //    for (i = 0, t = 0; t <= l2; ++t) {
        //    for (u = 0; u <= l2-t; ++u) {
        //    for (v = 0; v <= l2-t-u; ++v, ++i) {
        //        s = fac * R[e+t,f+u,g+v]
        //        jvec_kl_val += s * rho_ij[i];
        //        jvec_ij[i]  += s * rho_kl_val;
        //    } } }
        //    jvec_kl[k] += jvec_kl_val;
        //} } }
        for (int k = gout_id; k < nf3kl+gout_id; k += gout_stride) {
            __syncthreads();
            double vj_kl = 0.;
            if (k < nf3kl) {
                Fold3Index f3k = kl_fold3idx[k];
                int e = f3k.x;
                int f = f3k.y;
                int g = f3k.z;
                double fac = 1.;
                if ((e + f + g) % 2 != 0) {
                    fac = -1.;
                }
                for (int i = 0, t = 0; t <= lij; ++t) {
                for (int u = 0; u <= lij-t; ++u) {
                for (int v = 0; v <= lij-t-u; ++v, ++i) {
                    //double s = Rt[sq_id+ADDR(order,e+t,f+u,g+v)*nsq_per_block];
                    int ix = order-e-t;
                    int xoffset = ix*(ix+1)*(ix+2)/6;
                    int iy = ix-f-u;
                    int i2y = (iy+1)*(iy+2)/2;
                    double s = Rt[sq_id+(nf3ijkl-xoffset-i2y+g+v)*nsq_per_block];
                    vj_kl += fac * s * dm[dm_ij_pair0+i];
                } } }
                //atomicAdd(vj+dm_kl_pair0+k, vj_kl);
            }
            vj_cache[t_id] = vj_kl;
            for (int stride = threadsx/2; stride > 0; stride /= 2) {
                __syncthreads();
                if (tx < stride) {
                    vj_cache[t_id] += vj_cache[t_id + stride];
                }
            }
            __syncthreads();
            if (tx == 0 && task_kl0+ty < npairs_kl) {
                vj_kl_cache[sq_kl+k*bsizey] += vj_cache[t_id];
            }
        }

        for (int i = gout_id; i < nf3ij+gout_id; i += gout_stride) {
            __syncthreads();
            double vj_ij = 0.;
            if (i < nf3ij) {
                Fold3Index f3i = ij_fold3idx[i];
                int t = f3i.x;
                int u = f3i.y;
                int v = f3i.z;
                for (int k = 0, e = 0; e <= lkl; ++e) {
                for (int f = 0; f <= lkl-e; ++f) {
                for (int g = 0; g <= lkl-e-f; ++g, ++k) {
                    //double s = Rt[sq_id+ADDR(order,e+t,f+u,g+v)*nsq_per_block];
                    int ix = order-e-t;
                    int xoffset = ix*(ix+1)*(ix+2)/6;
                    int iy = ix-f-u;
                    int i2y = (iy+1)*(iy+2)/2;
                    double s = Rt[sq_id+(nf3ijkl-xoffset-i2y+g+v)*nsq_per_block];
                    double d = dm[dm_kl_pair0+k];
                    if ((e + f + g) % 2 == 0) {
                        vj_ij += s * d;
                    } else {
                        vj_ij -= s * d;
                    }
                } } }
                //atomicAdd(vj+dm_ij_pair0+i, vj_ij);
            }
            vj_cache[t_id] = vj_ij;
            for (int stride = threadsy/2; stride > 0; stride /= 2) {
                __syncthreads();
                if (ty < stride) {
                    vj_cache[t_id] += vj_cache[t_id + stride*threadsx];
                }
            }
            __syncthreads();
            if (ty == 0 && task_ij0+tx < npairs_ij) {
                vj_ij_cache[sq_ij+i*bsizex] += vj_cache[t_id];
            }
        }
        __syncthreads();
    } }

    for (int n = ty+threadsy*gout_id; n < nf3ij*TILEX; n += threadsy*gout_stride) {
        int i = n / TILEX;
        int tile = n % TILEX;
        int task_ij = blockIdx.x * bsizex + tile * threadsx + tx;
        if (task_ij < npairs_ij) {
            int pair_ij = pair_ij_mapping[task_ij];
            int dm_ij_pair0 = dm_pair_loc[pair_ij];
            int sq_ij = tx + tile * threadsx;
            atomicAdd(vj+dm_ij_pair0+i, vj_ij_cache[sq_ij+i*bsizex]);
        }
    }
    for (int n = tx+threadsx*gout_id; n < nf3kl*TILEY; n += threadsx*gout_stride) {
        int i = n / TILEY;
        int tile = n % TILEY;
        int task_kl = blockIdx.y * bsizey + tile * threadsy + ty;
        if (task_kl < npairs_kl) {
            int pair_kl = pair_kl_mapping[task_kl];
            int dm_kl_pair0 = dm_pair_loc[pair_kl];
            int sq_kl = ty + tile * threadsy;
            atomicAdd(vj+dm_kl_pair0+i, vj_kl_cache[sq_kl+i*bsizey]);
        }
    }
}
