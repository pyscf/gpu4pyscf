#include "hip/hip_runtime.h"
/*
 * Copyright 2021-2025 The PySCF Developers. All Rights Reserved.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */


template <int orderi, int orderj> __device__
void type1_cart_kernel(double *gctr,
                const int LI, const int LJ,
                const int ish, const int jsh, const int ksh,
                const int *ecpbas, const int *ecploc,
                const int *atm, const int *bas, const double *env)
{
    extern __shared__ double smem[];

    const int npi = bas[NPRIM_OF+ish*BAS_SLOTS];
    const int npj = bas[NPRIM_OF+jsh*BAS_SLOTS];
    const double *ai = env + bas[PTR_EXP+ish*BAS_SLOTS];
    const double *aj = env + bas[PTR_EXP+jsh*BAS_SLOTS];
    const double *ci = env + bas[PTR_COEFF+ish*BAS_SLOTS];
    const double *cj = env + bas[PTR_COEFF+jsh*BAS_SLOTS];
    const double *ri = env + atm[PTR_COORD+bas[ATOM_OF+ish*BAS_SLOTS]*ATM_SLOTS];
    const double *rj = env + atm[PTR_COORD+bas[ATOM_OF+jsh*BAS_SLOTS]*ATM_SLOTS];

    const int atm_id = ecpbas[ATOM_OF+ecploc[ksh]*BAS_SLOTS];
    const double *rc = env + atm[PTR_COORD+atm_id*ATM_SLOTS];

    double rca[3], rcb[3];
    rca[0] = rc[0] - ri[0];
    rca[1] = rc[1] - ri[1];
    rca[2] = rc[2] - ri[2];
    rcb[0] = rc[0] - rj[0];
    rcb[1] = rc[1] - rj[1];
    rcb[2] = rc[2] - rj[2];
    const double r2ca = rca[0]*rca[0] + rca[1]*rca[1] + rca[2]*rca[2];
    const double r2cb = rcb[0]*rcb[0] + rcb[1]*rcb[1] + rcb[2]*rcb[2];

    double ur = 0.0;
    for (int kbas = ecploc[ksh]; kbas < ecploc[ksh+1]; kbas++){
        ur += rad_part(kbas, ecpbas, env);
    }

    const int LIJ1 = LI+LJ+1;
    const int LIJ3 = LIJ1*LIJ1*LIJ1;

    double *rad_ang = smem;
    for (int i = threadIdx.x; i < LIJ3; i+=blockDim.x) {
        rad_ang[i] = 0;
    }
    __syncthreads();

    double *rad_all = rad_ang + LIJ3;
    const double fac = 16.0 * M_PI * M_PI * _common_fac[LI] * _common_fac[LJ];
    for (int ip = 0; ip < npi; ip++){
        for (int jp = 0; jp < npj; jp++){
            double rij[3];
            double ai_prim = ai[ip];
            double aj_prim = aj[jp];
            rij[0] = ai_prim * rca[0] + aj_prim * rcb[0];
            rij[1] = ai_prim * rca[1] + aj_prim * rcb[1];
            rij[2] = ai_prim * rca[2] + aj_prim * rcb[2];
            const double k = 2.0 * norm3d(rij[0], rij[1], rij[2]);
            const double aij = ai_prim + aj_prim;
            type1_rad_part(rad_all, LI+LJ, k, aij, ur);

            const double eij = exp(-ai_prim*r2ca - aj_prim*r2cb);
            const double eaij = eij * pow(-2.0*ai_prim, orderi) * pow(-2.0*aj_prim, orderj);
            const double ceij = eaij * ci[ip] * cj[jp];
            type1_rad_ang(rad_ang, LI+LJ, rij, rad_all, fac*ceij);
            __syncthreads();
        }
    }

    constexpr int NFI_MAX = (AO_LMAX+orderi+1)*(AO_LMAX+orderi+2)/2;
    constexpr int NFJ_MAX = (AO_LMAX+orderj+1)*(AO_LMAX+orderj+2)/2;
    double fi[3*NFI_MAX];
    double fj[3*NFJ_MAX];
    cache_fac(fi, LI, rca);
    cache_fac(fj, LJ, rcb);

    const int nfi = (LI+1) * (LI+2) / 2;
    const int nfj = (LJ+1) * (LJ+2) / 2;
    for (int ij = threadIdx.x; ij < nfi*nfj; ij+=blockDim.x){
        const int mi = ij%nfi;
        const int mj = ij/nfi;

        const int iy = _cart_pow_y[mi];
        const int iz = _cart_pow_z[mi];
        const int ix = LI - iy - iz;

        double* fx_i = fi + (ix+1)*ix/2;
        double* fy_i = fi + (iy+1)*iy/2 + nfi;
        double* fz_i = fi + (iz+1)*iz/2 + 2*nfi;

        const int jy = _cart_pow_y[mj];
        const int jz = _cart_pow_z[mj];
        const int jx = LJ - jy - jz;
        double* fx_j = fj + (jx+1)*jx/2;
        double* fy_j = fj + (jy+1)*jy/2 + nfj;
        double* fz_j = fj + (jz+1)*jz/2 + 2*nfj;

        // cache ifac and jfac in register
        double tmp = 0.0;
        for (int i1 = 0; i1 <= ix; i1++){
        for (int i2 = 0; i2 <= iy; i2++){
        for (int i3 = 0; i3 <= iz; i3++){
            double ifac = fx_i[i1] * fy_i[i2] * fz_i[i3];
            for (int j1 = 0; j1 <= jx; j1++){
            for (int j2 = 0; j2 <= jy; j2++){
            for (int j3 = 0; j3 <= jz; j3++){
                double jfac = fx_j[j1] * fy_j[j2] * fz_j[j3];
                const int ijr = (i1+j1)*LIJ1*LIJ1 + (i2+j2)*LIJ1 + (i3+j3);
                tmp += ifac * jfac * rad_ang[ijr];
            }}}
        }}}
        gctr[ij] = tmp;
    }
    return;
}


__global__
void type1_cart_ip1(double *gctr,
                const int LI, const int LJ,
                const int *ao_loc, const int nao,
                const int *tasks, const int ntasks,
                const int *ecpbas, const int *ecploc,
                const int *atm, const int *bas, const double *env)
{
    const int task_id = blockIdx.x;
    if (task_id >= ntasks){
        return;
    }

    const int ish = tasks[task_id];
    const int jsh = tasks[task_id + ntasks];
    const int ksh = tasks[task_id + 2*ntasks];
    const int ioff = ao_loc[ish];
    const int joff = ao_loc[jsh];
    const int ecp_id = ecpbas[ATOM_OF+ecploc[ksh]*BAS_SLOTS];
    gctr += 3*ecp_id*nao*nao + ioff*nao + joff;

    __shared__ double gctr_smem[NF_MAX*NF_MAX*3];
    for (int ij = threadIdx.x; ij < NF_MAX*NF_MAX*3; ij+=blockDim.x){
        gctr_smem[ij] = 0.0;
    }
    __syncthreads();

    constexpr int nfi_max = (AO_LMAX+2)*(AO_LMAX+3)/2;
    constexpr int nfj_max = (AO_LMAX+1)*(AO_LMAX+2)/2;
    __shared__ double buf[nfi_max*nfj_max];

    type1_cart_kernel<1,0>(buf, LI+1, LJ, ish, jsh, ksh, ecpbas, ecploc, atm, bas, env);
    __syncthreads();
    _li_down(gctr_smem, buf, LI, LJ);

    if (LI > 0){
        type1_cart_kernel<0,0>(buf, LI-1, LJ, ish, jsh, ksh, ecpbas, ecploc, atm, bas, env);
        __syncthreads();
        _li_up(gctr_smem, buf, LI, LJ);
    }

    const int nfi = (LI+1) * (LI+2) / 2;
    const int nfj = (LJ+1) * (LJ+2) / 2;
    for (int ij = threadIdx.x; ij < nfi*nfj; ij+=blockDim.x){
        const int i = ij%nfi;
        const int j = ij/nfi;
        double *gx = gctr;
        double *gy = gctr +   nao*nao;
        double *gz = gctr + 2*nao*nao;
        atomicAdd(gx+i*nao+j, gctr_smem[ij]);
        atomicAdd(gy+i*nao+j, gctr_smem[ij+nfi*nfj]);
        atomicAdd(gz+i*nao+j, gctr_smem[ij+2*nfi*nfj]);
    }
    return;
}

__global__
void type1_cart_ipipv(double *gctr,
                const int LI, const int LJ,
                const int *ao_loc, const int nao,
                const int *tasks, const int ntasks,
                const int *ecpbas, const int *ecploc,
                const int *atm, const int *bas, const double *env)
{
    const int task_id = blockIdx.x;
    if (task_id >= ntasks){
        return;
    }

    const int ish = tasks[task_id];
    const int jsh = tasks[task_id + ntasks];
    const int ksh = tasks[task_id + 2*ntasks];

    const int ioff = ao_loc[ish];
    const int joff = ao_loc[jsh];
    const int ecp_id = ecpbas[ATOM_OF+ecploc[ksh]*BAS_SLOTS];
    gctr += ioff*nao + joff + 9*ecp_id*nao*nao;

    constexpr int nfi2_max = (AO_LMAX+3)*(AO_LMAX+4)/2;
    constexpr int nfj_max = (AO_LMAX+1)*(AO_LMAX+2)/2;
    __shared__ double buf1[nfi2_max*nfj_max];
    type1_cart_kernel<2,0>(buf1, LI+2, LJ, ish, jsh, ksh, ecpbas, ecploc, atm, bas, env);
    __syncthreads();

    constexpr int nfi1_max = (AO_LMAX+2)*(AO_LMAX+3)/2;
    __shared__ double buf[3*nfi1_max*nfj_max];
    for (int i = threadIdx.x; i < 3*nfi1_max*nfj_max; i+=blockDim.x){
        buf[i] = 0.0;
    }
    __syncthreads();
    _li_down(buf, buf1, LI+1, LJ);

    type1_cart_kernel<1,0>(buf1, LI, LJ, ish, jsh, ksh, ecpbas, ecploc, atm, bas, env);
    __syncthreads();
    _li_up(buf, buf1, LI+1, LJ);
    _li_down_and_write(gctr, buf, LI, LJ, nao);

    if (LI > 0){
        for (int i = threadIdx.x; i < 3*nfi1_max*nfj_max; i+=blockDim.x){
            buf[i] = 0.0;
        }
        __syncthreads();
        _li_down(buf, buf1, LI-1, LJ);
        if (LI > 1){
            type1_cart_kernel<0,0>(buf1, LI-2, LJ, ish, jsh, ksh, ecpbas, ecploc, atm, bas, env);
            __syncthreads();
            _li_up(buf, buf1, LI-1, LJ);
        }
        _li_up_and_write(gctr, buf, LI, LJ, nao);
    }
    return;
}

__global__
void type1_cart_ipvip(double *gctr,
                const int LI, const int LJ,
                const int *ao_loc, const int nao,
                const int *tasks, const int ntasks,
                const int *ecpbas, const int *ecploc,
                const int *atm, const int *bas, const double *env)
{
    const int task_id = blockIdx.x;
    if (task_id >= ntasks){
        return;
    }

    const int ish = tasks[task_id];
    const int jsh = tasks[task_id + ntasks];
    const int ksh = tasks[task_id + 2*ntasks];

    const int ioff = ao_loc[ish];
    const int joff = ao_loc[jsh];
    const int ecp_id = ecpbas[ATOM_OF+ecploc[ksh]*BAS_SLOTS];
    gctr += ioff*nao + joff + 9*ecp_id*nao*nao;

    constexpr int nfi1_max = (AO_LMAX+2)*(AO_LMAX+3)/2;
    constexpr int nfj1_max = (AO_LMAX+2)*(AO_LMAX+3)/2;
    __shared__ double buf1[nfi1_max*nfj1_max];
    type1_cart_kernel<1,1>(buf1, LI+1, LJ+1, ish, jsh, ksh, ecpbas, ecploc, atm, bas, env);
    __syncthreads();

    constexpr int nfi_max = (AO_LMAX+1)*(AO_LMAX+2)/2;
    __shared__ double buf[3*nfi_max*nfj1_max];
    for (int i = threadIdx.x; i < 3*nfi_max*nfj1_max; i+=blockDim.x){
        buf[i] = 0.0;
    }
    __syncthreads();
    _li_down(buf, buf1, LI, LJ+1);
    if (LI > 0){
        type1_cart_kernel<0,1>(buf1, LI-1, LJ+1, ish, jsh, ksh, ecpbas, ecploc, atm, bas, env);
        __syncthreads();
        _li_up(buf, buf1, LI, LJ+1);
    }
    _lj_down_and_write(gctr, buf, LI, LJ, nao);

    if (LJ > 0){
        for (int i = threadIdx.x; i < 3*nfi_max*nfj1_max; i+=blockDim.x){
            buf[i] = 0.0;
        }
        __syncthreads();
        type1_cart_kernel<1,0>(buf1, LI+1, LJ-1, ish, jsh, ksh, ecpbas, ecploc, atm, bas, env);
        __syncthreads();
        _li_down(buf, buf1, LI, LJ-1);
        if (LI > 0){
            type1_cart_kernel<0,0>(buf1, LI-1, LJ-1, ish, jsh, ksh, ecpbas, ecploc, atm, bas, env);
            __syncthreads();
            _li_up(buf, buf1, LI, LJ-1);
        }
        _lj_up_and_write(gctr, buf, LI, LJ, nao);
    }
    return;
}
