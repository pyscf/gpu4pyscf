/*
 * Copyright 2021-2025 The PySCF Developers. All Rights Reserved.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include <stdio.h>
#include "ecp.h"
#include "bessel.cu"
#include "cart2sph.cu"
#include "gauss_chebyshev.cu"
#include "common.cu"
#include "ecp_type1.cu"
#include "ecp_type2.cu"

extern "C" {
int ECP_cart(double *gctr, 
            const int *ao_loc, const int nao, 
            const int *tasks, const int ntasks,
            const int *ecpbas, const int *ecploc, 
            const int *atm, const int *bas, const double *env, 
            int li, int lj, int lk){
    // one task per thread block
    dim3 threads(THREADS);
    dim3 blocks(ntasks);
    if (lk >= 0){
        int task_type = li * 100 + lj * 10 + lk;
        switch (task_type)
        {
        case 0:  type2_cart<0,0,0><<<blocks, threads>>>(gctr, ao_loc, nao, tasks, ntasks, ecpbas, ecploc, atm, bas, env); break;
        case 1:  type2_cart<0,0,1><<<blocks, threads>>>(gctr, ao_loc, nao, tasks, ntasks, ecpbas, ecploc, atm, bas, env); break;
        case 2:  type2_cart<0,0,2><<<blocks, threads>>>(gctr, ao_loc, nao, tasks, ntasks, ecpbas, ecploc, atm, bas, env); break;
        case 3:  type2_cart<0,0,3><<<blocks, threads>>>(gctr, ao_loc, nao, tasks, ntasks, ecpbas, ecploc, atm, bas, env); break;
        case 4:  type2_cart<0,0,4><<<blocks, threads>>>(gctr, ao_loc, nao, tasks, ntasks, ecpbas, ecploc, atm, bas, env); break;

        case 10:  type2_cart<0,1,0><<<blocks, threads>>>(gctr, ao_loc, nao, tasks, ntasks, ecpbas, ecploc, atm, bas, env); break;
        case 11:  type2_cart<0,1,1><<<blocks, threads>>>(gctr, ao_loc, nao, tasks, ntasks, ecpbas, ecploc, atm, bas, env); break;
        case 12:  type2_cart<0,1,2><<<blocks, threads>>>(gctr, ao_loc, nao, tasks, ntasks, ecpbas, ecploc, atm, bas, env); break;
        case 13:  type2_cart<0,1,3><<<blocks, threads>>>(gctr, ao_loc, nao, tasks, ntasks, ecpbas, ecploc, atm, bas, env); break;
        case 14:  type2_cart<0,1,4><<<blocks, threads>>>(gctr, ao_loc, nao, tasks, ntasks, ecpbas, ecploc, atm, bas, env); break;

        case 110: type2_cart<1,1,0><<<blocks, threads>>>(gctr, ao_loc, nao, tasks, ntasks, ecpbas, ecploc, atm, bas, env); break;
        case 111: type2_cart<1,1,1><<<blocks, threads>>>(gctr, ao_loc, nao, tasks, ntasks, ecpbas, ecploc, atm, bas, env); break;
        case 112: type2_cart<1,1,2><<<blocks, threads>>>(gctr, ao_loc, nao, tasks, ntasks, ecpbas, ecploc, atm, bas, env); break;
        case 113: type2_cart<1,1,3><<<blocks, threads>>>(gctr, ao_loc, nao, tasks, ntasks, ecpbas, ecploc, atm, bas, env); break;
        case 114: type2_cart<1,1,4><<<blocks, threads>>>(gctr, ao_loc, nao, tasks, ntasks, ecpbas, ecploc, atm, bas, env); break;
        
        case 20:  type2_cart<0,2,0><<<blocks, threads>>>(gctr, ao_loc, nao, tasks, ntasks, ecpbas, ecploc, atm, bas, env); break;
        case 21:  type2_cart<0,2,1><<<blocks, threads>>>(gctr, ao_loc, nao, tasks, ntasks, ecpbas, ecploc, atm, bas, env); break;
        case 22:  type2_cart<0,2,2><<<blocks, threads>>>(gctr, ao_loc, nao, tasks, ntasks, ecpbas, ecploc, atm, bas, env); break;
        case 23:  type2_cart<0,2,3><<<blocks, threads>>>(gctr, ao_loc, nao, tasks, ntasks, ecpbas, ecploc, atm, bas, env); break;
        case 24:  type2_cart<0,2,4><<<blocks, threads>>>(gctr, ao_loc, nao, tasks, ntasks, ecpbas, ecploc, atm, bas, env); break;

        case 30:  type2_cart<0,3,0><<<blocks, threads>>>(gctr, ao_loc, nao, tasks, ntasks, ecpbas, ecploc, atm, bas, env); break;
        case 31:  type2_cart<0,3,1><<<blocks, threads>>>(gctr, ao_loc, nao, tasks, ntasks, ecpbas, ecploc, atm, bas, env); break;
        case 32:  type2_cart<0,3,2><<<blocks, threads>>>(gctr, ao_loc, nao, tasks, ntasks, ecpbas, ecploc, atm, bas, env); break;
        case 33:  type2_cart<0,3,3><<<blocks, threads>>>(gctr, ao_loc, nao, tasks, ntasks, ecpbas, ecploc, atm, bas, env); break;
        case 34:  type2_cart<0,3,4><<<blocks, threads>>>(gctr, ao_loc, nao, tasks, ntasks, ecpbas, ecploc, atm, bas, env); break;
        
        case 120: type2_cart<1,2,0><<<blocks, threads>>>(gctr, ao_loc, nao, tasks, ntasks, ecpbas, ecploc, atm, bas, env); break;
        case 121: type2_cart<1,2,1><<<blocks, threads>>>(gctr, ao_loc, nao, tasks, ntasks, ecpbas, ecploc, atm, bas, env); break;
        case 122: type2_cart<1,2,2><<<blocks, threads>>>(gctr, ao_loc, nao, tasks, ntasks, ecpbas, ecploc, atm, bas, env); break;
        case 123: type2_cart<1,2,3><<<blocks, threads>>>(gctr, ao_loc, nao, tasks, ntasks, ecpbas, ecploc, atm, bas, env); break;
        case 124: type2_cart<1,2,4><<<blocks, threads>>>(gctr, ao_loc, nao, tasks, ntasks, ecpbas, ecploc, atm, bas, env); break;

        case 40:  type2_cart<0,4,0><<<blocks, threads>>>(gctr, ao_loc, nao, tasks, ntasks, ecpbas, ecploc, atm, bas, env); break;
        case 41:  type2_cart<0,4,1><<<blocks, threads>>>(gctr, ao_loc, nao, tasks, ntasks, ecpbas, ecploc, atm, bas, env); break;
        case 42:  type2_cart<0,4,2><<<blocks, threads>>>(gctr, ao_loc, nao, tasks, ntasks, ecpbas, ecploc, atm, bas, env); break;
        case 43:  type2_cart<0,4,3><<<blocks, threads>>>(gctr, ao_loc, nao, tasks, ntasks, ecpbas, ecploc, atm, bas, env); break;
        case 44:  type2_cart<0,4,4><<<blocks, threads>>>(gctr, ao_loc, nao, tasks, ntasks, ecpbas, ecploc, atm, bas, env); break;
           
        case 130: type2_cart<1,3,0><<<blocks, threads>>>(gctr, ao_loc, nao, tasks, ntasks, ecpbas, ecploc, atm, bas, env); break;
        case 131: type2_cart<1,3,1><<<blocks, threads>>>(gctr, ao_loc, nao, tasks, ntasks, ecpbas, ecploc, atm, bas, env); break;
        case 132: type2_cart<1,3,2><<<blocks, threads>>>(gctr, ao_loc, nao, tasks, ntasks, ecpbas, ecploc, atm, bas, env); break;
        case 133: type2_cart<1,3,3><<<blocks, threads>>>(gctr, ao_loc, nao, tasks, ntasks, ecpbas, ecploc, atm, bas, env); break;
        case 134: type2_cart<1,3,4><<<blocks, threads>>>(gctr, ao_loc, nao, tasks, ntasks, ecpbas, ecploc, atm, bas, env); break;
        
        case 220: type2_cart<2,2,0><<<blocks, threads>>>(gctr, ao_loc, nao, tasks, ntasks, ecpbas, ecploc, atm, bas, env); break;
        case 221: type2_cart<2,2,1><<<blocks, threads>>>(gctr, ao_loc, nao, tasks, ntasks, ecpbas, ecploc, atm, bas, env); break;
        case 222: type2_cart<2,2,2><<<blocks, threads>>>(gctr, ao_loc, nao, tasks, ntasks, ecpbas, ecploc, atm, bas, env); break;
        case 223: type2_cart<2,2,3><<<blocks, threads>>>(gctr, ao_loc, nao, tasks, ntasks, ecpbas, ecploc, atm, bas, env); break;
        case 224: type2_cart<2,2,4><<<blocks, threads>>>(gctr, ao_loc, nao, tasks, ntasks, ecpbas, ecploc, atm, bas, env); break;

        case 140: type2_cart<1,4,0><<<blocks, threads>>>(gctr, ao_loc, nao, tasks, ntasks, ecpbas, ecploc, atm, bas, env); break;
        case 141: type2_cart<1,4,1><<<blocks, threads>>>(gctr, ao_loc, nao, tasks, ntasks, ecpbas, ecploc, atm, bas, env); break;
        case 142: type2_cart<1,4,2><<<blocks, threads>>>(gctr, ao_loc, nao, tasks, ntasks, ecpbas, ecploc, atm, bas, env); break;
        case 143: type2_cart<1,4,3><<<blocks, threads>>>(gctr, ao_loc, nao, tasks, ntasks, ecpbas, ecploc, atm, bas, env); break;
        case 144: type2_cart<1,4,4><<<blocks, threads>>>(gctr, ao_loc, nao, tasks, ntasks, ecpbas, ecploc, atm, bas, env); break;
        
        case 230: type2_cart<2,3,0><<<blocks, threads>>>(gctr, ao_loc, nao, tasks, ntasks, ecpbas, ecploc, atm, bas, env); break;
        case 231: type2_cart<2,3,1><<<blocks, threads>>>(gctr, ao_loc, nao, tasks, ntasks, ecpbas, ecploc, atm, bas, env); break;
        case 232: type2_cart<2,3,2><<<blocks, threads>>>(gctr, ao_loc, nao, tasks, ntasks, ecpbas, ecploc, atm, bas, env); break;
        case 233: type2_cart<2,3,3><<<blocks, threads>>>(gctr, ao_loc, nao, tasks, ntasks, ecpbas, ecploc, atm, bas, env); break;
        case 234: type2_cart<2,3,4><<<blocks, threads>>>(gctr, ao_loc, nao, tasks, ntasks, ecpbas, ecploc, atm, bas, env); break;

        case 240: type2_cart<2,4,0><<<blocks, threads>>>(gctr, ao_loc, nao, tasks, ntasks, ecpbas, ecploc, atm, bas, env); break;
        case 241: type2_cart<2,4,1><<<blocks, threads>>>(gctr, ao_loc, nao, tasks, ntasks, ecpbas, ecploc, atm, bas, env); break;
        case 242: type2_cart<2,4,2><<<blocks, threads>>>(gctr, ao_loc, nao, tasks, ntasks, ecpbas, ecploc, atm, bas, env); break;
        case 243: type2_cart<2,4,3><<<blocks, threads>>>(gctr, ao_loc, nao, tasks, ntasks, ecpbas, ecploc, atm, bas, env); break;
        case 244: type2_cart<2,4,4><<<blocks, threads>>>(gctr, ao_loc, nao, tasks, ntasks, ecpbas, ecploc, atm, bas, env); break;
        
        case 330: type2_cart<3,3,0><<<blocks, threads>>>(gctr, ao_loc, nao, tasks, ntasks, ecpbas, ecploc, atm, bas, env); break;
        case 331: type2_cart<3,3,1><<<blocks, threads>>>(gctr, ao_loc, nao, tasks, ntasks, ecpbas, ecploc, atm, bas, env); break;
        case 332: type2_cart<3,3,2><<<blocks, threads>>>(gctr, ao_loc, nao, tasks, ntasks, ecpbas, ecploc, atm, bas, env); break;
        case 333: type2_cart<3,3,3><<<blocks, threads>>>(gctr, ao_loc, nao, tasks, ntasks, ecpbas, ecploc, atm, bas, env); break;
        case 334: type2_cart<3,3,4><<<blocks, threads>>>(gctr, ao_loc, nao, tasks, ntasks, ecpbas, ecploc, atm, bas, env); break;

        case 340: type2_cart<3,4,0><<<blocks, threads>>>(gctr, ao_loc, nao, tasks, ntasks, ecpbas, ecploc, atm, bas, env); break;
        case 341: type2_cart<3,4,1><<<blocks, threads>>>(gctr, ao_loc, nao, tasks, ntasks, ecpbas, ecploc, atm, bas, env); break;
        case 342: type2_cart<3,4,2><<<blocks, threads>>>(gctr, ao_loc, nao, tasks, ntasks, ecpbas, ecploc, atm, bas, env); break;
        case 343: type2_cart<3,4,3><<<blocks, threads>>>(gctr, ao_loc, nao, tasks, ntasks, ecpbas, ecploc, atm, bas, env); break;
        case 344: type2_cart<3,4,4><<<blocks, threads>>>(gctr, ao_loc, nao, tasks, ntasks, ecpbas, ecploc, atm, bas, env); break;

        case 440: type2_cart<4,4,0><<<blocks, threads>>>(gctr, ao_loc, nao, tasks, ntasks, ecpbas, ecploc, atm, bas, env); break;
        case 441: type2_cart<4,4,1><<<blocks, threads>>>(gctr, ao_loc, nao, tasks, ntasks, ecpbas, ecploc, atm, bas, env); break;
        case 442: type2_cart<4,4,2><<<blocks, threads>>>(gctr, ao_loc, nao, tasks, ntasks, ecpbas, ecploc, atm, bas, env); break;
        case 443: type2_cart<4,4,3><<<blocks, threads>>>(gctr, ao_loc, nao, tasks, ntasks, ecpbas, ecploc, atm, bas, env); break;
        case 444: type2_cart<4,4,4><<<blocks, threads>>>(gctr, ao_loc, nao, tasks, ntasks, ecpbas, ecploc, atm, bas, env); break;

        default: fprintf(stderr, "(%d,%d,%d) is not supported in ECP.\n", li, lj, lk); break;
        }
    } else {
        int task_type = li * 10 + lj;
        switch (task_type)
        {
        case 0:  type1_cart<0,0><<<blocks, threads>>>(gctr, ao_loc, nao, tasks, ntasks, ecpbas, ecploc, atm, bas, env); break;

        case 1:  type1_cart<0,1><<<blocks, threads>>>(gctr, ao_loc, nao, tasks, ntasks, ecpbas, ecploc, atm, bas, env); break;

        case 11: type1_cart<1,1><<<blocks, threads>>>(gctr, ao_loc, nao, tasks, ntasks, ecpbas, ecploc, atm, bas, env); break;
        case 2:  type1_cart<0,2><<<blocks, threads>>>(gctr, ao_loc, nao, tasks, ntasks, ecpbas, ecploc, atm, bas, env); break;

        case 3:  type1_cart<0,3><<<blocks, threads>>>(gctr, ao_loc, nao, tasks, ntasks, ecpbas, ecploc, atm, bas, env); break;
        case 12: type1_cart<1,2><<<blocks, threads>>>(gctr, ao_loc, nao, tasks, ntasks, ecpbas, ecploc, atm, bas, env); break;

        case 4:  type1_cart<0,4><<<blocks, threads>>>(gctr, ao_loc, nao, tasks, ntasks, ecpbas, ecploc, atm, bas, env); break;
        case 13: type1_cart<1,3><<<blocks, threads>>>(gctr, ao_loc, nao, tasks, ntasks, ecpbas, ecploc, atm, bas, env); break;
        case 22: type1_cart<2,2><<<blocks, threads>>>(gctr, ao_loc, nao, tasks, ntasks, ecpbas, ecploc, atm, bas, env); break;

        case 14: type1_cart<1,4><<<blocks, threads>>>(gctr, ao_loc, nao, tasks, ntasks, ecpbas, ecploc, atm, bas, env); break;
        case 23: type1_cart<2,3><<<blocks, threads>>>(gctr, ao_loc, nao, tasks, ntasks, ecpbas, ecploc, atm, bas, env); break;

        case 24: type1_cart<2,4><<<blocks, threads>>>(gctr, ao_loc, nao, tasks, ntasks, ecpbas, ecploc, atm, bas, env); break;
        case 33: type1_cart<3,3><<<blocks, threads>>>(gctr, ao_loc, nao, tasks, ntasks, ecpbas, ecploc, atm, bas, env); break;

        case 34: type1_cart<3,4><<<blocks, threads>>>(gctr, ao_loc, nao, tasks, ntasks, ecpbas, ecploc, atm, bas, env); break;

        case 44: type1_cart<4,4><<<blocks, threads>>>(gctr, ao_loc, nao, tasks, ntasks, ecpbas, ecploc, atm, bas, env); break;

        default: fprintf(stderr, "(%d,%d) is not supported in ECP.\n", li, lj); break;
        }
    }

    hipError_t err = hipGetLastError();
    if (err != hipSuccess) {
        fprintf(stderr, "CUDA Error in %s: %s\n", __func__, hipGetErrorString(err));
        return 1;
    }
    return 0;
    }
}
