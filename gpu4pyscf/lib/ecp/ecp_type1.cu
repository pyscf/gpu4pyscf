#include "hip/hip_runtime.h"
/*
 * Copyright 2021-2025 The PySCF Developers. All Rights Reserved.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

template <int LIJ> __device__
void type1_rad_part(double* __restrict__ rad_all, double k, double aij, double ur)
{
    constexpr int LIJ1 = LIJ + 1;
    const double kaij = k / (2*aij);
    const double fac = kaij * kaij * aij;

    double tmp = r128[threadIdx.x] - kaij;
    tmp = fac - aij*tmp*tmp;
    double bval[LIJ1];
    double rur;
    if (ur == 0 || tmp > CUTOFF || tmp < -(EXPCUTOFF+6.+30.)) {
        rur = 0;
        for (int i = 0; i < LIJ1; i++){
            bval[i] = 0;
        }
    } else {
        rur = ur * exp(tmp);
        _ine<LIJ>(bval, k*r128[threadIdx.x]);
    }

    for (int i = threadIdx.x; i <= LIJ1*LIJ1; i+=blockDim.x){
        rad_all[i] = 0.0;
    }
    __syncthreads();
    for (int lab = 0; lab <= LIJ; lab++){
        if (lab > 0){
            rur *= r128[threadIdx.x];
        }
        for (int i = lab%2; i <= LIJ; i+=2){
            //atomicAdd(rad_all+lab*LIJ1+i, rur*bval[i]);
            //rad_all[lab*LIJ1+i] = rur * bval[i];
            block_reduce(rur*bval[i], rad_all+lab*LIJ1+i);
        }
    }
    __syncthreads();
}

template <int LIJ> __device__
void type1_rad_ang(double *rad_ang, double *r, double *rad_all, double fac)
{
    double unitr[3];
    if (r[0]*r[0] + r[1]*r[1] + r[2]*r[2] < 1e-16){
        unitr[0] = 0;
        unitr[1] = 0;
        unitr[2] = 0;
    } else {
        double norm_r = -rnorm3d(r[0], r[1], r[2]);
        unitr[0] = r[0] * norm_r;
        unitr[1] = r[1] * norm_r;
        unitr[2] = r[2] * norm_r;
    }

    double omega_nuc[CART_CUM];
    ang_nuc_part<LIJ>(omega_nuc, unitr[0], unitr[1], unitr[2]);

    constexpr int LIJ1 = LIJ + 1;
    constexpr int LIJ2 = LIJ1 * LIJ1;

    // loop over i+j+k<=LIJ
    // TODO: find a closed form?
    for (int n = threadIdx.x; n < LIJ1*LIJ1*LIJ1; n+=blockDim.x){
        int i = n/LIJ1/LIJ1;
        int j = n/LIJ1%LIJ1;
        int k = n%LIJ1;
        if (i+j+k > LIJ){
            continue;
        }
        double *pout = rad_ang + i*LIJ2+j*LIJ1+k;
        double *prad = rad_all + (i+j+k)*LIJ1;
        // need_even to ensure (a+b+c+lmb) is even
        const int need_even = (i+j+k)%2;
        for (int lmb = need_even; lmb <= LIJ; lmb+=2) {
            double tmp = 0;
            double *pnuc = omega_nuc + _offset_cart[lmb];
            for (int n = 0; n < (lmb+1)*(lmb+2)/2; n++){
                const int ps = _cart_pow_y[n];
                const int pt = _cart_pow_z[n];
                const int pr = lmb - ps - pt;
                if ((i+pr)%2 || (j+ps)%2 || (k+pt)%2){
                    continue;
                }
                tmp += pnuc[n] * int_unit_xyz(i+pr, j+ps, k+pt);
            }
            //*pout += fac * prad[lmb] * tmp;
            atomicAdd(pout, fac*prad[lmb]*tmp);
        }
    }
}


template <int LI> __device__
void type1_cache_fac(double* __restrict__ ifac, double *ri){
    constexpr int LI1 = LI + 1;
    constexpr int nfi = (LI1+1)*LI1/2;
    double fx[nfi*3];
    cache_fac<LI>(fx, ri);

    double *fy = fx + nfi;
    double *fz = fy + nfi;
    constexpr int LI2 = LI1 * LI1;
    constexpr int LI3 = LI2 * LI1;
    for (int mi = threadIdx.x; mi < nfi; mi+=blockDim.x){
        int iy = _cart_pow_y[mi];
        int iz = _cart_pow_z[mi];
        int ix = LI - iy - iz;
        for (int i1 = 0; i1 <= ix; i1++){
        for (int i2 = 0; i2 <= iy; i2++){
        for (int i3 = 0; i3 <= iz; i3++){
            const int idx = mi*LI3 + i1*LI2 + i2*LI1 + i3;
            const int xoffset = (ix+1)*ix/2;
            const int yoffset = (iy+1)*iy/2;
            const int zoffset = (iz+1)*iz/2;
            ifac[idx] = fx[xoffset+i1] * fy[yoffset+i2] * fz[zoffset+i3];
        }}}
    }
}

template <int LI, int LJ> __global__
void type1_cart(double *gctr, 
                const int *ao_loc, const int nao, 
                const int *tasks, const int ntasks,
                const int *ecpbas, const int *ecploc, 
                const int *atm, const int *bas, const double *env)
{
    const int task_id = blockIdx.x;
    if (task_id >= ntasks){
        return;
    }

    const int ish = tasks[task_id];
    const int jsh = tasks[task_id + ntasks];
    const int ksh = tasks[task_id + 2*ntasks];

    const int npi = bas[NPRIM_OF+ish*BAS_SLOTS];
    const int npj = bas[NPRIM_OF+jsh*BAS_SLOTS];
    constexpr int nfi = (LI+1) * (LI+2) / 2;
    constexpr int nfj = (LJ+1) * (LJ+2) / 2;
    const double *ai = env + bas[PTR_EXP+ish*BAS_SLOTS];
    const double *aj = env + bas[PTR_EXP+jsh*BAS_SLOTS];
    const double *ci = env + bas[PTR_COEFF+ish*BAS_SLOTS];
    const double *cj = env + bas[PTR_COEFF+jsh*BAS_SLOTS];
    const double *ri = env + atm[PTR_COORD+bas[ATOM_OF+ish*BAS_SLOTS]*ATM_SLOTS];
    const double *rj = env + atm[PTR_COORD+bas[ATOM_OF+jsh*BAS_SLOTS]*ATM_SLOTS];

    const int atm_id = ecpbas[ATOM_OF+ecploc[ksh]*BAS_SLOTS];
    const double *rc = env + atm[PTR_COORD+atm_id*ATM_SLOTS];

    double rca[3], rcb[3];
    rca[0] = rc[0] - ri[0];
    rca[1] = rc[1] - ri[1];
    rca[2] = rc[2] - ri[2];
    rcb[0] = rc[0] - rj[0];
    rcb[1] = rc[1] - rj[1];
    rcb[2] = rc[2] - rj[2];
    const double r2ca = rca[0]*rca[0] + rca[1]*rca[1] + rca[2]*rca[2];
    const double r2cb = rcb[0]*rcb[0] + rcb[1]*rcb[1] + rcb[2]*rcb[2];

    double ur = 0.0;
    for (int kbas = ecploc[ksh]; kbas < ecploc[ksh+1]; kbas++){
        ur += rad_part(kbas, ecpbas, env);
    }
    
    constexpr int LIJ1 = LI+LJ+1;
    constexpr int LIJ3 = LIJ1*LIJ1*LIJ1;
    __shared__ double rad_ang[LIJ3]; // up to 5832 Bytes
    for (int i = threadIdx.x; i < LIJ3; i+=blockDim.x) {
        rad_ang[i] = 0;
    }
    __syncthreads();

    const double fac = 16.0 * M_PI * M_PI * _common_fac[LI] * _common_fac[LJ];
    for (int ip = 0; ip < npi; ip++){
        for (int jp = 0; jp < npj; jp++){
            double rij[3];
            rij[0] = ai[ip] * rca[0] + aj[jp] * rcb[0];
            rij[1] = ai[ip] * rca[1] + aj[jp] * rcb[1];
            rij[2] = ai[ip] * rca[2] + aj[jp] * rcb[2];
            const double k = 2.0 * norm3d(rij[0], rij[1], rij[2]);
            const double aij = ai[ip] + aj[jp];

            __shared__ double rad_all[LIJ1*LIJ1];
            type1_rad_part<LI+LJ>(rad_all, k, aij, ur);

            const double eij = exp(-ai[ip]*r2ca - aj[jp]*r2cb);
            const double ceij = eij * ci[ip] * cj[jp];
            type1_rad_ang<LI+LJ>(rad_ang, rij, rad_all, fac*ceij);
            __syncthreads();
        }
    }

    constexpr int LI1 = LI+1;
    constexpr int LJ1 = LJ+1;
    constexpr int LI2 = LI1*LI1;
    constexpr int LJ2 = LJ1*LJ1;
    constexpr int LI3 = LI1*LI2;
    constexpr int LJ3 = LJ1*LJ2;
    __shared__ double ifac[nfi*LI3]; // up to 15625 Bytes
    __shared__ double jfac[nfj*LJ3];

    type1_cache_fac<LJ>(jfac, rcb);
    type1_cache_fac<LI>(ifac, rca);
    __syncthreads();

    const int ioff = ao_loc[ish];
    const int joff = ao_loc[jsh];
    
    // TODO: unrolling with a code generator
    for (int ij = threadIdx.x; ij < nfi*nfj; ij+=blockDim.x){
        const int mi = ij%nfi;
        const int mj = ij/nfi;

        const int iy = _cart_pow_y[mi];
        const int iz = _cart_pow_z[mi];
        const int ix = LI - iy - iz;

        const int jy = _cart_pow_y[mj];
        const int jz = _cart_pow_z[mj];
        const int jx = LJ - jy - jz;

        // cache ifac and jfac in register
        double tmp = 0.0;
        for (int i1 = 0; i1 <= ix; i1++){
        for (int i2 = 0; i2 <= iy; i2++){
        for (int i3 = 0; i3 <= iz; i3++){
            for (int j1 = 0; j1 <= jx; j1++){
            for (int j2 = 0; j2 <= jy; j2++){
            for (int j3 = 0; j3 <= jz; j3++){
                const int ir = mi * LI3 + i1 * LI2 + i2 * LI1 + i3;
                const int jr = mj * LJ3 + j1 * LJ2 + j2 * LJ1 + j3;
                const int ijr = (i1+j1)*LIJ1*LIJ1 + (i2+j2)*LIJ1 + (i3+j3);
                tmp += ifac[ir] * jfac[jr] * rad_ang[ijr];
            }}}
        }}}
        atomicAdd(gctr + mi+ioff + (mj+joff)*nao, tmp);
        if (ish != jsh){
            atomicAdd(gctr + (mi+ioff)*nao + mj+joff, tmp);
        }
    }
    return;
}
