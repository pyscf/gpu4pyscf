#include "hip/hip_runtime.h"
/*
 * Copyright 2021-2025 The PySCF Developers. All Rights Reserved.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include "ecp.h"
#include "bessel.cu"

__device__
static double r99[] = {
    7.49149775547408580678e-09,2.39390171574704879731e-07,1.81360303841415770876e-06,
    7.61741913646307722274e-06,2.31485030718348028245e-05,5.73044613240147882038e-05,
    1.23105600339346032968e-04,2.38337096238994128328e-04,4.26098829770071851897e-04,
    7.15253566223816861225e-04,1.14076690801112601292e-03,1.74393542073036922346e-03,
    2.57250241525242007157e-03,3.68066393021282411979e-03,5.12897037092585605933e-03,
    6.98413189926139210684e-03,9.31873792827664360061e-03,1.22109028731325341965e-02,
    1.57438515926779931675e-02,2.00054587018748675220e-02,2.50877561523923375830e-02,
    3.10864232033296605806e-02,3.81002721950566280995e-02,4.62307424720755921754e-02,
    5.55814134610747023757e-02,6.62575463867129954565e-02,7.83656624864419448784e-02,
    9.20131639458153793854e-02,1.07308002185481865531e-01,1.24358396645808944037e-01,
    1.43272605876474390385e-01,1.64158751574681183172e-01,1.87124695242669281114e-01,
    2.12277966358133562963e-01,2.39725740366322215280e-01,2.69574864399791169767e-01,
    3.01931928396128146375e-01,3.36903379197803665157e-01,3.74595675262858196497e-01,
    4.15115479771713591362e-01,4.58569890166489968486e-01,5.05066702489160856970e-01,
    5.54714709280482720644e-01,6.07624030252335245450e-01,6.63906475444308141753e-01,
    7.23675941116369569883e-01,7.87048839211608508570e-01,8.54144561847640537700e-01,
    9.25085982966693309848e-01,1.00000000000000000000e+00,1.07901811919271173323e+00,
    1.16227708910521165819e+00,1.24991958777366174438e+00,1.34209497009947353874e+00,
    1.43896008327213209554e+00,1.54068015944685710039e+00,1.64742979654017807079e+00,
    1.75939403992477760852e+00,1.87676958006419325464e+00,1.99976608380819209643e+00,
    2.12860768027057822849e+00,2.26353462605831046162e+00,2.40480517927247383625e+00,
    2.55269771735934725143e+00,2.70751314081141725154e+00,2.86957761323600735182e+00,
    3.03924569885439943562e+00,3.21690397163072372422e+00,3.40297518669977883121e+00,
    3.59792312555240823002e+00,3.80225825286600205288e+00,4.01654435671977161348e+00,
    4.24140638764222543955e+00,4.47753976885303650590e+00,4.72572152485034457925e+00,
    4.98682367473171250793e+00,5.26182946969805254156e+00,5.55185323462325097665e+00,
    5.85816482123891901779e+00,6.18222002494881728296e+00,6.52569880330852480910e+00,
    6.89055383078500405247e+00,7.27907294007241034706e+00,7.69396050945778942065e+00,
    8.13844514586779865795e+00,8.61642457049883248033e+00,9.13266428605907520932e+00,
    9.69307592048363630965e+00,1.03051169664600053011e+01,1.09783815787195599967e+01,
    1.17255037073349104304e+01,1.25635944005437458770e+01,1.35166439102318687304e+01,
    1.46197880380327820404e+01,1.59274995188788217604e+01,1.75310364710511947806e+01,
    1.96014771359075226087e+01,2.25228992353034804808e+01,2.75208650628360445012e+01
};

__device__
static double w99[] = {
    3.74504465546597569538e-08,5.98025425874844469710e-07,3.01755904037220807623e-06,
    9.49315435270622856210e-06,2.30398689788116110507e-05,4.74313676460523959283e-05,
    8.71259201682535134800e-05,1.47179566351102945714e-04,2.33148687103363678368e-04,
    3.50984582880628406387e-04,5.06922938043571566502e-04,7.07371219265206943941e-04,
    9.58797101661354954881e-04,1.26762093081784132584e-03,1.64011501514828858875e-03,
    2.08231221338544105273e-03,2.59992585711949202726e-03,3.19828255500335511402e-03,
    3.88226889403487296012e-03,4.65629251553667869445e-03,5.52425752862067010601e-03,
    6.48955375754613900119e-03,7.55505892130658475436e-03,8.72315252730691305383e-03,
    9.99574003267965187358e-03,1.13742856869007685078e-02,1.28598524127975146619e-02,
    1.44531471005464283441e-02,1.61545697688108974566e-02,1.79642651752900291140e-02,
    1.98821756219053555337e-02,2.19080938846085149230e-02,2.40417153927295589033e-02,
    2.62826889781383951639e-02,2.86306657025656371984e-02,3.10853454466156094160e-02,
    3.36465211026480970347e-02,3.63141203538897561209e-02,3.90882451433852487477e-02,
    4.19692090394069017290e-02,4.49575727902660482460e-02,4.80541784332910032473e-02,
    5.12601823826394700778e-02,5.45770879714215714773e-02,5.80067779682611125991e-02,
    6.15515476299037800345e-02,6.52141388927818116406e-02,6.89977763505640606656e-02,
    7.29062057147122910550e-02,7.69437355140780432361e-02,8.11152828609839615659e-02,
    8.54264241987284650426e-02,8.98834520532275083049e-02,9.44934389444379158052e-02,
    9.92643097771207100211e-02,1.04204924232482093460e-01,1.09325170931016332765e-01,
    1.14636075443408688712e-01,1.20149924604410915374e-01,1.25880410051722568809e-01,
    1.31842794490549808373e-01,1.38054104903617747002e-01,1.44533357823557057076e-01,
    1.51301822908832206416e-01,1.58383332480235811124e-01,1.65804646467863830983e-01,
    1.73595884502318081877e-01,1.81791039811778243340e-01,1.90428593366022247402e-01,
    1.99552251623036053241e-01,2.09211837674749323579e-01,2.19464374100552722657e-01,
    2.30375407187545538923e-01,2.42020637456367176954e-01,2.54487942212678397436e-01,
    2.67879904418947323297e-01,2.82317001937544220791e-01,2.97941667216647110283e-01,
    3.14923507520003143068e-01,3.33466091844559053836e-01,3.53815881680646926455e-01,
    3.76274139285664444010e-01,4.01213039460866560670e-01,4.29097823819136747758e-01,
    4.60517817328173983960e-01,4.96230738223049638869e-01,5.37227459212045510561e-01,
    5.84829149853474561382e-01,6.40837406259777031536e-01,7.07774459213704520977e-01,
    7.89283521639834395600e-01,8.90829397324601646169e-01,1.02099961208170930682e+00,
    1.19410667684690441348e+00,1.43591306733334556078e+00,1.79794367421340983704e+00,
    2.40042315475528500457e+00,3.60402534195157464580e+00,7.21211908055371253568e+00
};

#define NGAUSS  99

__device__
static int _cart_pow_y[] = {
        0, 1, 0, 2, 1, 0, 3, 2, 1, 0, 4, 3, 2, 1, 0, 5, 4, 3, 2, 1,
        0, 6, 5, 4, 3, 2, 1, 0, 7, 6, 5, 4, 3, 2, 1, 0, 8, 7, 6, 5,
        4, 3, 2, 1, 0, 9, 8, 7, 6, 5, 4, 3, 2, 1, 0,10, 9, 8, 7, 6,
        5, 4, 3, 2, 1, 0,11,10, 9, 8, 7, 6, 5, 4, 3, 2, 1, 0,12,11,
       10, 9, 8, 7, 6, 5, 4, 3, 2, 1, 0,13,12,11,10, 9, 8, 7, 6, 5,
        4, 3, 2, 1, 0,14,13,12,11,10, 9, 8, 7, 6, 5, 4, 3, 2, 1, 0,
};

__device__
static int _cart_pow_z[] = {
        0, 0, 1, 0, 1, 2, 0, 1, 2, 3, 0, 1, 2, 3, 4, 0, 1, 2, 3, 4,
        5, 0, 1, 2, 3, 4, 5, 6, 0, 1, 2, 3, 4, 5, 6, 7, 0, 1, 2, 3,
        4, 5, 6, 7, 8, 0, 1, 2, 3, 4, 5, 6, 7, 8, 9, 0, 1, 2, 3, 4,
        5, 6, 7, 8, 9,10, 0, 1, 2, 3, 4, 5, 6, 7, 8, 9,10,11, 0, 1,
        2, 3, 4, 5, 6, 7, 8, 9,10,11,12, 0, 1, 2, 3, 4, 5, 6, 7, 8,
        9,10,11,12,13, 0, 1, 2, 3, 4, 5, 6, 7, 8, 9,10,11,12,13,14,
};

template<int L> __device__
static void ang_nuc_part(double *omega, double rx, double ry, double rz){
    if (L == 0){
        omega[0] = 0.282094791773878143;
    } else if (L == 1){
        omega[0] = 0.488602511902919921 * rx;
        omega[1] = 0.488602511902919921 * ry;
        omega[2] = 0.488602511902919921 * rz;
    } else if (L == 2){
        double g0 = rx * rx;
        double g1 = rx * ry;
        double g2 = rx * rz;
        double g3 = ry * ry;
        double g4 = ry * rz;
        double g5 = rz * rz;
        omega[0] = 1.092548430592079070 * g1;
        omega[1] = 1.092548430592079070 * g4;
        omega[2] = 0.630783130505040012 * g5 - 0.315391565252520002 * (g0 + g3);
        omega[3] = 1.092548430592079070 * g2;
        omega[4] = 0.546274215296039535 * (g0 - g3);
    } else if (L == 3){
        double g0 = rx * rx * rx;
        double g1 = rx * rx * ry;
        double g2 = rx * rx * rz;
        double g3 = rx * ry * ry;
        double g4 = rx * ry * rz;
        double g5 = rx * rz * rz;
        double g6 = ry * ry * ry;
        double g7 = ry * ry * rz;
        double g8 = ry * rz * rz;
        double g9 = rz * rz * rz;
        omega[0] = 1.770130769779930531 * g1 - 0.590043589926643510 * g6;
        omega[1] = 2.890611442640554055 * g4;
        omega[2] = 1.828183197857862944 * g8 - 0.457045799464465739 * (g1 + g6);
        omega[3] = 0.746352665180230782 * g9 - 1.119528997770346170 * (g2 + g7);
        omega[4] = 1.828183197857862944 * g5 - 0.457045799464465739 * (g0 + g3);
        omega[5] = 1.445305721320277020 * (g2 - g7);
        omega[6] = 0.590043589926643510 * g0 - 1.770130769779930530 * g3;
    } else if (L == 4){
        double g0  = rx * rx * rx * rx;
        double g1  = rx * rx * rx * ry;
        double g2  = rx * rx * rx * rz;
        double g3  = rx * rx * ry * ry;
        double g4  = rx * rx * ry * rz;
        double g5  = rx * rx * rz * rz;
        double g6  = rx * ry * ry * ry;
        double g7  = rx * ry * ry * rz;
        double g8  = rx * ry * rz * rz;
        double g9  = rx * rz * rz * rz;
        double g10 = ry * ry * ry * ry;
        double g11 = ry * ry * ry * rz;
        double g12 = ry * ry * rz * rz;
        double g13 = ry * rz * rz * rz;
        double g14 = rz * rz * rz * rz;
        omega[0] = 2.503342941796704538 * (g1 - g6);
        omega[1] = 5.310392309339791593 * g4 - 1.770130769779930530 * g11;
        omega[2] = 5.677048174545360108 * g8 - 0.946174695757560014 * (g1 + g6);
        omega[3] = 2.676186174229156671 * g13- 2.007139630671867500 * (g4 + g11);
        omega[4] = 0.317356640745612911 * (g0 + g10) + 0.634713281491225822 * g3 - 2.538853125964903290 * (g5 + g12) + 0.846284375321634430 * g14;
        omega[5] = 2.676186174229156671 * g9 - 2.007139630671867500 * (g2 + g7);
        omega[6] = 2.838524087272680054 * (g5 - g12) + 0.473087347878780009 * (g10 - g0);
        omega[7] = 1.770130769779930531 * g2 - 5.310392309339791590 * g7 ;
        omega[8] = 0.625835735449176134 * (g0  + g10) - 3.755014412695056800 * g3;
    }
}

template <int L>
__global__
void _ang_nuc_part(double *omega, double *x, int n){
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx >= n){
        return;
    }
    int offset = idx * (2*L+1);
    ang_nuc_part<L>(omega+offset, x[3*n], x[3*n+1], x[3*n+2]);
}

template <int LMAX> __device__
void type1_rad_part(double *rad_all, double k, double aij,
                    double *ur, int inc)
{
    constexpr int LMAX1 = LMAX + 1;
    double rur[LMAX1];
    double bval[NGAUSS*LMAX1];

    double kaij = k / (2*aij);
    double fac = kaij * kaij * aij;
    for (int n = 0; n < NGAUSS; n++){
        double tmp = r99[n*inc] - kaij;
        tmp = fac - aij*tmp*tmp;
        if (ur[n] == 0 || tmp > CUTOFF || tmp < -(EXPCUTOFF+6.+30.)) {
            rur[n] = 0;
            for (int i = 0; i < LMAX1; i++){
                bval[n*LMAX1 + i] = 0;
            }
        } else {
            rur[n] = ur[n] * exp(tmp);
            _ine(bval+n*LMAX1, LMAX, k*r99[n*inc]);
        }
    }

    for (int lab = 0; lab <= LMAX; lab++){
        if (lab > 0){
            for (int n = 0; n < NGAUSS; n++){
                rur[n] *= r99[n];
            }
        }
        double *prad = rad_all + lab * LMAX1;
        for (int i = lab%2; i <= LMAX; i+=2){
            double s = prad[i];
            for (int n = 0; n < NGAUSS; n++){
                s += rur[n] * bval[n*LMAX1+i];
            }
            prad[i] = s;
        }
    }
}

/*
template <int LMAX> __device__
void type1_rad_ang(double *rad_ang, double *r, double *rad_all)
{
    double unitr[3];
    if (r[0] == 0 && r[1] == 0 && r[2] == 0) {
        unitr[0] = 0;
        unitr[1] = 0;
        unitr[2] = 0;
    } else {
        double norm_r = -1/sqrt(r[0]*r[0] + r[1]*r[1] + r[2]*r[2]);
        unitr[0] = r[0] * norm_r;
        unitr[1] = r[1] * norm_r;
        unitr[2] = r[2] * norm_r;
    }

    double omega_nuc[CART_CUM];
    double *pnuc;
    for (int i = 0; i <= LMAX; i++) {
        pnuc = omega_nuc + _offset_cart[i];
        ang_nuc_in_cart(pnuc, i, unitr);
    }

    const int d1 = LMAX + 1;
    const int d2 = d1 * d1;
    const int d3 = d2 * d1;

    double *pout, *prad;
    for (int i = 0; i < d3; i++) { rad_ang[i] = 0; }
    for (int i = 0; i <= LMAX; i++) {
    for (int j = 0; j <= LMAX-i; j++) {
    for (int k = 0; k <= LMAX-i-j; k++) {
        pout = rad_ang + i*d2+j*d1+k;
        prad = rad_all + (i+j+k)*d1;
        // need_even to ensure (a+b+c+lmb) is even
        int need_even = (i+j+k)%2;
        for (int lmb = need_even; lmb <= LMAX; lmb+=2) {
            double tmp = 0;
            pnuc = omega_nuc + _offset_cart[lmb];
            for (int n = 0; n < (lmb+1)*(lmb+2)/2; n++){
                int ps = _cart_pow_y[n];
                int pt = _cart_pow_z[n];
                int pr = lmb - ps - pt;
                tmp += pnuc[n] * int_unit_xyz(i+pr, j+ps, k+pt);
            }
            *pout += prad[lmb] * tmp;
        }
    } } }
}
*/

/*
template <int LI, int LJ> __device__
static  void ECPtype1_cart(double *gctr, int *ecpbas, int necpbas,
                            int *atm, int natm,
                            int *bas, int nbas, double *env)
{
    if (necpbas == 0){
        return 0;
    }

    const int ish = blockIdx.x * blockDim.x + threadIdx.x;
    const int jsh = blockIdx.y * blockDim.y + threadIdx.y;
    const int npi = bas[NPRIM_OF+ish*BAS_SLOTS];
    const int npj = bas[NPRIM_OF+jsh*BAS_SLOTS];
    const int nfi = (LI+1) * (LI+2) / 2;
    const int nfj = (LI+1) * (LJ+2) / 2;
    const double *ai = env + bas[PTR_EXP+ish*BAS_SLOTS];
    const double *aj = env + bas[PTR_EXP+jsh*BAS_SLOTS];
    const double *ci = env + bas[PTR_COEFF+ish*BAS_SLOTS];
    const double *cj = env + bas[PTR_COEFF+jsh*BAS_SLOTS];
    const double *ri = env + atm[PTR_COORD+bas[ATOM_OF+ish*BAS_SLOTS]*ATM_SLOTS];
    const double *rj = env + atm[PTR_COORD+bas[ATOM_OF+jsh*BAS_SLOTS]*ATM_SLOTS];

    for (int iloc = 0; iloc < nslots; iloc++){
        if (ecpbas[ANG_OF+ecploc[iloc]*BAS_SLOTS] != -1 || ecpbas[SO_TYPE_OF+ecploc[iloc]*BAS_SLOTS] == 1) {
            continue;
            }

        int atm_id = ecpbas[ATOM_OF+ecploc[iloc]*BAS_SLOTS];
        rc = env + atm[PTR_COORD+atm_id*ATM_SLOTS];
        ecpshls = ecploc + iloc;

        rca[0] = rc[0] - ri[0];
        rca[1] = rc[1] - ri[1];
        rca[2] = rc[2] - ri[2];
        rcb[0] = rc[0] - rj[0];
        rcb[1] = rc[1] - rj[1];
        rcb[2] = rc[2] - rj[2];
    }

}
*/

extern "C" {
int ECPsph_ine(double *out, int order, double *zs, int n)
{
    int ntile = (n + THREADS - 1) / THREADS;
    dim3 threads(THREADS);
    dim3 blocks(ntile);
    _ine_kernel<<<blocks, threads>>>(out, order, zs, n);
    hipError_t err = hipGetLastError();
    if (err != hipSuccess) {
        return 1;
    }
    return 0;
}

int ECPang_nuc_part(double *omega, double *x, int n, const int l){
    int ntile = (n + THREADS - 1) / THREADS;
    dim3 threads(THREADS);
    dim3 blocks(ntile);
    switch (l){
    case 0: _ang_nuc_part<0><<<blocks, threads>>>(omega, x, n); break;
    case 1: _ang_nuc_part<1><<<blocks, threads>>>(omega, x, n); break;
    case 2: _ang_nuc_part<2><<<blocks, threads>>>(omega, x, n); break;
    case 3: _ang_nuc_part<3><<<blocks, threads>>>(omega, x, n); break;
    case 4: _ang_nuc_part<4><<<blocks, threads>>>(omega, x, n); break;
    default:
        break;
    }

    hipError_t err = hipGetLastError();
    if (err != hipSuccess) {
        return 1;
    }
    return 0;
}
}
