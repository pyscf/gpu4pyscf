/* Copyright 2023 The GPU4PySCF Authors. All Rights Reserved.
 *
 * This program is free software: you can redistribute it and/or modify
 * it under the terms of the GNU General Public License as published by
 * the Free Software Foundation, either version 3 of the License, or
 * (at your option) any later version.
 *
 * This program is distributed in the hope that it will be useful,
 * but WITHOUT ANY WARRANTY; without even the implied warranty of
 * MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
 * GNU General Public License for more details.
 *
 * You should have received a copy of the GNU General Public License
 * along with this program.  If not, see <http://www.gnu.org/licenses/>.
 */

#include <stdio.h>
#include <stdint.h>
#include <stdlib.h>
#include <string.h>
#include <hip/hip_runtime.h>

#include "gint.h"
#include "config.h"
#include "cuda_alloc.cuh"
#include "cint2e.cuh"
#include "g2e.h"

#include "rys_roots.cu"
#include "g2e.cu"
#include "gout3c2e.cu"
#include "g2e_root2.cu"
#include "g2e_root3.cu"
#include "g3c2e.cu"

static int GINTfill_int3c2e_tasks(ERITensor *eri, BasisProdOffsets *offsets, GINTEnvVars *envs, hipStream_t stream)
{
    int nrys_roots = envs->nrys_roots;
    int ntasks_ij = offsets->ntasks_ij;
    int ntasks_kl = offsets->ntasks_kl;
    assert(ntasks_kl < 65536*THREADSY);
    int type_ijkl;

    dim3 threads(THREADSX, THREADSY);
    dim3 blocks((ntasks_ij+THREADSX-1)/THREADSX, (ntasks_kl+THREADSY-1)/THREADSY);
    switch (nrys_roots) {
    case 1:
        type_ijkl = (envs->i_l << 3) | (envs->j_l << 2) | (envs->k_l << 1) | envs->l_l;
        switch (type_ijkl) {
        case 0b0000: GINTfill_int3c2e_kernel0000<<<blocks, threads, 0, stream>>>(*envs, *eri, *offsets); break;
        case 0b0010: GINTfill_int3c2e_kernel0010<<<blocks, threads, 0, stream>>>(*envs, *eri, *offsets); break;
        case 0b1000: GINTfill_int3c2e_kernel1000<<<blocks, threads, 0, stream>>>(*envs, *eri, *offsets); break;
        default:
            fprintf(stderr, "roots=1 type_ijkl %d\n", type_ijkl);
        }
        break;
    case 2:
        type_ijkl = (envs->i_l << 6) | (envs->j_l << 4) | (envs->k_l << 2) | envs->l_l;
        switch (type_ijkl) {
        case (0<<6)|(0<<4)|(2<<2)|0: GINTfill_int2e_kernel0020<<<blocks, threads, 0, stream>>>(*envs, *eri, *offsets); break;
        case (0<<6)|(0<<4)|(3<<2)|0: GINTfill_int2e_kernel0030<<<blocks, threads, 0, stream>>>(*envs, *eri, *offsets); break;
        case (1<<6)|(0<<4)|(1<<2)|0: GINTfill_int2e_kernel1010<<<blocks, threads, 0, stream>>>(*envs, *eri, *offsets); break;
        case (1<<6)|(0<<4)|(2<<2)|0: GINTfill_int2e_kernel1020<<<blocks, threads, 0, stream>>>(*envs, *eri, *offsets); break;
        case (1<<6)|(1<<4)|(0<<2)|0: GINTfill_int2e_kernel1100<<<blocks, threads, 0, stream>>>(*envs, *eri, *offsets); break;
        case (1<<6)|(1<<4)|(1<<2)|0: GINTfill_int2e_kernel1110<<<blocks, threads, 0, stream>>>(*envs, *eri, *offsets); break;
        case (2<<6)|(0<<4)|(0<<2)|0: GINTfill_int2e_kernel2000<<<blocks, threads, 0, stream>>>(*envs, *eri, *offsets); break;
        case (2<<6)|(0<<4)|(1<<2)|0: GINTfill_int2e_kernel2010<<<blocks, threads, 0, stream>>>(*envs, *eri, *offsets); break;
        case (2<<6)|(1<<4)|(0<<2)|0: GINTfill_int2e_kernel2100<<<blocks, threads, 0, stream>>>(*envs, *eri, *offsets); break;
        case (3<<6)|(0<<4)|(0<<2)|0: GINTfill_int2e_kernel3000<<<blocks, threads, 0, stream>>>(*envs, *eri, *offsets); break;
        default:
            GINTfill_int3c2e_kernel<2, GSIZE2_INT3C> <<<blocks, threads, 0, stream>>>(*envs, *eri, *offsets); break;
        }
        break;
    case 3:
        type_ijkl = (envs->i_l << 6) | (envs->j_l << 4) | (envs->k_l << 2) | envs->l_l;
        switch (type_ijkl) {
        case (1<<6)|(0<<4)|(3<<2)|0: GINTfill_int2e_kernel1030<<<blocks, threads, 0, stream>>>(*envs, *eri, *offsets); break;
        case (1<<6)|(1<<4)|(2<<2)|0: GINTfill_int2e_kernel1120<<<blocks, threads, 0, stream>>>(*envs, *eri, *offsets); break;
        case (1<<6)|(1<<4)|(3<<2)|0: GINTfill_int2e_kernel1130<<<blocks, threads, 0, stream>>>(*envs, *eri, *offsets); break;
        case (2<<6)|(0<<4)|(2<<2)|0: GINTfill_int2e_kernel2020<<<blocks, threads, 0, stream>>>(*envs, *eri, *offsets); break;
        case (2<<6)|(0<<4)|(3<<2)|0: GINTfill_int2e_kernel2030<<<blocks, threads, 0, stream>>>(*envs, *eri, *offsets); break;
        case (2<<6)|(1<<4)|(1<<2)|0: GINTfill_int2e_kernel2110<<<blocks, threads, 0, stream>>>(*envs, *eri, *offsets); break;
        case (2<<6)|(1<<4)|(2<<2)|0: GINTfill_int2e_kernel2120<<<blocks, threads, 0, stream>>>(*envs, *eri, *offsets); break;
        case (2<<6)|(2<<4)|(0<<2)|0: GINTfill_int2e_kernel2200<<<blocks, threads, 0, stream>>>(*envs, *eri, *offsets); break;
        case (2<<6)|(2<<4)|(1<<2)|0: GINTfill_int2e_kernel2210<<<blocks, threads, 0, stream>>>(*envs, *eri, *offsets); break;
        case (3<<6)|(0<<4)|(1<<2)|0: GINTfill_int2e_kernel3010<<<blocks, threads, 0, stream>>>(*envs, *eri, *offsets); break;
        case (3<<6)|(0<<4)|(2<<2)|0: GINTfill_int2e_kernel3020<<<blocks, threads, 0, stream>>>(*envs, *eri, *offsets); break;
        case (3<<6)|(1<<4)|(0<<2)|0: GINTfill_int2e_kernel3100<<<blocks, threads, 0, stream>>>(*envs, *eri, *offsets); break;
        case (3<<6)|(1<<4)|(1<<2)|0: GINTfill_int2e_kernel3110<<<blocks, threads, 0, stream>>>(*envs, *eri, *offsets); break;
        case (3<<6)|(2<<4)|(0<<2)|0: GINTfill_int2e_kernel3200<<<blocks, threads, 0, stream>>>(*envs, *eri, *offsets); break;
        default:
            GINTfill_int3c2e_kernel<3, GSIZE3_INT3C> <<<blocks, threads, 0, stream>>>(*envs, *eri, *offsets); break;
        }
        break;
    case 4: GINTfill_int3c2e_kernel<4, GSIZE4_INT3C> <<<blocks, threads, 0, stream>>>(*envs, *eri, *offsets); break;
    case 5: GINTfill_int3c2e_kernel<5, GSIZE5_INT3C> <<<blocks, threads, 0, stream>>>(*envs, *eri, *offsets); break;
    case 6: GINTfill_int3c2e_kernel<6, GSIZE6_INT3C> <<<blocks, threads, 0, stream>>>(*envs, *eri, *offsets); break;
    case 7: GINTfill_int3c2e_kernel<7, GSIZE7_INT3C> <<<blocks, threads, 0, stream>>>(*envs, *eri, *offsets); break;
    case 8: GINTfill_int3c2e_kernel<8, GSIZE8_INT3C> <<<blocks, threads, 0, stream>>>(*envs, *eri, *offsets); break;
    case 9: GINTfill_int3c2e_kernel<9, GSIZE9_INT3C> <<<blocks, threads, 0, stream>>>(*envs, *eri, *offsets); break;
    default:
        fprintf(stderr, "rys roots %d\n", nrys_roots);
        return 1;
    }

    hipError_t err = hipGetLastError();
    if (err != hipSuccess) {
        fprintf(stderr, "CUDA Error of GINTfill_int3c2e_kernel: %s\n", hipGetErrorString(err));
        return 1;
    }
    return 0;
}

extern "C" {
int GINTfill_int3c2e(hipStream_t stream, BasisProdCache *bpcache, double *eri, int nao,
                   int *strides, int *ao_offsets,
                   int *bins_locs_ij, int *bins_locs_kl, int nbins,
                   int cp_ij_id, int cp_kl_id, double omega)
{
    ContractionProdType *cp_ij = bpcache->cptype + cp_ij_id;
    ContractionProdType *cp_kl = bpcache->cptype + cp_kl_id;
    GINTEnvVars envs;
    int ng[4] = {0,0,0,0};

    GINTinit_EnvVars(&envs, cp_ij, cp_kl, ng);
    envs.omega = omega;

    if (envs.nrys_roots > 9) {
        return 2;
    }

    // TODO: improve the efficiency by unrolling
    if (envs.nrys_roots > 1) {
        int16_t *idx4c = (int16_t *)malloc(sizeof(int16_t) * envs.nf * 3);
        GINTg2e_index_xyz(idx4c, &envs);
        checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(c_idx4c), idx4c, sizeof(int16_t)*envs.nf*3));
        free(idx4c);
    }

    int kl_bin, ij_bin1;

    //checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(c_envs), &envs, sizeof(GINTEnvVars)));
    // move bpcache to constant memory
    checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(c_bpcache), bpcache, sizeof(BasisProdCache)));

    ERITensor eritensor;
    eritensor.stride_j = strides[1];
    eritensor.stride_k = strides[2];
    eritensor.stride_l = strides[3];
    eritensor.ao_offsets_i = ao_offsets[0];
    eritensor.ao_offsets_j = ao_offsets[1];
    eritensor.ao_offsets_k = ao_offsets[2];
    eritensor.ao_offsets_l = ao_offsets[3];
    eritensor.nao = nao;
    eritensor.data = eri;
    BasisProdOffsets offsets;

    int *bas_pairs_locs = bpcache->bas_pairs_locs;
    int *primitive_pairs_locs = bpcache->primitive_pairs_locs;
    for (kl_bin = 0; kl_bin < nbins; kl_bin++) {
        int bas_kl0 = bins_locs_kl[kl_bin];
        int bas_kl1 = bins_locs_kl[kl_bin+1];
        int ntasks_kl = bas_kl1 - bas_kl0;
        if (ntasks_kl <= 0) {
            continue;
        }
        // ij_bin + kl_bin < nbins <~> e_ij*e_kl < cutoff
        ij_bin1 = nbins - kl_bin;
        int bas_ij0 = bins_locs_ij[0];
        int bas_ij1 = bins_locs_ij[ij_bin1];
        int ntasks_ij = bas_ij1 - bas_ij0;
        if (ntasks_ij <= 0) {
            continue;
        }
        offsets.ntasks_ij = ntasks_ij;
        offsets.ntasks_kl = ntasks_kl;
        offsets.bas_ij = bas_pairs_locs[cp_ij_id] + bas_ij0;
        offsets.bas_kl = bas_pairs_locs[cp_kl_id] + bas_kl0;

        offsets.primitive_ij = primitive_pairs_locs[cp_ij_id] + bas_ij0 * envs.nprim_ij;
        offsets.primitive_kl = primitive_pairs_locs[cp_kl_id] + bas_kl0 * envs.nprim_kl;

        int err = -1;
        err = GINTfill_int3c2e_tasks(&eritensor, &offsets, &envs, stream);

        if (err != 0) {
            return err;
        }
    }

    return 0;
}
}
