#include "hip/hip_runtime.h"
/*
 * Copyright 2021-2024 The PySCF Developers. All Rights Reserved.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include "gvhf-rys/rys_roots.cuh"

#define SQRTPIE4        .8862269254527580136
#define PIE4            .7853981633974483096

template <int NROOTS> __device__
static void GINTrys_root(double x, double *rw)
{
    constexpr int off = NROOTS * (NROOTS - 1) / 2;
    const double t = sqrt(PIE4/x);

    if (x<3.0e-7){
        for (int rt_id = 0; rt_id < NROOTS; ++rt_id) {
            const double r = ROOT_SMALLX_R0[off+rt_id] + ROOT_SMALLX_R1[off+rt_id] * x;
            const double w = ROOT_SMALLX_W0[off+rt_id] + ROOT_SMALLX_W1[off+rt_id] * x;
            rw[rt_id] = r / (1 - r);
            rw[rt_id+NROOTS] = w;
        }
        return;
    }

    if (x>35+NROOTS*5){
        for (int rt_id = 0; rt_id < NROOTS; ++rt_id) {
            const double r = ROOT_LARGEX_R_DATA[off+rt_id] / x;
            const double w = ROOT_LARGEX_W_DATA[off+rt_id] * t;
            rw[rt_id] = r / (1 - r);
            rw[rt_id+NROOTS] = w;
        }
        return;
    }

    for (int rt_id = 0; rt_id < NROOTS; ++rt_id) {
        const int it = (int)(x * .4);
        double *datax = ROOT_RW_DATA + DEGREE1*INTERVALS * NROOTS*(NROOTS-1);
        const double u = (x - it * 2.5) * 0.8 - 1.;
        const double u2 = u * 2.;
        double *c = datax + (2*rt_id) * DEGREE1 * INTERVALS;
        //for i in range(2, degree + 1):
        //    c0, c1 = c[degree-i] - c1, c0 + c1*u2
        double c0 = c[it + DEGREE   *INTERVALS];
        double c1 = c[it +(DEGREE-1)*INTERVALS];
        double c2, c3;

        double r, w;
#pragma unroll
        for (int n = DEGREE-2; n > 0; n-=2) {
            c2 = c[it + n   *INTERVALS] - c1;
            c3 = c0 + c1*u2;
            c1 = c2 + c3*u2;
            c0 = c[it +(n-1)*INTERVALS] - c3;
        }
        if (DEGREE % 2 == 0) {
            c2 = c[it] - c1;
            c3 = c0 + c1*u2;
            r = c2 + c3*u;
        } else {
            r = c0 + c1*u;
        }

        // For weights
        c = datax + (2*rt_id+1) * DEGREE1 * INTERVALS;
        c0 = c[it + DEGREE   *INTERVALS];
        c1 = c[it +(DEGREE-1)*INTERVALS];

#pragma unroll
        for (int n = DEGREE-2; n > 0; n-=2) {
            c2 = c[it + n   *INTERVALS] - c1;
            c3 = c0 + c1*u2;
            c1 = c2 + c3*u2;
            c0 = c[it +(n-1)*INTERVALS] - c3;
        }
        if (DEGREE % 2 == 0) {
            c2 = c[it] - c1;
            c3 = c0 + c1*u2;
            w = c2 + c3*u;
        } else {
            w = c0 + c1*u;
        }

        rw[rt_id] = r / (1 - r);
        rw[rt_id+NROOTS] = w;
    }
}

template<> __device__
inline void GINTrys_root<1>(double x, double *rw)
{
    double tt = sqrt(x);
    double fmt0 = SQRTPIE4 / tt * erf(tt);
    rw[1] = fmt0;
    double e = exp(-x);
    double b = .5 / x;
    double fmt1 = b * (fmt0 - e);
    rw[0] = fmt1 / (fmt0 - fmt1);
    return;
}

template <int NROOTS> __device__
inline void GINTscale_u(double *u, double theta)
{
# pragma unroll
    for(int i = 0; i < NROOTS; i++){
        u[i] /= u[i] + 1 - u[i] * theta;
    }
}

__device__
static void GINTrys_root(int nroots, double x, double *rw)
{
    // roots and weights are distributed in each thread
    const int off = nroots * (nroots - 1) / 2;
    const double t = sqrt(PIE4/x);
    const int rt_id = threadIdx.x % nroots;
    if (x<3.0e-7){
        const double r = ROOT_SMALLX_R0[off+rt_id] + ROOT_SMALLX_R1[off+rt_id] * x;
        const double w = ROOT_SMALLX_W0[off+rt_id] + ROOT_SMALLX_W1[off+rt_id] * x;
        rw[0] = r / (1 - r);
        rw[1] = w;
        return;
    }

    if (x>35+nroots*5){
        const double r = ROOT_LARGEX_R_DATA[off+rt_id] / x;
        const double w = ROOT_LARGEX_W_DATA[off+rt_id] * t;
        rw[0] = r / (1 - r);
        rw[1] = w;
        return;
    }

    const int it = (int)(x * .4);
    double *datax = ROOT_RW_DATA + DEGREE1*INTERVALS * nroots*(nroots-1);
    const double u = (x - it * 2.5) * 0.8 - 1.;
    const double u2 = u * 2.;
    double *c = datax + (2*rt_id) * DEGREE1 * INTERVALS;
    //for i in range(2, degree + 1):
    //    c0, c1 = c[degree-i] - c1, c0 + c1*u2
    double c0 = c[it + DEGREE   *INTERVALS];
    double c1 = c[it +(DEGREE-1)*INTERVALS];
    double c2, c3;

    double r, w;
#pragma unroll
    for (int n = DEGREE-2; n > 0; n-=2) {
        c2 = c[it + n   *INTERVALS] - c1;
        c3 = c0 + c1*u2;
        c1 = c2 + c3*u2;
        c0 = c[it +(n-1)*INTERVALS] - c3;
    }
    if (DEGREE % 2 == 0) {
        c2 = c[it] - c1;
        c3 = c0 + c1*u2;
        r = c2 + c3*u;
    } else {
        r = c0 + c1*u;
    }

    // For weights
    c = datax + (2*rt_id+1) * DEGREE1 * INTERVALS;
    c0 = c[it + DEGREE   *INTERVALS];
    c1 = c[it +(DEGREE-1)*INTERVALS];

#pragma unroll
    for (int n = DEGREE-2; n > 0; n-=2) {
        c2 = c[it + n   *INTERVALS] - c1;
        c3 = c0 + c1*u2;
        c1 = c2 + c3*u2;
        c0 = c[it +(n-1)*INTERVALS] - c3;
    }
    if (DEGREE % 2 == 0) {
        c2 = c[it] - c1;
        c3 = c0 + c1*u2;
        w = c2 + c3*u;
    } else {
        w = c0 + c1*u;
    }

    rw[0] = r / (1 - r);
    rw[1] = w;
}
