/*
 * gpu4pyscf is a plugin to use Nvidia GPU in PySCF package
 *
 * Copyright (C) 2022 Qiming Sun
 *
 * This program is free software: you can redistribute it and/or modify
 * it under the terms of the GNU General Public License as published by
 * the Free Software Foundation, either version 3 of the License, or
 * (at your option) any later version.
 *
 * This program is distributed in the hope that it will be useful,
 * but WITHOUT ANY WARRANTY; without even the implied warranty of
 * MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
 * GNU General Public License for more details.
 *
 * You should have received a copy of the GNU General Public License
 * along with this program.  If not, see <http://www.gnu.org/licenses/>.
 */

#include <stdio.h>
#include <stdint.h>
#include <stdlib.h>
#include <string.h>
#include <hip/hip_runtime.h>

#include "gint.h"
#include "config.h"
#include "cuda_alloc.cuh"
#include "g2e.h"
/*
#include "cint2e.cuh"
#include "fill_ints.cu"
#include "g2e.cu"
#include "rys_roots.cu"
#include "g2e_root2.cu"
#include "g2e_root3.cu"
#include "g3c2e.cu"
#include "g3c2e_ip1.cu"
#include "g3c2e_ip2.cu"
*/
extern "C" { __host__
void GINTdel_basis_prod(BasisProdCache **pbp)
{
    BasisProdCache *bpcache = *pbp;
    if (bpcache == NULL) {
        return;
    }
    
    if (bpcache->cptype != NULL) {
        free(bpcache->cptype);
        free(bpcache->primitive_pairs_locs);
    }
    
    if (bpcache->aexyz != NULL) {
        free(bpcache->aexyz);
    }
    
    if (bpcache->a12 != NULL) {
        FREE(bpcache->bas_coords);
        FREE(bpcache->bas_pair2bra);
        FREE(bpcache->ao_loc);
        FREE(bpcache->a12);
    }

    free(bpcache);
    *pbp = NULL;
}

void GINTinit_basis_prod(BasisProdCache **pbp, double diag_fac, int *ao_loc,
                         int *bas_pair2shls, int *bas_pairs_locs, int ncptype,
                         int *atm, int natm, int *bas, int nbas, double *env)
{
    BasisProdCache *bpcache = (BasisProdCache *)malloc(sizeof(BasisProdCache));
    memset(bpcache, 0, sizeof(BasisProdCache));
    *pbp = bpcache;

    GINTinit_contraction_types(bpcache, bas_pair2shls, bas_pairs_locs, ncptype,
                               atm, natm, bas, nbas, env);
    int n_bas_pairs = bpcache->bas_pairs_locs[ncptype];
    int n_primitive_pairs = bpcache->primitive_pairs_locs[ncptype];
    double *aexyz = (double *)malloc(sizeof(double) * n_primitive_pairs * 7);
    GINTinit_aexyz(aexyz, bpcache, diag_fac, atm, natm, bas, nbas, env);
    bpcache->aexyz = aexyz;
    bpcache->bas_pair2shls = bas_pair2shls;

    // initialize ao_loc on GPU
    DEVICE_INIT(int, d_ao_loc, ao_loc, nbas+1);
    bpcache->ao_loc = d_ao_loc;

    // initialize basis coordinates on GPU memory
    bpcache->nbas = nbas;
    double *bas_coords = (double *)malloc(sizeof(double) * nbas * 3);
    GINTsort_bas_coordinates(bas_coords, atm, natm, bas, nbas, env);
    DEVICE_INIT(double, d_bas_coords, bas_coords, nbas * 3);
    bpcache->bas_coords = d_bas_coords;
    free(bas_coords);

    // initialize pair data on GPU memory
    DEVICE_INIT(double, d_aexyz, aexyz, n_primitive_pairs * 7);
    DEVICE_INIT(int, d_bas_pair2shls, bas_pair2shls, n_bas_pairs * 2);
    bpcache->a12 = d_aexyz;
    bpcache->e12 = d_aexyz + n_primitive_pairs * 1;
    bpcache->x12 = d_aexyz + n_primitive_pairs * 2;
    bpcache->y12 = d_aexyz + n_primitive_pairs * 3;
    bpcache->z12 = d_aexyz + n_primitive_pairs * 4;
    bpcache->a1  = d_aexyz + n_primitive_pairs * 5;
    bpcache->a2  = d_aexyz + n_primitive_pairs * 6;
    bpcache->bas_pair2bra = d_bas_pair2shls;
    bpcache->bas_pair2ket = d_bas_pair2shls + n_bas_pairs;
}
}

