#include "hip/hip_runtime.h"
/* Copyright 2024 The GPU4PySCF Authors. All Rights Reserved.
 *
 * This program is free software: you can redistribute it and/or modify
 * it under the terms of the GNU General Public License as published by
 * the Free Software Foundation, either version 3 of the License, or
 * (at your option) any later version.
 *
 * This program is distributed in the hope that it will be useful,
 * but WITHOUT ANY WARRANTY; without even the implied warranty of
 * MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
 * GNU General Public License for more details.
 *
 * You should have received a copy of the GNU General Public License
 * along with this program.  If not, see <http://www.gnu.org/licenses/>.
 */

#include <math.h>
#include "cint2e.cuh"

__global__
static void GINTfill_int3c1e_ip_kernel00(double* output, const BasisProdOffsets offsets, const int i_l, const int j_l, const int nprim_ij,
                                         const int stride_j, const int stride_ij, const int ao_offsets_i, const int ao_offsets_j,
                                         const double omega, const double* grid_points, const double* charge_exponents)
{
    const int ntasks_ij = offsets.ntasks_ij;
    const int ngrids = offsets.ntasks_kl;
    const int task_ij = blockIdx.x * blockDim.x + threadIdx.x;
    const int task_grid = blockIdx.y * blockDim.y + threadIdx.y;

    if (task_ij >= ntasks_ij || task_grid >= ngrids) {
        return;
    }
    const int bas_ij = offsets.bas_ij + task_ij;
    const int prim_ij = offsets.primitive_ij + task_ij * nprim_ij;
    const int* bas_pair2bra = c_bpcache.bas_pair2bra;
    const int* bas_pair2ket = c_bpcache.bas_pair2ket;
    const int ish = bas_pair2bra[bas_ij];
    const int jsh = bas_pair2ket[bas_ij];

    const double* __restrict__ a12 = c_bpcache.a12;
    const double* __restrict__ e12 = c_bpcache.e12;
    const double* __restrict__ x12 = c_bpcache.x12;
    const double* __restrict__ y12 = c_bpcache.y12;
    const double* __restrict__ z12 = c_bpcache.z12;

    const double* __restrict__ a_exponents = c_bpcache.a1;
    const int nbas = c_bpcache.nbas;
    const double* __restrict__ bas_x = c_bpcache.bas_coords;
    const double* __restrict__ bas_y = bas_x + nbas;
    const double* __restrict__ bas_z = bas_y + nbas;
    const double Ax = bas_x[ish];
    const double Ay = bas_y[ish];
    const double Az = bas_z[ish];

    const double* grid_point = grid_points + task_grid * 3;
    const double Cx = grid_point[0];
    const double Cy = grid_point[1];
    const double Cz = grid_point[2];
    const double charge_exponent = (charge_exponents != NULL) ? charge_exponents[task_grid] : 0.0;

    double deri_dAx = 0;
    double deri_dAy = 0;
    double deri_dAz = 0;
    double deri_dCx = 0;
    double deri_dCy = 0;
    double deri_dCz = 0;
    for (int ij = prim_ij; ij < prim_ij + nprim_ij; ij++) {
        const double aij = a12[ij];
        const double eij = e12[ij];
        const double Px  = x12[ij];
        const double Py  = y12[ij];
        const double Pz  = z12[ij];
        const double PCx = Px - Cx;
        const double PCy = Py - Cy;
        const double PCz = Pz - Cz;
        const double PAx = Px - Ax;
        const double PAy = Py - Ay;
        const double PAz = Pz - Az;
        const double minus_two_a = -2.0 * a_exponents[ij];
        const double one_over_two_p = 0.5 / aij;
        double a0 = aij;
        const double q_over_p_plus_q = charge_exponent > 0.0 ? charge_exponent / (aij + charge_exponent) : 1.0;
        const double sqrt_q_over_p_plus_q = charge_exponent > 0.0 ? sqrt(q_over_p_plus_q) : 1.0;
        a0 *= q_over_p_plus_q;
        const double theta = omega > 0.0 ? omega * omega / (omega * omega + a0) : 1.0;
        const double sqrt_theta = omega > 0.0 ? sqrt(theta) : 1.0;
        a0 *= theta;

        const double prefactor = 2.0 * M_PI / aij * eij * sqrt_theta * sqrt_q_over_p_plus_q;
        const double boys_input = a0 * (PCx * PCx + PCy * PCy + PCz * PCz);
        if (boys_input > 3.e-7) {
            const double sqrt_boys_input = sqrt(boys_input);
            const double R000_0 = SQRTPIE4 / sqrt_boys_input * erf(sqrt_boys_input);
            const double R000_1 = -a0 * (R000_0 - exp(-boys_input)) / boys_input;
            const double R100_0 = R000_1 * PCx;
            const double R010_0 = R000_1 * PCy;
            const double R001_0 = R000_1 * PCz;
            deri_dAx += prefactor * minus_two_a * (PAx * R000_0 + one_over_two_p * R100_0);
            deri_dAy += prefactor * minus_two_a * (PAy * R000_0 + one_over_two_p * R010_0);
            deri_dAz += prefactor * minus_two_a * (PAz * R000_0 + one_over_two_p * R001_0);
            deri_dCx += prefactor * R100_0;
            deri_dCy += prefactor * R010_0;
            deri_dCz += prefactor * R001_0;
        }
    }

    const int* ao_loc = c_bpcache.ao_loc;
    const int i0 = ao_loc[ish] - ao_offsets_i;
    const int j0 = ao_loc[jsh] - ao_offsets_j;
    output[i0 + j0 * stride_j + task_grid * stride_ij + 0 * stride_ij * ngrids] = deri_dAx;
    output[i0 + j0 * stride_j + task_grid * stride_ij + 1 * stride_ij * ngrids] = deri_dAy;
    output[i0 + j0 * stride_j + task_grid * stride_ij + 2 * stride_ij * ngrids] = deri_dAz;
    output[i0 + j0 * stride_j + task_grid * stride_ij + 3 * stride_ij * ngrids] = deri_dCx;
    output[i0 + j0 * stride_j + task_grid * stride_ij + 4 * stride_ij * ngrids] = deri_dCy;
    output[i0 + j0 * stride_j + task_grid * stride_ij + 5 * stride_ij * ngrids] = deri_dCz;
}

__global__
static void GINTfill_int3c1e_ip1_charge_contracted_kernel00(double* output, const BasisProdOffsets offsets, const int nprim_ij,
                                                            const int stride_j, const int stride_ij, const int ao_offsets_i, const int ao_offsets_j,
                                                            const double omega, const double* grid_points, const double* charge_exponents)
{
    const int ntasks_ij = offsets.ntasks_ij;
    const int ngrids = offsets.ntasks_kl;
    const int task_ij = blockIdx.x * blockDim.x + threadIdx.x;
    if (task_ij >= ntasks_ij) {
        return;
    }

    const int bas_ij = offsets.bas_ij + task_ij;
    const int prim_ij = offsets.primitive_ij + task_ij * nprim_ij;
    const int* bas_pair2bra = c_bpcache.bas_pair2bra;
    const int* bas_pair2ket = c_bpcache.bas_pair2ket;
    const int ish = bas_pair2bra[bas_ij];
    const int jsh = bas_pair2ket[bas_ij];

    const double* __restrict__ a12 = c_bpcache.a12;
    const double* __restrict__ e12 = c_bpcache.e12;
    const double* __restrict__ x12 = c_bpcache.x12;
    const double* __restrict__ y12 = c_bpcache.y12;
    const double* __restrict__ z12 = c_bpcache.z12;

    const double* __restrict__ a_exponents = c_bpcache.a1;
    const int nbas = c_bpcache.nbas;
    const double* __restrict__ bas_x = c_bpcache.bas_coords;
    const double* __restrict__ bas_y = bas_x + nbas;
    const double* __restrict__ bas_z = bas_y + nbas;
    const double Ax = bas_x[ish];
    const double Ay = bas_y[ish];
    const double Az = bas_z[ish];

    double deri_dAx_grid_sum = 0;
    double deri_dAy_grid_sum = 0;
    double deri_dAz_grid_sum = 0;
    for (int task_grid = blockIdx.y * blockDim.y + threadIdx.y; task_grid < ngrids; task_grid += gridDim.y * blockDim.y) {
        const double* grid_point = grid_points + task_grid * 4;
        const double Cx = grid_point[0];
        const double Cy = grid_point[1];
        const double Cz = grid_point[2];
        const double charge_exponent = (charge_exponents != NULL) ? charge_exponents[task_grid] : 0.0;

        double deri_dAx_per_grid = 0;
        double deri_dAy_per_grid = 0;
        double deri_dAz_per_grid = 0;
        for (int ij = prim_ij; ij < prim_ij + nprim_ij; ij++) {
            const double aij = a12[ij];
            const double eij = e12[ij];
            const double Px  = x12[ij];
            const double Py  = y12[ij];
            const double Pz  = z12[ij];
            const double PCx = Px - Cx;
            const double PCy = Py - Cy;
            const double PCz = Pz - Cz;
            const double PAx = Px - Ax;
            const double PAy = Py - Ay;
            const double PAz = Pz - Az;
            const double minus_two_a = -2.0 * a_exponents[ij];
            const double one_over_two_p = 0.5 / aij;
            double a0 = aij;
            const double q_over_p_plus_q = charge_exponent > 0.0 ? charge_exponent / (aij + charge_exponent) : 1.0;
            const double sqrt_q_over_p_plus_q = charge_exponent > 0.0 ? sqrt(q_over_p_plus_q) : 1.0;
            a0 *= q_over_p_plus_q;
            const double theta = omega > 0.0 ? omega * omega / (omega * omega + a0) : 1.0;
            const double sqrt_theta = omega > 0.0 ? sqrt(theta) : 1.0;
            a0 *= theta;

            const double prefactor = 2.0 * M_PI / aij * eij * sqrt_theta * sqrt_q_over_p_plus_q;
            const double boys_input = a0 * (PCx * PCx + PCy * PCy + PCz * PCz);
            if (boys_input > 3.e-7) {
                const double sqrt_boys_input = sqrt(boys_input);
                const double R000_0 = SQRTPIE4 / sqrt_boys_input * erf(sqrt_boys_input);
                const double R000_1 = -a0 * (R000_0 - exp(-boys_input)) / boys_input;
                deri_dAx_per_grid += prefactor * minus_two_a * (PAx * R000_0 + one_over_two_p * R000_1 * PCx);
                deri_dAy_per_grid += prefactor * minus_two_a * (PAy * R000_0 + one_over_two_p * R000_1 * PCy);
                deri_dAz_per_grid += prefactor * minus_two_a * (PAz * R000_0 + one_over_two_p * R000_1 * PCz);
            }
        }

        const double charge = grid_point[3];
        deri_dAx_grid_sum += deri_dAx_per_grid * charge;
        deri_dAy_grid_sum += deri_dAy_per_grid * charge;
        deri_dAz_grid_sum += deri_dAz_per_grid * charge;
    }

    const int* ao_loc = c_bpcache.ao_loc;
    const int i0 = ao_loc[ish] - ao_offsets_i;
    const int j0 = ao_loc[jsh] - ao_offsets_j;
    atomicAdd(output + (i0 + j0 * stride_j + 0 * stride_ij), deri_dAx_grid_sum);
    atomicAdd(output + (i0 + j0 * stride_j + 1 * stride_ij), deri_dAy_grid_sum);
    atomicAdd(output + (i0 + j0 * stride_j + 2 * stride_ij), deri_dAz_grid_sum);
}

__global__
static void GINTfill_int3c1e_ip2_density_contracted_kernel00(double* output, const double* density, const HermiteDensityOffsets hermite_density_offsets,
                                                             const BasisProdOffsets offsets, const int nprim_ij,
                                                             const double omega, const double* grid_points, const double* charge_exponents)
{
    const int ntasks_ij = offsets.ntasks_ij;
    const int ngrids = offsets.ntasks_kl;
    const int task_grid = blockIdx.y * blockDim.y + threadIdx.y;
    if (task_grid >= ngrids) {
        return;
    }

    const double* grid_point = grid_points + task_grid * 3;
    const double Cx = grid_point[0];
    const double Cy = grid_point[1];
    const double Cz = grid_point[2];
    const double charge_exponent = (charge_exponents != NULL) ? charge_exponents[task_grid] : 0.0;

    double deri_dCx_pair_sum = 0.0;
    double deri_dCy_pair_sum = 0.0;
    double deri_dCz_pair_sum = 0.0;
    for (int task_ij = blockIdx.x * blockDim.x + threadIdx.x; task_ij < ntasks_ij; task_ij += gridDim.x * blockDim.x) {
        const int bas_ij = offsets.bas_ij + task_ij;
        const int prim_ij = offsets.primitive_ij + task_ij * nprim_ij;

        const double* __restrict__ a12 = c_bpcache.a12;
        const double* __restrict__ e12 = c_bpcache.e12;
        const double* __restrict__ x12 = c_bpcache.x12;
        const double* __restrict__ y12 = c_bpcache.y12;
        const double* __restrict__ z12 = c_bpcache.z12;

        double deri_dCx_per_pair = 0;
        double deri_dCy_per_pair = 0;
        double deri_dCz_per_pair = 0;
        for (int ij = prim_ij; ij < prim_ij + nprim_ij; ij++) {
            const double aij = a12[ij];
            const double eij = e12[ij];
            const double Px  = x12[ij];
            const double Py  = y12[ij];
            const double Pz  = z12[ij];
            const double PCx = Px - Cx;
            const double PCy = Py - Cy;
            const double PCz = Pz - Cz;
            double a0 = aij;
            const double q_over_p_plus_q = charge_exponent > 0.0 ? charge_exponent / (aij + charge_exponent) : 1.0;
            const double sqrt_q_over_p_plus_q = charge_exponent > 0.0 ? sqrt(q_over_p_plus_q) : 1.0;
            a0 *= q_over_p_plus_q;
            const double theta = omega > 0.0 ? omega * omega / (omega * omega + a0) : 1.0;
            const double sqrt_theta = omega > 0.0 ? sqrt(theta) : 1.0;
            a0 *= theta;

            const double prefactor = 2.0 * M_PI / aij * eij * sqrt_theta * sqrt_q_over_p_plus_q;
            const double boys_input = a0 * (PCx * PCx + PCy * PCy + PCz * PCz);
            if (boys_input > 3.e-7) {
                const double sqrt_boys_input = sqrt(boys_input);
                const double R000_0 = SQRTPIE4 / sqrt_boys_input * erf(sqrt_boys_input);
                const double R000_1 = -a0 * (R000_0 - exp(-boys_input)) / boys_input;
                deri_dCx_per_pair += prefactor * R000_1 * PCx;
                deri_dCy_per_pair += prefactor * R000_1 * PCy;
                deri_dCz_per_pair += prefactor * R000_1 * PCz;
            }
        }

        const double D = density[bas_ij - hermite_density_offsets.pair_offset_of_angular_pair + hermite_density_offsets.density_offset_of_angular_pair];
        deri_dCx_pair_sum += deri_dCx_per_pair * D;
        deri_dCy_pair_sum += deri_dCy_per_pair * D;
        deri_dCz_pair_sum += deri_dCz_per_pair * D;
    }
    atomicAdd(output + task_grid + 0 * ngrids, deri_dCx_pair_sum);
    atomicAdd(output + task_grid + 1 * ngrids, deri_dCy_pair_sum);
    atomicAdd(output + task_grid + 2 * ngrids, deri_dCz_pair_sum);
}
