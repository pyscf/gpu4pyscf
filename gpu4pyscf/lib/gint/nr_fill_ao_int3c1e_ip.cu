/* Copyright 2024 The GPU4PySCF Authors. All Rights Reserved.
 *
 * This program is free software: you can redistribute it and/or modify
 * it under the terms of the GNU General Public License as published by
 * the Free Software Foundation, either version 3 of the License, or
 * (at your option) any later version.
 *
 * This program is distributed in the hope that it will be useful,
 * but WITHOUT ANY WARRANTY; without even the implied warranty of
 * MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
 * GNU General Public License for more details.
 *
 * You should have received a copy of the GNU General Public License
 * along with this program.  If not, see <http://www.gnu.org/licenses/>.
 */

#include <stdio.h>
#include <stdint.h>
#include <stdlib.h>
#include <string.h>
#include <hip/hip_runtime.h>

#include "gint.h"
#include "gint1e.h"
#include "cuda_alloc.cuh"
#include "cint2e.cuh"

#include "rys_roots.cu"
#include "g1e.cu"
#include "g1e_ip_root_1.cu"
#include "g3c1e_ip.cu"

static int GINTfill_int3c1e_ip_tasks(double* output, const BasisProdOffsets offsets, const int i_l, const int j_l, const int nprim_ij,
                                     const int stride_j, const int stride_ij, const int ao_offsets_i, const int ao_offsets_j,
                                     const double omega, const double* grid_points, const double* charge_exponents, const hipStream_t stream)
{
    const int nrys_roots = (i_l + j_l + 1) / 2 + 1;
    const int ntasks_ij = offsets.ntasks_ij;
    const int ngrids = offsets.ntasks_kl;

    const dim3 threads(THREADSX, THREADSY);
    const dim3 blocks((ntasks_ij+THREADSX-1)/THREADSX, (ngrids+THREADSY-1)/THREADSY);
    int type_ijkl;
    switch (nrys_roots) {
    case 1:
        type_ijkl = (i_l + 1) * 10 + j_l;
        switch (type_ijkl) {
        case 10: GINTfill_int3c1e_ip_kernel00<<<blocks, threads, 0, stream>>>(output, offsets, i_l, j_l, nprim_ij, stride_j, stride_ij, ao_offsets_i, ao_offsets_j, omega, grid_points, charge_exponents); break;
        default:
            fprintf(stderr, "roots=1 type_ijkl %d\n", type_ijkl);
        }
        break;
    case 2: GINTfill_int3c1e_ip_kernel_general<2, GSIZE2_INT3C_1E> <<<blocks, threads, 0, stream>>>(output, offsets, i_l, j_l, nprim_ij, stride_j, stride_ij, ao_offsets_i, ao_offsets_j, omega, grid_points, charge_exponents); break;
    case 3: GINTfill_int3c1e_ip_kernel_general<3, GSIZE3_INT3C_1E> <<<blocks, threads, 0, stream>>>(output, offsets, i_l, j_l, nprim_ij, stride_j, stride_ij, ao_offsets_i, ao_offsets_j, omega, grid_points, charge_exponents); break;
    case 4: GINTfill_int3c1e_ip_kernel_general<4, GSIZE4_INT3C_1E> <<<blocks, threads, 0, stream>>>(output, offsets, i_l, j_l, nprim_ij, stride_j, stride_ij, ao_offsets_i, ao_offsets_j, omega, grid_points, charge_exponents); break;
    case 5: GINTfill_int3c1e_ip_kernel_general<5, GSIZE5_INT3C_1E> <<<blocks, threads, 0, stream>>>(output, offsets, i_l, j_l, nprim_ij, stride_j, stride_ij, ao_offsets_i, ao_offsets_j, omega, grid_points, charge_exponents); break;
    default:
        fprintf(stderr, "rys roots %d\n", nrys_roots);
        return 1;
    }

    hipError_t err = hipGetLastError();
    if (err != hipSuccess) {
        fprintf(stderr, "CUDA Error in %s: %s\n", __func__, hipGetErrorString(err));
        return 1;
    }
    return 0;
}

static int GINTfill_int3c1e_ip1_charge_contracted_tasks(double* output, const BasisProdOffsets offsets, const int i_l, const int j_l, const int nprim_ij,
                                                        const int stride_j, const int stride_ij, const int ao_offsets_i, const int ao_offsets_j,
                                                        const double omega, const double* grid_points, const double* charge_exponents,
                                                        const int n_charge_sum_per_thread, const hipStream_t stream)
{
    const int ntasks_ij = offsets.ntasks_ij;
    const int ngrids = (offsets.ntasks_kl + n_charge_sum_per_thread - 1) / n_charge_sum_per_thread;

    const dim3 threads(THREADSX, THREADSY);
    const dim3 blocks((ntasks_ij+THREADSX-1)/THREADSX, (ngrids+THREADSY-1)/THREADSY);
    const int type_ij = i_l * 10 + j_l;
    switch (type_ij) {
    case 00: GINTfill_int3c1e_ip1_charge_contracted_kernel00<<<blocks, threads, 0, stream>>>(output, offsets, nprim_ij, stride_j, stride_ij, ao_offsets_i, ao_offsets_j, omega, grid_points, charge_exponents); break;
    case 01: GINTfill_int3c1e_ip1_charge_contracted_kernel_expanded<0, 1> <<<blocks, threads, 0, stream>>>(output, offsets, nprim_ij, stride_j, stride_ij, ao_offsets_i, ao_offsets_j, omega, grid_points, charge_exponents); break;
    case 02: GINTfill_int3c1e_ip1_charge_contracted_kernel_expanded<0, 2> <<<blocks, threads, 0, stream>>>(output, offsets, nprim_ij, stride_j, stride_ij, ao_offsets_i, ao_offsets_j, omega, grid_points, charge_exponents); break;
    case 03: GINTfill_int3c1e_ip1_charge_contracted_kernel_expanded<0, 3> <<<blocks, threads, 0, stream>>>(output, offsets, nprim_ij, stride_j, stride_ij, ao_offsets_i, ao_offsets_j, omega, grid_points, charge_exponents); break;
    case 04: GINTfill_int3c1e_ip1_charge_contracted_kernel_expanded<0, 4> <<<blocks, threads, 0, stream>>>(output, offsets, nprim_ij, stride_j, stride_ij, ao_offsets_i, ao_offsets_j, omega, grid_points, charge_exponents); break;
    case 10: GINTfill_int3c1e_ip1_charge_contracted_kernel_expanded<1, 0> <<<blocks, threads, 0, stream>>>(output, offsets, nprim_ij, stride_j, stride_ij, ao_offsets_i, ao_offsets_j, omega, grid_points, charge_exponents); break;
    case 11: GINTfill_int3c1e_ip1_charge_contracted_kernel_expanded<1, 1> <<<blocks, threads, 0, stream>>>(output, offsets, nprim_ij, stride_j, stride_ij, ao_offsets_i, ao_offsets_j, omega, grid_points, charge_exponents); break;
    case 12: GINTfill_int3c1e_ip1_charge_contracted_kernel_expanded<1, 2> <<<blocks, threads, 0, stream>>>(output, offsets, nprim_ij, stride_j, stride_ij, ao_offsets_i, ao_offsets_j, omega, grid_points, charge_exponents); break;
    case 13: GINTfill_int3c1e_ip1_charge_contracted_kernel_expanded<1, 3> <<<blocks, threads, 0, stream>>>(output, offsets, nprim_ij, stride_j, stride_ij, ao_offsets_i, ao_offsets_j, omega, grid_points, charge_exponents); break;
    case 20: GINTfill_int3c1e_ip1_charge_contracted_kernel_expanded<2, 0> <<<blocks, threads, 0, stream>>>(output, offsets, nprim_ij, stride_j, stride_ij, ao_offsets_i, ao_offsets_j, omega, grid_points, charge_exponents); break;
    case 21: GINTfill_int3c1e_ip1_charge_contracted_kernel_expanded<2, 1> <<<blocks, threads, 0, stream>>>(output, offsets, nprim_ij, stride_j, stride_ij, ao_offsets_i, ao_offsets_j, omega, grid_points, charge_exponents); break;
    case 22: GINTfill_int3c1e_ip1_charge_contracted_kernel_expanded<2, 2> <<<blocks, threads, 0, stream>>>(output, offsets, nprim_ij, stride_j, stride_ij, ao_offsets_i, ao_offsets_j, omega, grid_points, charge_exponents); break;
    case 30: GINTfill_int3c1e_ip1_charge_contracted_kernel_expanded<3, 0> <<<blocks, threads, 0, stream>>>(output, offsets, nprim_ij, stride_j, stride_ij, ao_offsets_i, ao_offsets_j, omega, grid_points, charge_exponents); break;
    case 31: GINTfill_int3c1e_ip1_charge_contracted_kernel_expanded<3, 1> <<<blocks, threads, 0, stream>>>(output, offsets, nprim_ij, stride_j, stride_ij, ao_offsets_i, ao_offsets_j, omega, grid_points, charge_exponents); break;
    case 40: GINTfill_int3c1e_ip1_charge_contracted_kernel_expanded<4, 0> <<<blocks, threads, 0, stream>>>(output, offsets, nprim_ij, stride_j, stride_ij, ao_offsets_i, ao_offsets_j, omega, grid_points, charge_exponents); break;
    default:
        const int nrys_roots = (i_l + j_l + 1) / 2 + 1;
        switch (nrys_roots) {
        case 4: GINTfill_int3c1e_ip1_charge_contracted_kernel_general<4, GSIZE4_INT3C_1E> <<<blocks, threads, 0, stream>>>(output, offsets, i_l, j_l, nprim_ij, stride_j, stride_ij, ao_offsets_i, ao_offsets_j, omega, grid_points, charge_exponents); break;
        case 5: GINTfill_int3c1e_ip1_charge_contracted_kernel_general<5, GSIZE5_INT3C_1E> <<<blocks, threads, 0, stream>>>(output, offsets, i_l, j_l, nprim_ij, stride_j, stride_ij, ao_offsets_i, ao_offsets_j, omega, grid_points, charge_exponents); break;
        default:
            fprintf(stderr, "type_ij = %d, nrys_roots = %d out of range\n", type_ij, nrys_roots);
            return 1;
        }
    }

    hipError_t err = hipGetLastError();
    if (err != hipSuccess) {
        fprintf(stderr, "CUDA Error in %s: %s\n", __func__, hipGetErrorString(err));
        return 1;
    }
    return 0;
}

static int GINTfill_int3c1e_ip2_density_contracted_tasks(double* output, const double* density, const HermiteDensityOffsets hermite_density_offsets,
                                                         const BasisProdOffsets offsets, const int i_l, const int j_l, const int nprim_ij,
                                                         const double omega, const double* grid_points, const double* charge_exponents,
                                                         const int n_pair_sum_per_thread, const hipStream_t stream)
{
    const int ntasks_ij = (offsets.ntasks_ij + n_pair_sum_per_thread - 1) / n_pair_sum_per_thread;
    const int ngrids = offsets.ntasks_kl;

    const dim3 threads(THREADSX, THREADSY);
    const dim3 blocks((ntasks_ij+THREADSX-1)/THREADSX, (ngrids+THREADSY-1)/THREADSY);
    switch (i_l + j_l) {
    case  0: GINTfill_int3c1e_ip2_density_contracted_kernel00<<<blocks, threads, 0, stream>>>(output, density, hermite_density_offsets, offsets, nprim_ij, omega, grid_points, charge_exponents); break;
    case  1: GINTfill_int3c1e_ip2_density_contracted_kernel_general< 1> <<<blocks, threads, 0, stream>>>(output, density, hermite_density_offsets, offsets, nprim_ij, omega, grid_points, charge_exponents); break;
    case  2: GINTfill_int3c1e_ip2_density_contracted_kernel_general< 2> <<<blocks, threads, 0, stream>>>(output, density, hermite_density_offsets, offsets, nprim_ij, omega, grid_points, charge_exponents); break;
    case  3: GINTfill_int3c1e_ip2_density_contracted_kernel_general< 3> <<<blocks, threads, 0, stream>>>(output, density, hermite_density_offsets, offsets, nprim_ij, omega, grid_points, charge_exponents); break;
    case  4: GINTfill_int3c1e_ip2_density_contracted_kernel_general< 4> <<<blocks, threads, 0, stream>>>(output, density, hermite_density_offsets, offsets, nprim_ij, omega, grid_points, charge_exponents); break;
    case  5: GINTfill_int3c1e_ip2_density_contracted_kernel_general< 5> <<<blocks, threads, 0, stream>>>(output, density, hermite_density_offsets, offsets, nprim_ij, omega, grid_points, charge_exponents); break;
    case  6: GINTfill_int3c1e_ip2_density_contracted_kernel_general< 6> <<<blocks, threads, 0, stream>>>(output, density, hermite_density_offsets, offsets, nprim_ij, omega, grid_points, charge_exponents); break;
    case  7: GINTfill_int3c1e_ip2_density_contracted_kernel_general< 7> <<<blocks, threads, 0, stream>>>(output, density, hermite_density_offsets, offsets, nprim_ij, omega, grid_points, charge_exponents); break;
    case  8: GINTfill_int3c1e_ip2_density_contracted_kernel_general< 8> <<<blocks, threads, 0, stream>>>(output, density, hermite_density_offsets, offsets, nprim_ij, omega, grid_points, charge_exponents); break;
    // Up to g + g = 8 now
    default:
        fprintf(stderr, "i_l + j_l = %d out of range\n", i_l + j_l);
        return 1;
    }

    hipError_t err = hipGetLastError();
    if (err != hipSuccess) {
        fprintf(stderr, "CUDA Error in %s: %s\n", __func__, hipGetErrorString(err));
        return 1;
    }
    return 0;
}

extern "C" {
int GINTfill_int3c1e_ip(const hipStream_t stream, const BasisProdCache* bpcache,
                        const double* grid_points, const double* charge_exponents, const int ngrids,
                        double* integrals, const int nao,
                        const int* strides, const int* ao_offsets,
                        const int* bins_locs_ij, int nbins,
                        const int cp_ij_id, const double omega)
{
    const ContractionProdType *cp_ij = bpcache->cptype + cp_ij_id;
    const int i_l = cp_ij->l_bra;
    const int j_l = cp_ij->l_ket;
    const int nrys_roots = (i_l + j_l + 1) / 2 + 1;
    const int nprim_ij = cp_ij->nprim_12;

    if (nrys_roots > MAX_NROOTS_INT3C_1E + 1) {
        fprintf(stderr, "nrys_roots = %d too high\n", nrys_roots);
        return 2;
    }

    checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(c_bpcache), bpcache, sizeof(BasisProdCache)));

    const int* bas_pairs_locs = bpcache->bas_pairs_locs;
    const int* primitive_pairs_locs = bpcache->primitive_pairs_locs;
    for (int ij_bin = 0; ij_bin < nbins; ij_bin++) {
        const int bas_ij0 = bins_locs_ij[ij_bin];
        const int bas_ij1 = bins_locs_ij[ij_bin + 1];
        const int ntasks_ij = bas_ij1 - bas_ij0;
        if (ntasks_ij <= 0) {
            continue;
        }

        BasisProdOffsets offsets;
        offsets.ntasks_ij = ntasks_ij;
        offsets.ntasks_kl = ngrids;
        offsets.bas_ij = bas_pairs_locs[cp_ij_id] + bas_ij0;
        offsets.bas_kl = -1;
        offsets.primitive_ij = primitive_pairs_locs[cp_ij_id] + bas_ij0 * nprim_ij;
        offsets.primitive_kl = -1;

        const int err = GINTfill_int3c1e_ip_tasks(integrals, offsets, i_l, j_l, nprim_ij,
                                                  strides[0], strides[1], ao_offsets[0], ao_offsets[1],
                                                  omega, grid_points, charge_exponents, stream);

        if (err != 0) {
            return err;
        }
    }

    return 0;
}

int GINTfill_int3c1e_ip1_charge_contracted(const hipStream_t stream, const BasisProdCache* bpcache,
                                           const double* grid_points, const double* charge_exponents, const int ngrids,
                                           double* integral_charge_contracted, const int nao,
                                           const int* strides, const int* ao_offsets,
                                           const int* bins_locs_ij, int nbins,
                                           const int cp_ij_id, const double omega, const int n_charge_sum_per_thread)
{
    const ContractionProdType *cp_ij = bpcache->cptype + cp_ij_id;
    const int i_l = cp_ij->l_bra;
    const int j_l = cp_ij->l_ket;
    const int nrys_roots = (i_l + j_l + 1) / 2 + 1;
    const int nprim_ij = cp_ij->nprim_12;

    if (nrys_roots > MAX_NROOTS_INT3C_1E + 1) {
        fprintf(stderr, "nrys_roots = %d too high\n", nrys_roots);
        return 2;
    }

    checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(c_bpcache), bpcache, sizeof(BasisProdCache)));

    const int* bas_pairs_locs = bpcache->bas_pairs_locs;
    const int* primitive_pairs_locs = bpcache->primitive_pairs_locs;
    for (int ij_bin = 0; ij_bin < nbins; ij_bin++) {
        const int bas_ij0 = bins_locs_ij[ij_bin];
        const int bas_ij1 = bins_locs_ij[ij_bin + 1];
        const int ntasks_ij = bas_ij1 - bas_ij0;
        if (ntasks_ij <= 0) {
            continue;
        }

        BasisProdOffsets offsets;
        offsets.ntasks_ij = ntasks_ij;
        offsets.ntasks_kl = ngrids;
        offsets.bas_ij = bas_pairs_locs[cp_ij_id] + bas_ij0;
        offsets.bas_kl = -1;
        offsets.primitive_ij = primitive_pairs_locs[cp_ij_id] + bas_ij0 * nprim_ij;
        offsets.primitive_kl = -1;

        const int err = GINTfill_int3c1e_ip1_charge_contracted_tasks(integral_charge_contracted, offsets, i_l, j_l, nprim_ij,
                                                                     strides[0], strides[1], ao_offsets[0], ao_offsets[1],
                                                                     omega, grid_points, charge_exponents, n_charge_sum_per_thread, stream);

        if (err != 0) {
            return err;
        }
    }

    return 0;
}

int GINTfill_int3c1e_ip2_density_contracted(const hipStream_t stream, const BasisProdCache* bpcache,
                                            const double* grid_points, const double* charge_exponents, const int ngrids,
                                            const double* dm_pair_ordered, const int* density_offset,
                                            double* integral_density_contracted,
                                            const int* bins_locs_ij, int nbins,
                                            const int cp_ij_id, const double omega, const int n_pair_sum_per_thread)
{
    const ContractionProdType *cp_ij = bpcache->cptype + cp_ij_id;
    const int i_l = cp_ij->l_bra;
    const int j_l = cp_ij->l_ket;
    const int nrys_roots = (i_l + j_l + 1) / 2 + 1;
    const int nprim_ij = cp_ij->nprim_12;

    if (nrys_roots > MAX_NROOTS_INT3C_1E + 1) {
        fprintf(stderr, "nrys_roots = %d too high\n", nrys_roots);
        return 2;
    }

    checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(c_bpcache), bpcache, sizeof(BasisProdCache)));

    const int* bas_pairs_locs = bpcache->bas_pairs_locs;
    const int* primitive_pairs_locs = bpcache->primitive_pairs_locs;
    for (int ij_bin = 0; ij_bin < nbins; ij_bin++) {
        const int bas_ij0 = bins_locs_ij[ij_bin];
        const int bas_ij1 = bins_locs_ij[ij_bin + 1];
        const int ntasks_ij = bas_ij1 - bas_ij0;
        if (ntasks_ij <= 0) {
            continue;
        }

        BasisProdOffsets offsets;
        offsets.ntasks_ij = ntasks_ij;
        offsets.ntasks_kl = ngrids;
        offsets.bas_ij = bas_pairs_locs[cp_ij_id] + bas_ij0;
        offsets.bas_kl = -1;
        offsets.primitive_ij = primitive_pairs_locs[cp_ij_id] + bas_ij0 * nprim_ij;
        offsets.primitive_kl = -1;

        HermiteDensityOffsets hermite_density_offsets;
        hermite_density_offsets.density_offset_of_angular_pair = density_offset[cp_ij_id];
        hermite_density_offsets.pair_offset_of_angular_pair = bas_pairs_locs[cp_ij_id];
        hermite_density_offsets.n_pair_of_angular_pair = bas_pairs_locs[cp_ij_id + 1] - bas_pairs_locs[cp_ij_id];

        const int err = GINTfill_int3c1e_ip2_density_contracted_tasks(integral_density_contracted, dm_pair_ordered, hermite_density_offsets,
                                                                      offsets, i_l, j_l, nprim_ij,
                                                                      omega, grid_points, charge_exponents, n_pair_sum_per_thread, stream);

        if (err != 0) {
            return err;
        }
    }

    return 0;
}
}
