#include "hip/hip_runtime.h"
/*
 * Copyright 2021-2025 The PySCF Developers. All Rights Reserved.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include <stdio.h>
#include <stdint.h>
#include <stdlib.h>
#include <string.h>
#include <hip/hip_runtime.h>

#include "gvhf-rys/vhf.cuh"

__device__ static
void _fill_sr_vk_tasks(int &ntasks, int &pair_kl0, uint32_t *bas_kl_idx, uint32_t bas_ij,
                       RysIntEnvVars &envs, BoundsInfo &bounds)
{
    int thread_id = threadIdx.x + blockDim.x * threadIdx.y;
    int threads = blockDim.x * blockDim.y;
    if (thread_id == 0) {
        ntasks = 0;
    }
    __syncthreads();
    int *bas = envs.bas;
    int nbas = envs.nbas;
    uint32_t *pair_kl_mapping = bounds.pair_kl_mapping;
    int ish = bas_ij / nbas;
    int jsh = bas_ij % nbas;
    float *q_cond = bounds.q_cond;
    float *s_estimator = bounds.s_estimator;
    float *dm_cond = bounds.dm_cond;
    float *diffuse_exps = s_estimator + nbas*nbas;
    double *env = envs.env;
    double *ri = env + bas[ish*BAS_SLOTS+PTR_BAS_COORD];
    double *rj = env + bas[jsh*BAS_SLOTS+PTR_BAS_COORD];
    float ai = diffuse_exps[ish];
    float aj = diffuse_exps[jsh];
    float aij = ai + aj;
    float aj_aij = aj / aij;
    float xi = ri[0];
    float yi = ri[1];
    float zi = ri[2];
    float xj = rj[0];
    float yj = rj[1];
    float zj = rj[2];
    float xjxi = xj - xi;
    float yjyi = yj - yi;
    float zjzi = zj - zi;
    float xpa = xjxi * aj_aij;
    float ypa = yjyi * aj_aij;
    float zpa = zjzi * aj_aij;
    float xij = xi + xpa;
    float yij = yi + ypa;
    float zij = zi + zpa;
    float cutoff = bounds.cutoff;
    float q_ij = q_cond[bas_ij];
    float s_ij = s_estimator[bas_ij];
    float kl_cutoff = cutoff - q_ij;
    float skl_cutoff = cutoff - s_ij;
    float omega = env[PTR_RANGE_OMEGA];
    float omega2 = omega * omega;
    float theta_ij = omega2 * aij / (aij + omega2);

    int _pair_kl0 = pair_kl0;
    for (;_pair_kl0 < bounds.npairs_kl && ntasks < QUEUE_DEPTH - 1024; _pair_kl0 += threads) {
        int pair_kl = _pair_kl0 + thread_id;
        if (pair_kl >= bounds.npairs_kl) {
            break;
        }

        int bas_kl = pair_kl_mapping[pair_kl];
        float q_kl = q_cond[bas_kl];
        if (q_kl < kl_cutoff) {
            continue;
        }
        int ksh = bas_kl / nbas;
        int lsh = bas_kl % nbas;
        float d_cutoff = kl_cutoff - q_kl;
        float dm_ik = dm_cond[ish*nbas+ksh];
        float dm_il = dm_cond[ish*nbas+lsh];
        if (dm_ik > d_cutoff || dm_il > d_cutoff) {
            double *rk = env + bas[ksh*BAS_SLOTS+PTR_BAS_COORD];
            double *rl = env + bas[lsh*BAS_SLOTS+PTR_BAS_COORD];
            float ak = diffuse_exps[ksh];
            float al = diffuse_exps[lsh];
            float akl = ak + al;
            float al_akl = al / akl;
            float xk = rk[0];
            float yk = rk[1];
            float zk = rk[2];
            float xl = rl[0];
            float yl = rl[1];
            float zl = rl[2];
            float xlxk = xl - xk;
            float ylyk = yl - yk;
            float zlzk = zl - zk;
            float xqc = xlxk * al_akl;
            float yqc = ylyk * al_akl;
            float zqc = zlzk * al_akl;
            float xkl = xk + xqc;
            float ykl = yk + yqc;
            float zkl = zk + zqc;
            float theta = theta_ij * akl / (theta_ij + akl);
            float xpq = xij - xkl;
            float ypq = yij - ykl;
            float zpq = zij - zkl;
            float rr = xpq*xpq + ypq*ypq + zpq*zpq;
            float theta_rr = logf(rr + 1.f) + theta * rr;
            float d_cutoff = skl_cutoff - s_estimator[bas_kl] + theta_rr;
            if (dm_ik > d_cutoff || dm_il > d_cutoff) {
                int off = atomicAdd(&ntasks, 1);
                bas_kl_idx[off] = bas_kl;
            }
        }
    }
    __syncthreads();
    if (thread_id == 0) {
        pair_kl0 = _pair_kl0;
    }
    if (threadIdx.y == 0) {
        bas_kl_idx[ntasks+thread_id] = pair_kl_mapping[0];
    }
    __syncthreads();
}
