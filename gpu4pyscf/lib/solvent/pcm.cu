/* Copyright 2023 The GPU4PySCF Authors. All Rights Reserved.
 *
 * This program is free software: you can redistribute it and/or modify
 * it under the terms of the GNU General Public License as published by
 * the Free Software Foundation, either version 3 of the License, or
 * (at your option) any later version.
 *
 * This program is distributed in the hope that it will be useful,
 * but WITHOUT ANY WARRANTY; without even the implied warranty of
 * MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
 * GNU General Public License for more details.
 *
 * You should have received a copy of the GNU General Public License
 * along with this program.  If not, see <http://www.gnu.org/licenses/>.
 */

#include <hip/hip_runtime.h>
#include <stdio.h>

#define THREADS        32
#define SQRT2_PI       0.7978845608028654
#define SQRT_PI        1.7724538509055159

// D and S matrix in J. Chem. Phys. 133, 244111 (2010)
__global__
static void _pcm_d_s(double *matrix_d, double *matrix_s,
                    const double *coords, const double *norm_vec, const double *r_vdw,
                    const double *charge_exp, const double *switch_fun,
                    int n)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;
    if (i >= n || j >= n){
        return;
    }

    // calculate xi
    double ei = charge_exp[i];
    double ej = charge_exp[j];
    double xi_ij = ei * ej / sqrt(ei*ei + ej*ej);

    // calculate r
    double xi = coords[3*i];
    double yi = coords[3*i+1];
    double zi = coords[3*i+2];
    double xj = coords[3*j];
    double yj = coords[3*j+1];
    double zj = coords[3*j+2];
    double dx = xi - xj;
    double dy = yi - yj;
    double dz = zi - zj;
    double rij = norm3d(dx, dy, dz);

    double xi_r_ij = xi_ij * rij;
    if (i == j) rij = 1.0;
    double s = erf(xi_r_ij) / rij;
    if (i == j) s = charge_exp[i] * SQRT2_PI / switch_fun[i];
    matrix_s[i*n+j] = s;

    if (matrix_d != NULL){
        double nxj = norm_vec[3*j];
        double nyj = norm_vec[3*j+1];
        double nzj = norm_vec[3*j+2];

        double nrij = 0.0;
        nrij += (xi - xj) * nxj;
        nrij += (yi - yj) * nyj;
        nrij += (zi - zj) * nzj;

        double rij2 = rij*rij;
        double rij3 = rij2*rij;
        double xi_r2_ij = xi_r_ij * xi_r_ij;
        double d = s * nrij / rij2 - 2.0*xi_r_ij/SQRT_PI*exp(-xi_r2_ij)*nrij/rij3;
        if (i == j) d = -charge_exp[i] * SQRT2_PI / (2.0*r_vdw[i]);
        matrix_d[i*n+j] = d;
    }
}

__global__
static void _pcm_dD_dS(double *matrix_dd, double *matrix_ds,
                    const double *coords, const double *norm_vec, const double *r_vdw,
                    const double *charge_exp, const double *switch_fun,
                    int n)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;
    if (i >= n || j >= n){
        return;
    }

    // calculate xi
    double ei = charge_exp[i];
    double ej = charge_exp[j];
    double xi_ij = ei * ej / sqrt(ei*ei + ej*ej);

    // calculate r
    double dx = coords[3*i]   - coords[3*j];
    double dy = coords[3*i+1] - coords[3*j+1];
    double dz = coords[3*i+2] - coords[3*j+2];
    double rij = norm3d(dx, dy, dz);

    double xi_r_ij = xi_ij * rij;
    double xi_r2_ij = xi_r_ij * xi_r_ij;
    if (i == j) rij = 1.0;
    double rij2 = rij*rij;

    double dS_dr = -(erf(xi_r_ij) -  2.0*xi_r_ij/ SQRT_PI * exp(-xi_r2_ij)) / rij2;
    if (i == j) dS_dr = 0.0;
    double dx_rij = dx / rij;
    double dy_rij = dy / rij;
    double dz_rij = dz / rij;

    matrix_ds[i*n+j       ] = dS_dr * dx_rij;
    matrix_ds[i*n+j +  n*n] = dS_dr * dy_rij;
    matrix_ds[i*n+j +2*n*n] = dS_dr * dz_rij;

    if (matrix_dd != NULL){
        double nxj = norm_vec[3*j];
        double nyj = norm_vec[3*j+1];
        double nzj = norm_vec[3*j+2];
        double nj_rij = dx*nxj + dy*nyj + dz*nzj;
        double rij3 = rij2*rij;
        double dD_dri = 4.0*xi_r2_ij*xi_ij / SQRT_PI*exp(-xi_r2_ij)*nj_rij/rij3;
        if (i == j) dD_dri = 0.0;

        nj_rij = 3.0*nj_rij/rij2;
        matrix_dd[i*n+j        ] = dD_dri*dx_rij + dS_dr*(-nxj/rij + nj_rij*dx_rij);
        matrix_dd[i*n+j +   n*n] = dD_dri*dy_rij + dS_dr*(-nyj/rij + nj_rij*dy_rij);
        matrix_dd[i*n+j + 2*n*n] = dD_dri*dz_rij + dS_dr*(-nzj/rij + nj_rij*dz_rij);
    }
}

extern "C" {
int pcm_d_s(hipStream_t stream, double *matrix_d, double *matrix_s,
                    const double *coords, const double *norm_vec, const double *r_vdw,
                    const double *charge_exp, const double *switch_fun,
                    int n)
{
    int ntilex = (n + THREADS - 1) / THREADS;
    int ntiley = (n + THREADS - 1) / THREADS;
    dim3 threads(THREADS, THREADS);
    dim3 blocks(ntilex, ntiley);
    _pcm_d_s<<<blocks, threads, 0, stream>>>(matrix_d, matrix_s, coords, norm_vec, r_vdw, charge_exp, switch_fun, n);
    hipError_t err = hipGetLastError();
    if (err != hipSuccess) {
        return 1;
    }
    return 0;
}

int pcm_dd_ds(hipStream_t stream, double *matrix_dD, double *matrix_dS,
                    const double *coords, const double *norm_vec, const double *r_vdw,
                    const double *charge_exp, const double *switch_fun,
                    int n)
{
    int ntilex = (n + THREADS - 1) / THREADS;
    int ntiley = (n + THREADS - 1) / THREADS;
    dim3 threads(THREADS, THREADS);
    dim3 blocks(ntilex, ntiley);
    _pcm_dD_dS<<<blocks, threads, 0, stream>>>(matrix_dD, matrix_dS, coords, norm_vec, r_vdw, charge_exp, switch_fun, n);
    hipError_t err = hipGetLastError();
    if (err != hipSuccess) {
        return 1;
    }
    return 0;
}
}
