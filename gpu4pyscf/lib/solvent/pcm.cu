/*
 * Copyright 2021-2024 The PySCF Developers. All Rights Reserved.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include <hip/hip_runtime.h>
#include <stdio.h>

#define THREADS        32
#define SQRT2_PI       0.7978845608028654
#define SQRT_PI        1.7724538509055159

// D and S matrix in J. Chem. Phys. 133, 244111 (2010)
__global__
static void _pcm_d_s(double *matrix_d, double *matrix_s,
                    const double *coords, const double *norm_vec, const double *r_vdw,
                    const double *charge_exp, const double *switch_fun,
                    int n)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;
    if (i >= n || j >= n){
        return;
    }

    // calculate xi
    double ei = charge_exp[i];
    double ej = charge_exp[j];
    double xi_ij = ei * ej / sqrt(ei*ei + ej*ej);

    // calculate r
    double xi = coords[3*i];
    double yi = coords[3*i+1];
    double zi = coords[3*i+2];
    double xj = coords[3*j];
    double yj = coords[3*j+1];
    double zj = coords[3*j+2];
    double dx = xi - xj;
    double dy = yi - yj;
    double dz = zi - zj;
    double rij = norm3d(dx, dy, dz);

    double xi_r_ij = xi_ij * rij;
    if (i == j) rij = 1.0;
    double s = erf(xi_r_ij) / rij;
    if (i == j) s = charge_exp[i] * SQRT2_PI / switch_fun[i];
    matrix_s[i*n+j] = s;

    if (matrix_d != NULL){
        double nxj = norm_vec[3*j];
        double nyj = norm_vec[3*j+1];
        double nzj = norm_vec[3*j+2];

        double nrij = 0.0;
        nrij += (xi - xj) * nxj;
        nrij += (yi - yj) * nyj;
        nrij += (zi - zj) * nzj;

        double rij2 = rij*rij;
        double rij3 = rij2*rij;
        double xi_r2_ij = xi_r_ij * xi_r_ij;
        double d = s * nrij / rij2 - 2.0*xi_r_ij/SQRT_PI*exp(-xi_r2_ij)*nrij/rij3;
        if (i == j) d = -charge_exp[i] * SQRT2_PI / (2.0*r_vdw[i]);
        matrix_d[i*n+j] = d;
    }
}

__global__
static void _pcm_dD_dS(double *matrix_dd, double *matrix_ds,
                       const double *coords, const double *norm_vec,
                       const double *charge_exp,
                       int n)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;
    if (i >= n || j >= n){
        return;
    }

    // calculate xi
    double ei = charge_exp[i];
    double ej = charge_exp[j];
    double xi_ij = ei * ej / sqrt(ei*ei + ej*ej);

    // calculate r
    double dx = coords[3*i]   - coords[3*j];
    double dy = coords[3*i+1] - coords[3*j+1];
    double dz = coords[3*i+2] - coords[3*j+2];
    double rij = norm3d(dx, dy, dz);

    double xi_r_ij = xi_ij * rij;
    double xi_r2_ij = xi_r_ij * xi_r_ij;
    if (i == j) rij = 1.0;
    double rij2 = rij*rij;

    double dS_dr = -(erf(xi_r_ij) -  2.0*xi_r_ij/ SQRT_PI * exp(-xi_r2_ij)) / rij2;
    if (i == j) dS_dr = 0.0;
    double dx_rij = dx / rij;
    double dy_rij = dy / rij;
    double dz_rij = dz / rij;

    matrix_ds[i*n+j       ] = dS_dr * dx_rij;
    matrix_ds[i*n+j +  n*n] = dS_dr * dy_rij;
    matrix_ds[i*n+j +2*n*n] = dS_dr * dz_rij;

    if (matrix_dd != NULL){
        double nxj = norm_vec[3*j];
        double nyj = norm_vec[3*j+1];
        double nzj = norm_vec[3*j+2];
        double nj_rij = dx*nxj + dy*nyj + dz*nzj;
        double rij3 = rij2*rij;
        double dD_dri = 4.0*xi_r2_ij*xi_ij / SQRT_PI*exp(-xi_r2_ij)*nj_rij/rij3;
        if (i == j) dD_dri = 0.0;

        nj_rij = 3.0*nj_rij/rij2;
        matrix_dd[i*n+j        ] = dD_dri*dx_rij + dS_dr*(-nxj/rij + nj_rij*dx_rij);
        matrix_dd[i*n+j +   n*n] = dD_dri*dy_rij + dS_dr*(-nyj/rij + nj_rij*dy_rij);
        matrix_dd[i*n+j + 2*n*n] = dD_dri*dz_rij + dS_dr*(-nzj/rij + nj_rij*dz_rij);
    }
}

__global__
static void _pcm_left_multiply_dS(double *output, const double* right_vector,
                                  const double *coords, const double *charge_exp,
                                  int n)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i >= n) {
        return;
    }

    const double rix = coords[3*i  ];
    const double riy = coords[3*i+1];
    const double riz = coords[3*i+2];
    const double ei = charge_exp[i];

    double sum_x = 0.0;
    double sum_y = 0.0;
    double sum_z = 0.0;
    for (int j = threadIdx.y; j < n; j += blockDim.y) {
        // calculate xi
        const double ej = charge_exp[j];
        const double xi_ij = ei * ej / sqrt(ei*ei + ej*ej);

        // calculate r
        const double dx = rix - coords[3*j  ];
        const double dy = riy - coords[3*j+1];
        const double dz = riz - coords[3*j+2];
        double rij = norm3d(dx, dy, dz);

        const double xi_r_ij = xi_ij * rij;
        const double xi_r2_ij = xi_r_ij * xi_r_ij;
        if (i == j) rij = 1.0;
        const double rij2 = rij*rij;

        double dS_dr = -(erf(xi_r_ij) -  2.0*xi_r_ij/ SQRT_PI * exp(-xi_r2_ij)) / rij2;
        if (i == j) dS_dr = 0.0;
        const double dx_rij = dx / rij;
        const double dy_rij = dy / rij;
        const double dz_rij = dz / rij;

        const double dSx = dS_dr * dx_rij;
        const double dSy = dS_dr * dy_rij;
        const double dSz = dS_dr * dz_rij;

        const double right_vector_j = right_vector[j];
        sum_x += dSx * right_vector_j;
        sum_y += dSy * right_vector_j;
        sum_z += dSz * right_vector_j;
    }

    __shared__ double sum_shared[THREADS * THREADS];

    sum_shared[threadIdx.y * THREADS + threadIdx.x] = sum_x;
    __syncthreads();
    for (int stride = THREADS / 2; stride > 0; stride >>= 1) {
        if (threadIdx.y < stride) {
            sum_shared[threadIdx.y * THREADS + threadIdx.x] += sum_shared[(threadIdx.y + stride) * THREADS + threadIdx.x];
        }
        __syncthreads();
    }
    if (threadIdx.y == 0) {
        output[        i] = sum_shared[threadIdx.x];
    }

    sum_shared[threadIdx.y * THREADS + threadIdx.x] = sum_y;
    __syncthreads();
    for (int stride = THREADS / 2; stride > 0; stride >>= 1) {
        if (threadIdx.y < stride) {
            sum_shared[threadIdx.y * THREADS + threadIdx.x] += sum_shared[(threadIdx.y + stride) * THREADS + threadIdx.x];
        }
        __syncthreads();
    }
    if (threadIdx.y == 0) {
        output[n     + i] = sum_shared[threadIdx.x];
    }

    sum_shared[threadIdx.y * THREADS + threadIdx.x] = sum_z;
    __syncthreads();
    for (int stride = THREADS / 2; stride > 0; stride >>= 1) {
        if (threadIdx.y < stride) {
            sum_shared[threadIdx.y * THREADS + threadIdx.x] += sum_shared[(threadIdx.y + stride) * THREADS + threadIdx.x];
        }
        __syncthreads();
    }
    if (threadIdx.y == 0) {
        output[n * 2 + i] = sum_shared[threadIdx.x];
    }
}

__global__
static void _pcm_d2D_d2S(double *matrix_d2D, double *matrix_d2S,
                         const double *coords, const double *norm_vec,
                         const double *charge_exp,
                         int n)
{
    const int i = blockIdx.x * blockDim.x + threadIdx.x;
    const int j = blockIdx.y * blockDim.y + threadIdx.y;
    if (i >= n || j >= n) {
        return;
    }

    // calculate xi
    const double ei = charge_exp[i];
    const double ej = charge_exp[j];
    const double eij = ei * ej / sqrt(ei*ei + ej*ej);

    // calculate r
    const double dx = coords[3*i]   - coords[3*j];
    const double dy = coords[3*i+1] - coords[3*j+1];
    const double dz = coords[3*i+2] - coords[3*j+2];
    const double rij = norm3d(dx, dy, dz);
    const double rij_1 = (i != j) ? (1.0 / rij) : 0.0; // This guarantees that if i == j, all matrix elements = 0
    const double rij_2 = rij_1 * rij_1;
    const double rij_3 = rij_2 * rij_1;
    const double rij_4 = rij_2 * rij_2;
    const double rij_5 = rij_2 * rij_3;
    const double eij2 = eij * eij;

    const double eij_rij = eij * rij;
    const double erf_eij_rij = erf(eij_rij);
    const double exp_minus_eij2_rij2 = exp(-eij_rij * eij_rij);
    const double two_eij_over_sqrt_pi = 2.0 * eij / SQRT_PI;
    const double two_eij_over_sqrt_pi_exp_minus_eij2_rij2 = exp_minus_eij2_rij2 * two_eij_over_sqrt_pi;

    const double S_direct_product_prefactor = -two_eij_over_sqrt_pi_exp_minus_eij2_rij2 * (3 * rij_4 + 2 * eij2 * rij_2)
                                              + 3 * rij_5 * erf_eij_rij;
    const double S_xyz_diagonal_prefactor = two_eij_over_sqrt_pi_exp_minus_eij2_rij2 * rij_2 - rij_3 * erf_eij_rij;

    const int n2 = n * n;
    matrix_d2S[i*n + j         ] = dx * dx * S_direct_product_prefactor + S_xyz_diagonal_prefactor;
    matrix_d2S[i*n + j + n2    ] = dx * dy * S_direct_product_prefactor;
    matrix_d2S[i*n + j + n2 * 2] = dx * dz * S_direct_product_prefactor;
    matrix_d2S[i*n + j + n2 * 3] = dy * dx * S_direct_product_prefactor;
    matrix_d2S[i*n + j + n2 * 4] = dy * dy * S_direct_product_prefactor + S_xyz_diagonal_prefactor;
    matrix_d2S[i*n + j + n2 * 5] = dy * dz * S_direct_product_prefactor;
    matrix_d2S[i*n + j + n2 * 6] = dz * dx * S_direct_product_prefactor;
    matrix_d2S[i*n + j + n2 * 7] = dz * dy * S_direct_product_prefactor;
    matrix_d2S[i*n + j + n2 * 8] = dz * dz * S_direct_product_prefactor + S_xyz_diagonal_prefactor;

    if (matrix_d2D != NULL) {
        const double nxj = norm_vec[3*j];
        const double nyj = norm_vec[3*j+1];
        const double nzj = norm_vec[3*j+2];
        const double nj_rij = dx * nxj + dy * nyj + dz * nzj;

        const double eij4 = eij2 * eij2;
        const double rij_6 = rij_4 * rij_2;
        const double rij_7 = rij_4 * rij_3;

        const double D_direct_product_prefactor = (-two_eij_over_sqrt_pi_exp_minus_eij2_rij2 * (15 * rij_6 + 10 * eij2 * rij_4 + 4 * eij4 * rij_2)
                                                   + 15 * rij_7 * erf_eij_rij) * nj_rij;
        matrix_d2D[i*n + j         ] = D_direct_product_prefactor * dx * dx - S_direct_product_prefactor * (dx * nxj + dx * nxj + nj_rij);
        matrix_d2D[i*n + j + n2    ] = D_direct_product_prefactor * dx * dy - S_direct_product_prefactor * (dy * nxj + dx * nyj);
        matrix_d2D[i*n + j + n2 * 2] = D_direct_product_prefactor * dx * dz - S_direct_product_prefactor * (dz * nxj + dx * nzj);
        matrix_d2D[i*n + j + n2 * 3] = D_direct_product_prefactor * dy * dx - S_direct_product_prefactor * (dx * nyj + dy * nxj);
        matrix_d2D[i*n + j + n2 * 4] = D_direct_product_prefactor * dy * dy - S_direct_product_prefactor * (dy * nyj + dy * nyj + nj_rij);
        matrix_d2D[i*n + j + n2 * 5] = D_direct_product_prefactor * dy * dz - S_direct_product_prefactor * (dz * nyj + dy * nzj);
        matrix_d2D[i*n + j + n2 * 6] = D_direct_product_prefactor * dz * dx - S_direct_product_prefactor * (dx * nzj + dz * nxj);
        matrix_d2D[i*n + j + n2 * 7] = D_direct_product_prefactor * dz * dy - S_direct_product_prefactor * (dy * nzj + dz * nyj);
        matrix_d2D[i*n + j + n2 * 8] = D_direct_product_prefactor * dz * dz - S_direct_product_prefactor * (dz * nzj + dz * nzj + nj_rij);
    }
}

__global__
static void _pcm_d2F_to_d2Sii(const double* F, const double* dF, const double* d2F, const double* charge_exp,
                              double* d2Sii, const int n_atom, const int n_grid)
{
    const int i_grid = blockIdx.x * blockDim.x + threadIdx.x;
    const int ij_atom = blockIdx.y * blockDim.y + threadIdx.y;
    if (i_grid >= n_grid || ij_atom >= n_atom * n_atom) {
        return;
    }

    const int i_atom = ij_atom / n_atom;
    const int j_atom = ij_atom % n_atom;

    const double zeta = charge_exp[i_grid];
    const double F_value = F[i_grid];
    const double F_1 = 1.0 / F_value;
    const double F_2 = F_1 * F_1;
    const double combined_factor = SQRT2_PI * zeta * F_2;

    const double dFix = dF[(i_atom * 3    ) * n_grid + i_grid];
    const double dFiy = dF[(i_atom * 3 + 1) * n_grid + i_grid];
    const double dFiz = dF[(i_atom * 3 + 2) * n_grid + i_grid];
    const double dFjx = dF[(j_atom * 3    ) * n_grid + i_grid];
    const double dFjy = dF[(j_atom * 3 + 1) * n_grid + i_grid];
    const double dFjz = dF[(j_atom * 3 + 2) * n_grid + i_grid];

    const double d2Fixjx = d2F[((i_atom * n_atom + j_atom) * 9 + 0 * 3    ) * n_grid + i_grid];
    const double d2Fixjy = d2F[((i_atom * n_atom + j_atom) * 9 + 0 * 3 + 1) * n_grid + i_grid];
    const double d2Fixjz = d2F[((i_atom * n_atom + j_atom) * 9 + 0 * 3 + 2) * n_grid + i_grid];
    const double d2Fiyjx = d2F[((i_atom * n_atom + j_atom) * 9 + 1 * 3    ) * n_grid + i_grid];
    const double d2Fiyjy = d2F[((i_atom * n_atom + j_atom) * 9 + 1 * 3 + 1) * n_grid + i_grid];
    const double d2Fiyjz = d2F[((i_atom * n_atom + j_atom) * 9 + 1 * 3 + 2) * n_grid + i_grid];
    const double d2Fizjx = d2F[((i_atom * n_atom + j_atom) * 9 + 2 * 3    ) * n_grid + i_grid];
    const double d2Fizjy = d2F[((i_atom * n_atom + j_atom) * 9 + 2 * 3 + 1) * n_grid + i_grid];
    const double d2Fizjz = d2F[((i_atom * n_atom + j_atom) * 9 + 2 * 3 + 2) * n_grid + i_grid];

    d2Sii[((i_atom * n_atom + j_atom) * 9 + 0 * 3    ) * n_grid + i_grid] = combined_factor * (2 * F_1 * dFix * dFjx - d2Fixjx);
    d2Sii[((i_atom * n_atom + j_atom) * 9 + 0 * 3 + 1) * n_grid + i_grid] = combined_factor * (2 * F_1 * dFix * dFjy - d2Fixjy);
    d2Sii[((i_atom * n_atom + j_atom) * 9 + 0 * 3 + 2) * n_grid + i_grid] = combined_factor * (2 * F_1 * dFix * dFjz - d2Fixjz);
    d2Sii[((i_atom * n_atom + j_atom) * 9 + 1 * 3    ) * n_grid + i_grid] = combined_factor * (2 * F_1 * dFiy * dFjx - d2Fiyjx);
    d2Sii[((i_atom * n_atom + j_atom) * 9 + 1 * 3 + 1) * n_grid + i_grid] = combined_factor * (2 * F_1 * dFiy * dFjy - d2Fiyjy);
    d2Sii[((i_atom * n_atom + j_atom) * 9 + 1 * 3 + 2) * n_grid + i_grid] = combined_factor * (2 * F_1 * dFiy * dFjz - d2Fiyjz);
    d2Sii[((i_atom * n_atom + j_atom) * 9 + 2 * 3    ) * n_grid + i_grid] = combined_factor * (2 * F_1 * dFiz * dFjx - d2Fizjx);
    d2Sii[((i_atom * n_atom + j_atom) * 9 + 2 * 3 + 1) * n_grid + i_grid] = combined_factor * (2 * F_1 * dFiz * dFjy - d2Fizjy);
    d2Sii[((i_atom * n_atom + j_atom) * 9 + 2 * 3 + 2) * n_grid + i_grid] = combined_factor * (2 * F_1 * dFiz * dFjz - d2Fizjz);
}

extern "C" {
int pcm_d_s(hipStream_t stream, double *matrix_d, double *matrix_s,
                    const double *coords, const double *norm_vec, const double *r_vdw,
                    const double *charge_exp, const double *switch_fun,
                    int n)
{
    int ntilex = (n + THREADS - 1) / THREADS;
    int ntiley = (n + THREADS - 1) / THREADS;
    dim3 threads(THREADS, THREADS);
    dim3 blocks(ntilex, ntiley);
    _pcm_d_s<<<blocks, threads, 0, stream>>>(matrix_d, matrix_s, coords, norm_vec, r_vdw, charge_exp, switch_fun, n);
    hipError_t err = hipGetLastError();
    if (err != hipSuccess) {
        return 1;
    }
    return 0;
}

int pcm_dd_ds(hipStream_t stream, double *matrix_dD, double *matrix_dS,
              const double *coords, const double *norm_vec,
              const double *charge_exp,
              int n)
{
    int ntilex = (n + THREADS - 1) / THREADS;
    int ntiley = (n + THREADS - 1) / THREADS;
    dim3 threads(THREADS, THREADS);
    dim3 blocks(ntilex, ntiley);
    _pcm_dD_dS<<<blocks, threads, 0, stream>>>(matrix_dD, matrix_dS, coords, norm_vec, charge_exp, n);
    hipError_t err = hipGetLastError();
    if (err != hipSuccess) {
        return 1;
    }
    return 0;
}

int pcm_left_multiply_ds(const hipStream_t stream, double *output, const double *right_vector,
                         const double *coords, const double *charge_exp,
                         int n)
{
    int ntilex = (n + THREADS - 1) / THREADS;
    dim3 threads(THREADS, THREADS);
    dim3 blocks(ntilex, 1);
    _pcm_left_multiply_dS<<<blocks, threads, 0, stream>>>(output, right_vector, coords, charge_exp, n);
    hipError_t err = hipGetLastError();
    if (err != hipSuccess) {
        return 1;
    }
    return 0;
}

int pcm_d2d_d2s(hipStream_t stream, double *matrix_d2D, double *matrix_d2S,
                const double *coords, const double *norm_vec,
                const double *charge_exp,
                int n)
{
    const int ntilex = (n + THREADS - 1) / THREADS;
    const int ntiley = (n + THREADS - 1) / THREADS;
    const dim3 threads(THREADS, THREADS);
    const dim3 blocks(ntilex, ntiley);
    _pcm_d2D_d2S<<<blocks, threads, 0, stream>>>(matrix_d2D, matrix_d2S, coords, norm_vec, charge_exp, n);
    hipError_t err = hipGetLastError();
    if (err != hipSuccess) {
        return 1;
    }
    return 0;
}

int pcm_d2f_to_d2sii(hipStream_t stream, const double* F, const double* dF, const double* d2F, const double* charge_exp,
                     double* d2Sii, const int n_atom, const int n_grid)
{
    const int ntilex = (n_grid + THREADS - 1) / THREADS;
    const int ntiley = (n_atom * n_atom + THREADS - 1) / THREADS;
    const dim3 threads(THREADS, THREADS);
    const dim3 blocks(ntilex, ntiley);
    _pcm_d2F_to_d2Sii<<<blocks, threads, 0, stream>>>(F, dF, d2F, charge_exp, d2Sii, n_atom, n_grid);
    hipError_t err = hipGetLastError();
    if (err != hipSuccess) {
        return 1;
    }
    return 0;
}
}
